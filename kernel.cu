#include "hip/hip_runtime.h"
#include "kernel.h"
#include "structs.h"
#include "params.h"

#include <math.h>

#include <hip/hip_cooperative_groups.h>

#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>

#define __STDC_FORMAT_MACROS
#include <inttypes.h>

//namespace cg = cooperative_groups;
using namespace cooperative_groups;



__device__ void print(unsigned int tid, unsigned int value)
{
	if(0 == tid)
	{
		printf("threadIdx.x 0, value = %d\n", value);
	}
}



__global__ void sortByWorkLoadGlobal(
		DTYPE * database,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		schedulingCell * sortedCells)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(*nNonEmptyCells <= tid)
	{
		return;
	}

	unsigned int cell = gridCellLookupArr[tid].idx;
	unsigned int nbNeighborPoints = 0;
	unsigned int tmpId = indexLookupArr[ index[cell].indexmin ];

	DTYPE point[GPUNUMDIM];
	for(int i = 0; i < GPUNUMDIM; ++i)
	{
			point[i] = database[tmpId * GPUNUMDIM + i];
	}

	unsigned int nDCellIDs[NUMINDEXEDDIM];

	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for(int n = 0; n < NUMINDEXEDDIM; n++)
	{
		nDCellIDs[n] = (point[n] - minArr[n]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[n] - 1);;
		unsigned int nDMaxCellIDs = min(nCells[n] - 1, nDCellIDs[n] + 1);

		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		rangeFilteredCellIdsMin[n] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[n] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{
			for (int x = 0; x < NUMINDEXEDDIM; x++){
				indexes[x] = loopRng[x];
			}

			uint64_t cellID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			struct gridCellLookup tmp;
			tmp.gridLinearID = cellID;
			if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
			{
				struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
				unsigned int GridIndex = resultBinSearch->idx;
				nbNeighborPoints += index[GridIndex].indexmax - index[GridIndex].indexmin + 1;

			}

		}

	sortedCells[tid].nbPoints = nbNeighborPoints;
	sortedCells[tid].cellId = cell;

}



/******************************************************************************/


//TODO use the unicomp pattern
__global__ void sortByWorkLoadUnicomp(
		DTYPE * database,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE* minArr,
		unsigned int * nCells,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		schedulingCell * sortedCells)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(*nNonEmptyCells <= tid)
	{
		return;
	}

	int cell = gridCellLookupArr[tid].idx;
	int nbNeighborPoints = 0;
	int tmpId = indexLookupArr[ index[cell].indexmin ];

	DTYPE point[NUMINDEXEDDIM];

	unsigned int nDCellIDs[NUMINDEXEDDIM];

	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for(int n = 0; n < NUMINDEXEDDIM; n++)
	{
		point[n] = database[tmpId * NUMINDEXEDDIM + n];
		nDCellIDs[n] = (point[n] - minArr[n]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[n] - 1);;
		unsigned int nDMaxCellIDs = min(nCells[n] - 1, nDCellIDs[n] + 1);

		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		if (1 == foundMin && 1 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs;
			rangeFilteredCellIdsMax[n] = nDMaxCellIDs;
			//printf("\nmin and max");
		}
		else if (1 == foundMin && 0 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs;
			rangeFilteredCellIdsMax[n] = nDMinCellIDs + 1;
			//printf("\nmin not max");
		}
		else if (0 == foundMin && 1 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs + 1;
			rangeFilteredCellIdsMax[n] = nDMaxCellIDs;
			//printf("\nmax not min");
		}
		else{
			//printf("\nneither");
			rangeFilteredCellIdsMin[n] = nDMinCellIDs + 1;
			rangeFilteredCellIdsMax[n] = nDMinCellIDs + 1;
		}
	}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{
			for (int x = 0; x < NUMINDEXEDDIM; x++){
				indexes[x] = loopRng[x];
			}

			uint64_t cellID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			struct gridCellLookup tmp;
			tmp.gridLinearID = cellID;
			if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
			{
				struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
				unsigned int GridIndex = resultBinSearch->idx;
				nbNeighborPoints += index[GridIndex].indexmax - index[GridIndex].indexmin + 1;
			}

		}

	sortedCells[tid].nbPoints = nbNeighborPoints;
	sortedCells[tid].cellId = cell;

}



/******************************************************************************/



__global__ void sortByWorkLoadLidUnicomp(
		DTYPE* database,
		DTYPE* epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE* minArr,
		unsigned int * nCells,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		schedulingCell * sortedCells)
{

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(*nNonEmptyCells <= tid)
	{
		return;
	}

	int cell = gridCellLookupArr[tid].idx;
	int nbNeighborPoints = 0;
	int tmpId = indexLookupArr[ index[cell].indexmin ];

	DTYPE point[NUMINDEXEDDIM];

	unsigned int nDCellIDs[NUMINDEXEDDIM];

	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for(int n = 0; n < NUMINDEXEDDIM; n++)
	{
		point[n] = database[tmpId * NUMINDEXEDDIM + n];
		nDCellIDs[n] = (point[n] - minArr[n]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[n] - 1);;
		unsigned int nDMaxCellIDs = min(nCells[n] - 1, nDCellIDs[n] + 1);

		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (n * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		if (1 == foundMin && 1 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs;
			rangeFilteredCellIdsMax[n] = nDMaxCellIDs;
			//printf("\nmin and max");
		}
		else if (1 == foundMin && 0 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs;
			rangeFilteredCellIdsMax[n] = nDMinCellIDs + 1;
			//printf("\nmin not max");
		}
		else if (0 == foundMin && 1 == foundMax){
			rangeFilteredCellIdsMin[n] = nDMinCellIDs + 1;
			rangeFilteredCellIdsMax[n] = nDMaxCellIDs;
			//printf("\nmax not min");
		}
		else{
			//printf("\nneither");
			rangeFilteredCellIdsMin[n] = nDMinCellIDs + 1;
			rangeFilteredCellIdsMax[n] = nDMinCellIDs + 1;
		}
	}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (int x = 0; x < NUMINDEXEDDIM; x++){
		indexes[x] = nDCellIDs[x];
	}

	uint64_t originCellID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{
			for (int x = 0; x < NUMINDEXEDDIM; x++){
				indexes[x] = loopRng[x];
			}

			uint64_t cellID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			if(originCellID <= cellID)
			{
				struct gridCellLookup tmp;
				tmp.gridLinearID = cellID;
				if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
				{
					struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
					unsigned int GridIndex = resultBinSearch->idx;
					nbNeighborPoints += index[GridIndex].indexmax - index[GridIndex].indexmin + 1;
				}
			}

		}

	sortedCells[tid].nbPoints = nbNeighborPoints;
	sortedCells[tid].cellId = cell;

}



/******************************************************************************/



__device__ uint64_t getLinearID_nDimensionsGPU(
		unsigned int * indexes,
		unsigned int * dimLen,
		unsigned int nDimensions)
{
    uint64_t offset = 0;
	uint64_t multiplier = 1;

	for (int i = 0; i < nDimensions; i++)
	{
		offset += (uint64_t) indexes[i] * multiplier;
		multiplier *= dimLen[i];
	}

	return offset;
}



/******************************************************************************/



__forceinline__ __device__ void evalPoint(
		unsigned int* indexLookupArr,
		int k,
		DTYPE* database,
		DTYPE* epsilon,
		DTYPE* point,
		unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx,
		bool differentCell)
{
	// unsigned int tid = blockIdx.x * BLOCKSIZE + threadIdx.x;

	DTYPE runningTotalDist = 0;
	unsigned int dataIdx = indexLookupArr[k];

	for(int l = 0; l < GPUNUMDIM; l++){
		runningTotalDist += ( database[dataIdx * GPUNUMDIM + l] - point[l])
				* (database[dataIdx * GPUNUMDIM + l] - point[l] );
	}

	if(sqrt(runningTotalDist) <= (*epsilon)){
	//if(runningTotalDist <= ((*epsilon) * (*epsilon))){
		unsigned int idx = atomicAdd(cnt, int(1));
		pointIDKey[idx] = pointIdx; // --> HERE
		pointInDistVal[idx] = dataIdx;

		if(differentCell)
		{
			unsigned int idx = atomicAdd(cnt, int(1));
			pointIDKey[idx] = dataIdx;
			// pointIDKey[tid] = dataIdx;
			pointInDistVal[idx] = pointIdx;
			// pointInDistVal[tid] = pointIdx;
		}
	}
}



/******************************************************************************/



__device__ void evaluateCell(
		unsigned int* nCells,
		unsigned int* indexes,
		struct gridCellLookup * gridCellLookupArr,
		unsigned int* nNonEmptyCells,
		DTYPE* database, DTYPE* epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		DTYPE* point, unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx,
		bool differentCell,
		unsigned int* nDCellIDs)
{
	//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
	//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)
	uint64_t calcLinearID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	struct gridCellLookup tmp;
	tmp.gridLinearID = calcLinearID;
	//find if the cell is non-empty
	if(thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
	{
		//compute the neighbors for the adjacent non-empty cell
		struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
		unsigned int GridIndex = resultBinSearch->idx;

		for(int k = index[GridIndex].indexmin; k <= index[GridIndex].indexmax; k++){
			evalPoint(indexLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx, differentCell);
		}
	}
}



/******************************************************************************/



__forceinline__ __device__ void evalPointUnicompOrigin(
		unsigned int* indexLookupArr,
		int k,
		DTYPE* database,
		DTYPE* epsilon,
		DTYPE* point,
		unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx)
{
	DTYPE runningTotalDist = 0;
	unsigned int dataIdx = indexLookupArr[k];

	for (int l = 0; l < GPUNUMDIM; l++)
	{
		runningTotalDist += (database[dataIdx * GPUNUMDIM + l] - point[l]) * (database[dataIdx * GPUNUMDIM + l] - point[l]);
	}

	if (sqrt(runningTotalDist) <= (*epsilon)){
	//if(runningTotalDist <= ((*epsilon) * (*epsilon))){
		unsigned int idx = atomicAdd(cnt, int(1));
		// assert(idx < 2000000);
		pointIDKey[idx] = pointIdx; // --> HERE
		pointInDistVal[idx] = dataIdx;
	}
}



/******************************************************************************/



__device__ void evaluateCellUnicompOrigin(
		unsigned int* nCells,
		unsigned int* indexes,
		struct gridCellLookup * gridCellLookupArr,
		unsigned int* nNonEmptyCells,
		DTYPE* database, DTYPE* epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		DTYPE* point, unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx,
		unsigned int* nDCellIDs,
		unsigned int nbThreads,
		unsigned int numThread)
{
	//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
	//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)
	uint64_t calcLinearID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	struct gridCellLookup tmp;
	tmp.gridLinearID = calcLinearID;
	//find if the cell is non-empty
	if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
	{
		//compute the neighbors for the adjacent non-empty cell
		struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
		unsigned int GridIndex = resultBinSearch->idx;

		int begin = index[GridIndex].indexmin;
		int end = index[GridIndex].indexmax;
		int nbElem = end - begin + 1;
		if(numThread < nbElem)
		{
			int size = nbElem / nbThreads;
			int oneMore = nbElem - (size * nbThreads);
			if(nbElem == (size * nbThreads))
			{
				begin += size * numThread;
				end = begin + size - 1;
			}else{
				begin += numThread * size + ((numThread < oneMore)?numThread:oneMore);
				end = begin + size - 1 + (numThread < oneMore);
			}

			for(int k = begin; k <= end; k++)
			{
				evalPointUnicompOrigin(indexLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			}
		}
	}
}



/******************************************************************************/



__forceinline__ __device__ void evalPointUnicompAdjacent(
		unsigned int* indexLookupArr,
		int k,
		DTYPE* database,
		DTYPE* epsilon,
		DTYPE* point,
		unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx)
{
	DTYPE runningTotalDist = 0;
	unsigned int dataIdx = indexLookupArr[k];

	for (int l = 0; l < GPUNUMDIM; l++)
	{
		runningTotalDist += (database[dataIdx * GPUNUMDIM + l] - point[l]) * (database[dataIdx * GPUNUMDIM + l] - point[l]);
	}

	if (sqrt(runningTotalDist) <= (*epsilon)){
	//if(runningTotalDist <= ((*epsilon) * (*epsilon))){
		unsigned int idx = atomicAdd(cnt, int(2));
		pointIDKey[idx] = pointIdx;
		pointInDistVal[idx] = dataIdx;
		pointIDKey[idx + 1] = dataIdx;
		pointInDistVal[idx + 1] = pointIdx;
	}
}



/******************************************************************************/



__device__ void evaluateCellUnicompAdjacent(
		unsigned int* nCells,
		unsigned int* indexes,
		struct gridCellLookup * gridCellLookupArr,
		unsigned int* nNonEmptyCells,
		DTYPE* database, DTYPE* epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		DTYPE* point, unsigned int* cnt,
		int* pointIDKey,
		int* pointInDistVal,
		int pointIdx,
		unsigned int* nDCellIDs,
		unsigned int nbThreads,
		unsigned int numThread)
{
	//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
	//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)
	uint64_t calcLinearID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);

	struct gridCellLookup tmp;
	tmp.gridLinearID = calcLinearID;
	//find if the cell is non-empty
	if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
	{
		//compute the neighbors for the adjacent non-empty cell
		struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
		unsigned int GridIndex = resultBinSearch->idx;

		int begin = index[GridIndex].indexmin;
		int end = index[GridIndex].indexmax;
		int nbElem = end - begin + 1;
		if(numThread < nbElem)
		{
			int size = nbElem / nbThreads;
			int oneMore = nbElem - (size * nbThreads);
			if(nbElem == (size * nbThreads))
			{
				begin += size * numThread;
				end = begin + size - 1;
			}else{
				begin += numThread * size + ((numThread < oneMore)?numThread:oneMore);
				end = begin + size - 1 + (numThread < oneMore);
			}

			for(int k = begin; k <= end; k++)
			{
				evalPointUnicompAdjacent(indexLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			}
		}
	}
}



/******************************************************************************/



// used to represent the head of the "queue" when sampling the dataset
__device__ int counterEstimator = 0;

__global__ void kernelNDGridIndexBatchEstimatorAdaptive(
		unsigned int sampleBegin,
		unsigned int sampleEnd,
		unsigned int * N,
		unsigned int * sampleOffset,
		DTYPE * database,
		DTYPE * sortedDatabase,
		unsigned int * originPointIndex,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets)
{

	unsigned int tid = blockIdx.x * BLOCKSIZE + threadIdx.x;

	if((*N) <= tid)
	{
		return;
	}

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; i++)
	{
			point[i] = database[ originPointIndex[tid] * GPUNUMDIM + i ];
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; i++)
	{

		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)


		///////////////////////////
		//Take the intersection of the ranges for each dimension between
		//the point and the filtered set of cells in each dimension
		//Ranges in a given dimension that have points in them that are non-empty in a dimension will be tested
		///////////////////////////

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		//we go throgh each dimension and compare the range of the query points min/max cell ids to the filtered ones
		//find out which ones in the range exist based on the min/max
		//then determine the appropriate ranges

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		// cases:
		// found the min and max
		// found the min and not max
		//found the max and not the min
		//you don't find the min or max -- then only check the mid
		//you always find the mid because it's in the cell of the point you're looking for

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	///////////////////////////////////////
	//End taking intersection
	//////////////////////////////////////

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; ++x)
			{
				indexes[x] = loopRng[x];
			}

			uint64_t calcLinearID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
			//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)

			struct gridCellLookup tmp;
			tmp.gridLinearID = calcLinearID;

			if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
			{
				//in the GPU implementation we go directly to computing neighbors so that we don't need to
				//store a buffer of the cells to check
				//cellsToCheck->push_back(calcLinearID);

				//HERE WE COMPUTE THE NEIGHBORS FOR THE CELL
				//XXXXXXXXXXXXXXXXXXXXXXXXX

				struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr+(*nNonEmptyCells), gridCellLookup(tmp));
				unsigned int GridIndex = resultBinSearch->idx;

				for (int k = index[GridIndex].indexmin; k <= index[GridIndex].indexmax; ++k)
				{
					DTYPE runningTotalDist = 0;
					unsigned int dataIdx = indexLookupArr[k];

					for (int l = 0; l < GPUNUMDIM; ++l)
					{
						runningTotalDist += (database[dataIdx * GPUNUMDIM + l]  - point[l])
								* (database[dataIdx * GPUNUMDIM + l] - point[l]);
					}

					if (sqrt(runningTotalDist) <= (*epsilon))
					{
						unsigned int idx = atomicAdd(cnt, int(1));
					}
				}
			}
		} //end loop body

}



__global__ void kernelNDGridIndexBatchEstimatorUnicompAdaptive(
		unsigned int sampleBegin,
		unsigned int sampleEnd,
		unsigned int * N,
		unsigned int * sampleOffset,
		DTYPE * database,
		DTYPE * sortedDatabase,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets)
{

	unsigned int tid = blockIdx.x * BLOCKSIZE + threadIdx.x;

	if((*N) <= tid)
	{
		return;
	}

	#if SORT_BY_WORKLOAD
		unsigned int pointID = atomicAdd(&counterEstimator, int(1));
		pointID = pointID * (*sampleOffset) * GPUNUMDIM;
	#else
		unsigned int pointID = tid  * (*sampleOffset) * GPUNUMDIM;
	#endif

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; ++i){
		#if SORT_BY_WORKLOAD
			point[i] = sortedDatabase[pointID + i];
		#else
			point[i] = database[pointID + i];
		#endif
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];

	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; ++i)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)


		///////////////////////////
		//Take the intersection of the ranges for each dimension between
		//the point and the filtered set of cells in each dimension
		//Ranges in a given dimension that have points in them that are non-empty in a dimension will be tested
		///////////////////////////

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		//we go throgh each dimension and compare the range of the query points min/max cell ids to the filtered ones
		//find out which ones in the range exist based on the min/max
		//then determine the appropriate ranges

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin=1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax=1;
		}

		// cases:
		// found the min and max
		// found the min and not max
		// found the max and not the min
		// you don't find the min or max -- then only check the mid
		// you always find the mid because it's in the cell of the point you're looking for

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	///////////////////////////////////////
	//End taking intersection
	//////////////////////////////////////

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for(int i = 0; i < NUMINDEXEDDIM; ++i)
	{
		indexes[i] = nDCellIDs[i];
	}

	#include "stamploopsEstimator.h"

}



__global__ void kernelNDGridIndexBatchEstimatorLidUnicompAdaptive(
		unsigned int sampleBegin,
		unsigned int sampleEnd,
		unsigned int * N,
		unsigned int * sampleOffset,
		DTYPE * database,
		DTYPE * sortedDatabase,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets)
{

	unsigned int tid = blockIdx.x * BLOCKSIZE + threadIdx.x;

	if((*N) <= tid)
	{
		return;
	}

	#if SORT_BY_WORKLOAD
		unsigned int pointID = atomicAdd(&counterEstimator, int(1));
		// pointID = pointID * (*sampleOffset) * GPUNUMDIM;
		pointID = pointID * (*sampleOffset / 1.25) * GPUNUMDIM;
	#else
		unsigned int pointID = tid  * (*sampleOffset) * GPUNUMDIM;
	#endif

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; ++i){
		#if SORT_BY_WORKLOAD
			point[i] = sortedDatabase[pointID + i];
		#else
			point[i] = database[pointID + i];
		#endif
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];

	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; ++i)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)


		///////////////////////////
		//Take the intersection of the ranges for each dimension between
		//the point and the filtered set of cells in each dimension
		//Ranges in a given dimension that have points in them that are non-empty in a dimension will be tested
		///////////////////////////

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		//we go throgh each dimension and compare the range of the query points min/max cell ids to the filtered ones
		//find out which ones in the range exist based on the min/max
		//then determine the appropriate ranges

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin=1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) ],
				gridCellNDMask + gridCellNDMaskOffsets[ (i * 2) + 1 ] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax=1;
		}

		// cases:
		// found the min and max
		// found the min and not max
		//found the max and not the min
		//you don't find the min or max -- then only check the mid
		//you always find the mid because it's in the cell of the point you're looking for

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	///////////////////////////////////////
	//End taking intersection
	//////////////////////////////////////

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	uint64_t cellID = getLinearID_nDimensionsGPU(nDCellIDs, nCells, NUMINDEXEDDIM);

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; ++x)
			{
				indexes[x] = loopRng[x];
			}

			uint64_t calcLinearID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
			//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)

			// condition for the linear id unicomp pattern
			if(cellID <= calcLinearID)
			{
				struct gridCellLookup tmp;
				tmp.gridLinearID = calcLinearID;

				if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
				{
					//in the GPU implementation we go directly to computing neighbors so that we don't need to
					//store a buffer of the cells to check
					//cellsToCheck->push_back(calcLinearID);

					//HERE WE COMPUTE THE NEIGHBORS FOR THE CELL
					//XXXXXXXXXXXXXXXXXXXXXXXXX

					struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr+(*nNonEmptyCells), gridCellLookup(tmp));
					unsigned int GridIndex = resultBinSearch->idx;

					for (int k = index[GridIndex].indexmin; k <= index[GridIndex].indexmax; ++k)
					{
						DTYPE runningTotalDist = 0;
						unsigned int dataIdx = indexLookupArr[k];

						for (int l = 0; l < GPUNUMDIM; ++l)
						{
							runningTotalDist += (database[dataIdx * GPUNUMDIM + l]  - point[l])
									* (database[dataIdx * GPUNUMDIM + l] - point[l]);
						}

						if (sqrt(runningTotalDist) <= (*epsilon))
						{
							unsigned int idx = atomicAdd(cnt, int(2));
						}
					}
				}
			}
		} //end loop body

}



/******************************************************************************/



// __device__ int counter = 0;

// Global memory kernel - Initial version ("GPU")
__global__ void kernelNDGridIndexGlobal(
		unsigned int * batchBegin,
		unsigned int * N,
		unsigned int * offset,
		unsigned int * batchNum,
		DTYPE * database,
		DTYPE * sortedCells,
		unsigned int * originPointIndex,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		int * pointIDKey,
		int * pointInDistVal)
{

	unsigned int tid = (blockIdx.x * BLOCKSIZE + threadIdx.x);

	if (*N <= tid)
	{
		return;
	}

	unsigned int pointId = atomicAdd(batchBegin, int(1));

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; i++)
	{
		point[i] = database[ originPointIndex[pointId] * GPUNUMDIM + i ];
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; i++)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; x++)
			{
				indexes[x] = loopRng[x];
			}

			evaluateCell(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index,
					indexLookupArr, point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], false, nDCellIDs);

		} //end loop body

}





// Global memory kernel - Unicomp version ("Unicomp")
__global__ void kernelNDGridIndexGlobalUnicomp(
		unsigned int * batchBegin,
		unsigned int * N,
		unsigned int * offset,
		unsigned int * batchNum,
		DTYPE * database,
		DTYPE * sortedCells,
		unsigned int * originPointIndex,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		int * pointIDKey,
		int * pointInDistVal)
{

	unsigned int tid = (blockIdx.x * BLOCKSIZE + threadIdx.x);

	if (*N <= tid)
	{
		return;
	}

	unsigned int pointId = atomicAdd(batchBegin, int(1));

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; i++)
	{
		point[i] = sortedCells[pointId * GPUNUMDIM + i];
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; i++)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	///////////////////////////////////////
	//End taking intersection
	//////////////////////////////////////

    unsigned int indexes[NUMINDEXEDDIM];
    unsigned int loopRng[NUMINDEXEDDIM];

	for(int i = 0; i < NUMINDEXEDDIM; i++)
	{
		indexes[i] = nDCellIDs[i];
	}

	evaluateCell(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon,
			index, indexLookupArr, point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], false, nDCellIDs);
	#include "unicompWorkQueue.h"

}





// Global memory kernel - Linear ID comparison (Need to find a name : L-Unicomp ? Lin-Unicomp ? LId-Unicomp ?)
__global__ void kernelNDGridIndexGlobalLinearIDUnicomp(
		unsigned int * batchBegin,
		unsigned int * N,
		unsigned int * offset,
		unsigned int * batchNum,
		DTYPE * database,
		DTYPE * sortedCells,
		unsigned int * originPointIndex,
		DTYPE * epsilon,
		struct grid * index,
		unsigned int * indexLookupArr,
		struct gridCellLookup * gridCellLookupArr,
		DTYPE * minArr,
		unsigned int * nCells,
		unsigned int * cnt,
		unsigned int * nNonEmptyCells,
		unsigned int * gridCellNDMask,
		unsigned int * gridCellNDMaskOffsets,
		int * pointIDKey,
		int * pointInDistVal)
{

	unsigned int tid = (blockIdx.x * BLOCKSIZE + threadIdx.x);

	if (*N <= tid)
	{
		return;
	}

	unsigned int pointId = atomicAdd(batchBegin, int(1));

	//make a local copy of the point
	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; i++)
	{
		point[i] = sortedCells[pointId * GPUNUMDIM + i];
	}

	//calculate the coords of the Cell for the point
	//and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMin[NUMINDEXEDDIM];
	unsigned int rangeFilteredCellIdsMax[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; i++)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		unsigned int nDMinCellIDs = max(0, nDCellIDs[i] - 1); //boundary conditions (don't go beyond cell 0)
		unsigned int nDMaxCellIDs = min(nCells[i] - 1, nDCellIDs[i] + 1); //boundary conditions (don't go beyond the maximum number of cells)

		//compare the point's range of cell IDs in each dimension to the filter mask
		//only 2 possible values (you always find the middle point in the range), because that's the cell of the point itself
		bool foundMin = 0;
		bool foundMax = 0;

		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMinCellIDs)){ //extra +1 here is because we include the upper bound
			foundMin = 1;
		}
		if(thrust::binary_search(thrust::seq, gridCellNDMask + gridCellNDMaskOffsets[(i * 2)],
				gridCellNDMask + gridCellNDMaskOffsets[(i * 2) + 1] + 1, nDMaxCellIDs)){ //extra +1 here is because we include the upper bound
			foundMax = 1;
		}

		rangeFilteredCellIdsMin[i] = (1 == foundMin) ? nDMinCellIDs : (nDMinCellIDs + 1);
		rangeFilteredCellIdsMax[i] = (1 == foundMax) ? nDMaxCellIDs : (nDMinCellIDs + 1);
	}

	///////////////////////////////////////
	//End taking intersection
	//////////////////////////////////////

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	uint64_t cellID = getLinearID_nDimensionsGPU(nDCellIDs, nCells, NUMINDEXEDDIM);
	for(int i = 0; i < NUMINDEXEDDIM; i++) {
		indexes[i] = nDCellIDs[i];
	}

	evaluateCellUnicompOrigin(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index, indexLookupArr,
			point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], nDCellIDs, 1, 0);

	// cuts a third of the iterations, that are not necessary, in 2D
	// rangeFilteredCellIdsMin[NUMINDEXEDDIM - 1] = max(rangeFilteredCellIdsMin[NUMINDEXEDDIM - 1],
	// 													nDCellIDs[NUMINDEXEDDIM - 1]);

	for (loopRng[0] = rangeFilteredCellIdsMin[0]; loopRng[0] <= rangeFilteredCellIdsMax[0]; loopRng[0]++)
		for (loopRng[1] = rangeFilteredCellIdsMin[1]; loopRng[1] <= rangeFilteredCellIdsMax[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; x++)
			{
				indexes[x] = loopRng[x];
			}

			uint64_t neighborID = getLinearID_nDimensionsGPU(indexes, nCells, NUMINDEXEDDIM);
			if(cellID < neighborID)
			{
				evaluateCellUnicompAdjacent(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, index, indexLookupArr,
						point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], nDCellIDs, 1, 0);
			}

		} //end loop body

}
