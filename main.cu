#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <math.h>
#include <set>

#include "omp.h"

#include "main.h"
#include "params.h"
#include "structs.h"
#include "SortByWorkload.h"
#include "WorkQueue.h"
#include "GPU.h"

#include "Point.hpp"
#include "Util.hpp"

using std::cout;
using std::endl;

int main(int argc, char * argv[])
{

    printf("\n\n========== Hybrid-EGO ==========\n\n");

    if(NB_ARGS != argc)
    {
        printf("Expected %d args, found %d\n", NB_ARGS, argc);
        printf("Args: filename epsilon dim searchmode\n");
        return 1;
    }

    char filename[256];
    strcpy(filename, argv[FILENAME_ARG]);

    // int datasetSize = atoi(argv[DATASETSIZE_ARG]);
    DTYPE epsilon = atof(argv[EPSILON_ARG]);
    int dim = atoi(argv[DIM_ARG]);
    int searchMode = atoi(argv[SEARCHMODE_ARG]);

    if(GPUNUMDIM != dim)
    {
        printf("Error: dim should be equals to GPUNUMDIM (params.h)\n");
        return 1;
    }

    if(epsilon <= 0.0)
    {
        printf("Error: epsilon should be positive\n");
        return 1;
    }

    printf("Dataset: %s\n", filename);
    printf("Epsilon: %f\n", epsilon);
    printf("Dimensionality: %d\n", dim);
    printf("Search mode: %d\n", searchMode);

    std::vector< std::vector<DTYPE> > NDdataPoints;
    double tBeginReadDataset = omp_get_wtime();
    importNDDataset(&NDdataPoints, filename);
    double tEndReadDataset = omp_get_wtime();
    printf("Time to read the dataset: %f\n", tEndReadDataset - tBeginReadDataset);

    unsigned int DBSIZE = NDdataPoints.size();
    setQueueIndexCPU(DBSIZE);

    sortInNDBins(&NDdataPoints);

    Point * A = new Point[DBSIZE + 1];
    for(int i = 0; i < DBSIZE; ++i)
    {
        pPoint p = & A[i];
        p->id = i;
        for(int j = 0; j < NUMINDEXEDDIM; ++j)
        {
            p->x[j] = NDdataPoints[i][j];
        }
    }
    Point * B = A;

    DTYPE * minArr = new DTYPE [NUMINDEXEDDIM];
    DTYPE * maxArr = new DTYPE [NUMINDEXEDDIM];
    unsigned int * nCells = new unsigned int [NUMINDEXEDDIM];
    uint64_t totalCells = 0;
    unsigned int nNonEmptyCells = 0;

    generateNDGridDimensions(&NDdataPoints, epsilon, minArr, maxArr, nCells, &totalCells);
    printf("[GPU] ~ Total cells (including empty): %lu\n", totalCells);

    struct grid * index;
    struct gridCellLookup * gridCellLookupArr;
    unsigned int * gridCellNDMask;
    unsigned int * nNDMaskElems = new unsigned int;
    unsigned int * gridCellNDMaskOffsets = new unsigned int [NUMINDEXEDDIM * 2];
    unsigned int * indexLookupArr = new unsigned int[NDdataPoints.size()];

    populateNDGridIndexAndLookupArray(&NDdataPoints, epsilon, &gridCellLookupArr, &index,
            indexLookupArr, minArr,  nCells, totalCells, &nNonEmptyCells, &gridCellNDMask,
            gridCellNDMaskOffsets, nNDMaskElems);

    //Neighbortable storage -- the result
    neighborTableLookup * neighborTable = new neighborTableLookup [NDdataPoints.size()];
    // neighborTableLookup * neighborTable = new neighborTableLookup[DBSIZE * fraction];
    std::vector<struct neighborDataPtrs> pointersToNeighbors(DBSIZE);

    DTYPE * database = new DTYPE [DBSIZE * GPUNUMDIM];
    for(unsigned int i = 0; i < DBSIZE; ++i)
    {
        for(unsigned int j = 0; j < GPUNUMDIM; ++j)
        {
            database[i * GPUNUMDIM + j] = NDdataPoints[i][j];
        }
        // std::copy(NDdataPoints[i].begin(), NDdataPoints[i].end(), database + i * GPUNUMDIM);
    }

    DTYPE * dev_epsilon;
    DTYPE * dev_database;
    struct grid * dev_index;
    unsigned int * dev_indexLookupArr;
    struct gridCellLookup * dev_gridCellLookupArr;
    DTYPE * dev_minArr;
    unsigned int * dev_nCells;
    unsigned int * dev_nNonEmptyCells;
    unsigned int * dev_gridCellNDMask;
    unsigned int * dev_gridCellNDMaskOffsets;

    unsigned int * originPointIndex;
    unsigned int * dev_originPointIndex;

    double tStartSort = omp_get_wtime();
    sortByWorkLoad(searchMode, &DBSIZE, &epsilon, &dev_epsilon, database, &dev_database, index, &dev_index, indexLookupArr, &dev_indexLookupArr,
            gridCellLookupArr, &dev_gridCellLookupArr, minArr, &dev_minArr, nCells, &dev_nCells, &nNonEmptyCells, &dev_nNonEmptyCells,
            gridCellNDMask, &dev_gridCellNDMask, gridCellNDMaskOffsets, &dev_gridCellNDMaskOffsets, nNDMaskElems, &originPointIndex, &dev_originPointIndex,
            nullptr);
    double tEndSort = omp_get_wtime();
    double sortTime = tEndSort - tStartSort;

    uint64_t totalNeighbors = 0;
    uint64_t totalNeighborsCPU = 0;

    omp_set_nested(1);
	omp_set_dynamic(0);

    double gpuTime = 0.0;
    double egoTime = 0.0;
    double egoReorder = 0.0;
    double egoSort = 0.0;

    double tStart = omp_get_wtime();
    #pragma omp parallel num_threads(2)
    {
        int tid = omp_get_thread_num();

        if(0 == tid) // GPU part
        {
            double tBeginGPU = omp_get_wtime();
            distanceTableNDGridBatches(searchMode, &DBSIZE, &epsilon, dev_epsilon, database, dev_database, index, dev_index,
                    indexLookupArr, dev_indexLookupArr, gridCellLookupArr, dev_gridCellLookupArr, minArr, dev_minArr, nCells, dev_nCells,
                    &nNonEmptyCells, dev_nNonEmptyCells, gridCellNDMask, dev_gridCellNDMask, gridCellNDMaskOffsets, dev_gridCellNDMaskOffsets,
                    nNDMaskElems, originPointIndex, dev_originPointIndex, neighborTable, &pointersToNeighbors, &totalNeighbors);
            double tEndGPU = omp_get_wtime();
            gpuTime = tEndGPU - tBeginGPU;
        }
        else // Super-EGO part
        {
            if(searchMode != SM_GPU)
            {
                unsigned int A_sz = DBSIZE;
                unsigned int B_sz = DBSIZE;

                double tBeginEgo = omp_get_wtime();

                printf("[EGO] ~ Reordering the dimensions\n");
                double tStartReorder = omp_get_wtime();
                Util::reorderDim(A, A_sz, B, B_sz);
                double tEndReorder = omp_get_wtime();
                egoReorder = tEndReorder - tStartReorder;

                printf("[EGO] ~ EGO sorting of A\n");
                double tStartEGOSort = omp_get_wtime();
                // std::sort(sortedDatabaseTmp, sortedDatabaseTmp + (*nNonEmptyCells),
                //         [](const schedulingCell& a, const schedulingCell& b){ return a.nbPoints > b.nbPoints; });
                // auto egoSortLamba = [](const void * v1, const void * v2) -> int
                // {
                //     pPoint p1 = (pPoint)v1;
                // 	pPoint p2 = (pPoint)v2;
                //
                // 	for (int i = 0; i < GPUNUMDIM; i++)
                // 	{
                // 		int d = ((int) (p1->x[i]/Util::eps)) - ((int) (p2->x[i]/Util::eps));
                //
                // 		if (d != 0)
                // 			return d;
                // 	}
                //
                // 	return 0;
                // }
                // std::sort(A, A + A_sz, egoSortLamba);
                qsort(A, A_sz, sizeof(Point), pcmp);
                double tEndEGOSort = omp_get_wtime();
                egoSort = tEndEGOSort - tStartEGOSort;

                unsigned int * egoMapping = new unsigned int[DBSIZE];
                for(int i = 0; i < DBSIZE; ++i)
                {
                    pPoint p = &A[i];
                    egoMapping[p->id] = i;
                }

                totalNeighborsCPU = Util::multiThreadJoinWorkQueue(A, A_sz, B, B_sz, CPU_THREADS, egoMapping);

                double tEndEgo = omp_get_wtime();
                egoTime = tEndEgo - tBeginEgo;

                delete[] A;

            } // searchMode
        } // Super-EGO
    } // parallel section
    double tEnd = omp_get_wtime();

    printf("[RESULT] ~ Total result set size: %lu\n", totalNeighbors + totalNeighborsCPU);
    printf("   [RESULT] ~ Total result set size on the GPU: %lu\n", totalNeighbors);
    printf("   [RESULT] ~ Total result set size on the CPU: %lu\n", totalNeighborsCPU);

    printf("[RESULT] ~ Total execution time: %f\n", (tEnd - tStart) + sortTime);
    printf("   [RESULT] ~ Total execution time for the GPU: %f\n", gpuTime);
    printf("   [RESULT] ~ Total execution time for the CPU: %f (Reorder: %f, sort: %f)\n", egoTime, egoReorder, egoSort);

    if(egoTime < gpuTime)
    {
        printf("[RESULT] ~ The CPU ended before the GPU, with a difference of: %f\n", gpuTime - egoTime);
    }else{
        printf("[RESULT] ~ The GPU ended before the CPU, with a difference of: %f\n", egoTime - gpuTime);
    }

    NDdataPoints.clear();
    NDdataPoints.shrink_to_fit();
    pointersToNeighbors.clear();
    pointersToNeighbors.shrink_to_fit();

    delete[] minArr;
    delete[] maxArr;
    delete[] nCells;
    delete nNDMaskElems;
    delete[] gridCellNDMaskOffsets;
    delete[] indexLookupArr;
    delete[] neighborTable;
    delete[] database;

    hipFree(dev_epsilon);
    hipFree(dev_database);
    hipFree(dev_index);
    hipFree(dev_indexLookupArr);
    hipFree(dev_gridCellLookupArr);
    hipFree(dev_minArr);
    hipFree(dev_nCells);
    hipFree(dev_nNonEmptyCells);
    hipFree(dev_gridCellNDMask);
    hipFree(dev_gridCellNDMaskOffsets);

    delete[] originPointIndex;
    hipFree(dev_originPointIndex);

    return 0;
}


void generateNDGridDimensions(
        std::vector< std::vector <DTYPE> > * NDdataPoints,
        DTYPE epsilon,
        DTYPE* minArr,
        DTYPE* maxArr,
        unsigned int * nCells,
        uint64_t * totalCells)
{

    printf("\n\n*****************************  Generating grid dimensions  *****************************\n");

    printf("\nNumber of dimensions data: %d, Number of dimensions indexed: %d", GPUNUMDIM, NUMINDEXEDDIM);

    //make the min/max values for each grid dimension the first data element
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        minArr[j] = (*NDdataPoints)[0][j];
        maxArr[j] = (*NDdataPoints)[0][j];
    }



    for(int i = 1; i < NDdataPoints->size(); ++i)
    {
        for(int j = 0; j < NUMINDEXEDDIM; j++)
        {
            if ((*NDdataPoints)[i][j] < minArr[j])
            {
                minArr[j] = (*NDdataPoints)[i][j];
            }
            if((*NDdataPoints)[i][j] > maxArr[j])
            {
                maxArr[j] = (*NDdataPoints)[i][j];
            }
        }
    }

    printf("\n");
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        printf("Data Dim: %d, min/max: %f, %f\n", j, minArr[j], maxArr[j]);
    }

    //add buffer around each dim so no weirdness later with putting data into cells
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        minArr[j] -= epsilon;
        maxArr[j] += epsilon;
    }

    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        printf("Appended by epsilon Dim: %d, min/max: %f, %f\n", j, minArr[j], maxArr[j]);
    }

    //calculate the number of cells:
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        nCells[j] = ceil((maxArr[j] - minArr[j]) / epsilon);
        printf("Number of cells dim: %d: %d\n", j, nCells[j]);
    }

    //calc total cells: num cells in each dim multiplied
    uint64_t tmpTotalCells = nCells[0];
    for(int j = 1; j < NUMINDEXEDDIM; j++)
    {
        tmpTotalCells *= nCells[j];
    }

    *totalCells = tmpTotalCells;

}


struct cmpStruct
{
    cmpStruct(std::vector< std::vector<DTYPE> > points) {this->points = points;}
    bool operator() (int a, int b)
    {
        return points[a][0] < points[b][0];
    }

    std::vector< std::vector<DTYPE> > points;
};


void populateNDGridIndexAndLookupArray(
        std::vector<std::vector <DTYPE> > * NDdataPoints,
        DTYPE epsilon,
        struct gridCellLookup ** gridCellLookupArr,
        struct grid ** index,
        unsigned int * indexLookupArr,
        DTYPE* minArr,
        unsigned int * nCells,
        uint64_t totalCells,
        unsigned int * nNonEmptyCells,
        unsigned int ** gridCellNDMask,
        unsigned int * gridCellNDMaskOffsets,
        unsigned int * nNDMaskElems)
{

    /////////////////////////////////
    // Populate grid lookup array
    // and corresponding indicies in the lookup array
    /////////////////////////////////
    printf("\n\n*****************************  Populating Grid Index and lookup array  *****************************\n");
    // printf("\nSize of dataset: %lu", NDdataPoints->size());


    ///////////////////////////////
    // First, we need to figure out how many non-empty cells there will be
    // For memory allocation
    // Need to do a scan of the dataset and calculate this
    // Also need to keep track of the list of uniquie linear grid cell IDs for inserting into the grid
    ///////////////////////////////
    std::set<uint64_t> uniqueGridCellLinearIds;
    std::vector<uint64_t>uniqueGridCellLinearIdsVect; //for random access

    for (int i = 0; i < NDdataPoints->size(); ++i)
    {
        unsigned int tmpNDCellIdx[NUMINDEXEDDIM];
        for (int j = 0; j < NUMINDEXEDDIM; j++)
        {
            tmpNDCellIdx[j] = (((*NDdataPoints)[i][j] - minArr[j]) / epsilon);
        }
        uint64_t linearID = getLinearID_nDimensions(tmpNDCellIdx, nCells, NUMINDEXEDDIM);
        uniqueGridCellLinearIds.insert(linearID);

    }

    // printf("uniqueGridCellLinearIds: %d",uniqueGridCellLinearIds.size());

    //copy the set to the vector (sets can't do binary searches -- no random access)
    std::copy(uniqueGridCellLinearIds.begin(), uniqueGridCellLinearIds.end(), std::back_inserter(uniqueGridCellLinearIdsVect));


    ///////////////////////////////////////////////


    std::vector<uint64_t> * gridElemIDs;
    gridElemIDs = new std::vector<uint64_t>[uniqueGridCellLinearIds.size()];

    //Create ND array mask:
    //This mask determines which cells in each dimension has points in them.
    std::set<unsigned int> NDArrMask[NUMINDEXEDDIM];

    std::vector<uint64_t>::iterator lower;


    for (int i = 0; i < NDdataPoints->size(); i++)
    {
        unsigned int tmpNDCellID[NUMINDEXEDDIM];
        for (int j = 0; j < NUMINDEXEDDIM; j++)
        {
            tmpNDCellID[j] = (((*NDdataPoints)[i][j] - minArr[j]) / epsilon);

            //add value to the ND array mask
            NDArrMask[j].insert(tmpNDCellID[j]);
        }

        //get the linear id of the cell
        uint64_t linearID = getLinearID_nDimensions(tmpNDCellID, nCells, NUMINDEXEDDIM);
        //printf("\nlinear id: %d",linearID);
        //if (linearID > totalCells){

        //	printf("\n\nERROR Linear ID is: %lu, total cells is only: %lu\n\n", linearID, totalCells);
        //}

        //find the index in gridElemIds that corresponds to this grid cell linear id

        lower = std::lower_bound(uniqueGridCellLinearIdsVect.begin(), uniqueGridCellLinearIdsVect.end(), linearID);
        uint64_t gridIdx = lower - uniqueGridCellLinearIdsVect.begin();
        gridElemIDs[gridIdx].push_back(i);
    }




    ///////////////////////////////
    //Here we fill a temporary index with points, and then copy the non-empty cells to the actual index
    ///////////////////////////////

    struct grid * tmpIndex = new grid[uniqueGridCellLinearIdsVect.size()];

    int cnt = 0;



    //populate temp index and lookup array

    for (int i = 0; i < uniqueGridCellLinearIdsVect.size(); i++)
    {
        tmpIndex[i].indexmin = cnt;
        for (int j = 0; j < gridElemIDs[i].size(); j++)
        {
            if ((NDdataPoints->size() - 1) < j)
            {
                printf("\n\n***ERROR Value of a data point is larger than the dataset! %d\n\n", j);
                return;
            }
            indexLookupArr[cnt] = gridElemIDs[i][j];
            cnt++;
        }
        tmpIndex[i].indexmax = cnt - 1;
    }

    // printf("\nExiting grid populate method early!");
    // return;

    printf("\nFull cells: %d (%f, fraction full)", (unsigned int)uniqueGridCellLinearIdsVect.size(), uniqueGridCellLinearIdsVect.size() * 1.0 / double(totalCells));
    printf("\nEmpty cells: %ld (%f, fraction empty)", totalCells - (unsigned int)uniqueGridCellLinearIdsVect.size(), (totalCells - uniqueGridCellLinearIdsVect.size() * 1.0) / double(totalCells));

    *nNonEmptyCells = uniqueGridCellLinearIdsVect.size();


    printf("\nSize of index that would be sent to GPU (GiB) -- (if full index sent), excluding the data lookup arr: %f", (double)sizeof(struct grid) * (totalCells) / (1024.0 * 1024.0 * 1024.0));
    printf("\nSize of compressed index to be sent to GPU (GiB) , excluding the data and grid lookup arr: %f", (double)sizeof(struct grid) * (uniqueGridCellLinearIdsVect.size() * 1.0) / (1024.0 * 1024.0 * 1024.0));


    /////////////////////////////////////////
    //copy the tmp index into the actual index that only has the non-empty cells

    //allocate memory for the index that will be sent to the GPU
    *index = new grid[uniqueGridCellLinearIdsVect.size()];
    *gridCellLookupArr = new struct gridCellLookup[uniqueGridCellLinearIdsVect.size()];

    cmpStruct theStruct(*NDdataPoints);

    for (int i = 0; i < uniqueGridCellLinearIdsVect.size(); i++)
    {
        (*index)[i].indexmin = tmpIndex[i].indexmin;
        (*index)[i].indexmax = tmpIndex[i].indexmax;
        (*gridCellLookupArr)[i].idx = i;
        (*gridCellLookupArr)[i].gridLinearID = uniqueGridCellLinearIdsVect[i];
    }

    printf("\nWhen copying from entire index to compressed index: number of non-empty cells: %lu", uniqueGridCellLinearIdsVect.size());

    //copy NDArrMask from set to an array

    //find the total size and allocate the array

    unsigned int cntNDOffsets = 0;
    unsigned int cntNonEmptyNDMask = 0;
    for (int i = 0; i < NUMINDEXEDDIM; i++)
    {
        cntNonEmptyNDMask += NDArrMask[i].size();
    }
    *gridCellNDMask = new unsigned int[cntNonEmptyNDMask];

    *nNDMaskElems = cntNonEmptyNDMask;


    //copy the offsets to the array
    for (int i = 0; i < NUMINDEXEDDIM; i++)
    {
        //Min
        gridCellNDMaskOffsets[(i * 2)] = cntNDOffsets;
        for (std::set<unsigned int>::iterator it = NDArrMask[i].begin(); it != NDArrMask[i].end(); ++it)
        {
            (*gridCellNDMask)[cntNDOffsets] = *it;
            cntNDOffsets++;
        }
        //max
        gridCellNDMaskOffsets[(i * 2) + 1]  =cntNDOffsets - 1;
    }

    delete [] tmpIndex;
}


uint64_t getLinearID_nDimensions(
        unsigned int * indexes,
        unsigned int * dimLen,
        unsigned int nDimensions)
{
    uint64_t index = 0;
    uint64_t multiplier = 1;
    for (int i = 0; i < nDimensions; i++)
    {
        index += (uint64_t)indexes[i] * multiplier;
        multiplier *= dimLen[i];
    }

    return index;
}
