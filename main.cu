#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <vector>
#include <math.h>
#include <set>
#include <algorithm>

#include "omp.h"

#include "main.h"
#include "params.h"
#include "structs.h"
#include "SortByWorkload.h"
#include "WorkQueue.h"
#include "GPU.h"

#include "Point.hpp"
#include "Util.hpp"

// #include <boost/sort/sort.hpp>
#include <boost_1_72_0/sort/sort.hpp>
// #include <parallel/algorithm>

using std::cout;
using std::endl;


bool egoSortFunction(Point const& p1, Point const& p2)
{
    for (int i = 0; i < GPUNUMDIM; i++)
	{
		if ( (int) (p1.x[i] / Util::eps) < (int)(p2.x[i] / Util::eps) ) return true;
		if ( (int) (p1.x[i] / Util::eps) > (int)(p2.x[i] / Util::eps) ) return false;
	}

	return false;
}


int main(int argc, char * argv[])
{
    double tStartStart = omp_get_wtime();
    printf("\n\n========== Hybrid-EGO ==========\n\n\n");

    if(NB_ARGS_MAX != argc && (NB_ARGS_MAX - 1) != argc)
    {
        fprintf(stderr, "[MAIN] ~ Expected %d or %d args, found %d\n", NB_ARGS_MAX, NB_ARGS_MAX - 1, argc);
        fprintf(stderr, "[MAIN] ~ Args: filename epsilon dim searchmode [CPU-GPU partitioning]\n");
        return 1;
    }

    char filename[256];
    strcpy(filename, argv[FILENAME_ARG]);
    DTYPE epsilon = atof(argv[EPSILON_ARG]);
    int dim = atoi(argv[DIM_ARG]);
    int searchMode = atoi(argv[SEARCHMODE_ARG]);
    float staticPartition;

    // Static partitioning between CPU and GPU components
    if (SM_HYBRID_STATIC == searchMode)
    {
        if (NB_ARGS_MAX == argc)
        {
            staticPartition = atof(argv[STATIC_PART_ARG]);
            if (staticPartition <= 0.0 || 1.0 <= staticPartition)
            {
                fprintf(stderr, "[MAIN] ~ Error: the partitioning should be between ]0.0, 1.0[");
                return 1;
            }
        } else {
            fprintf(stderr, "[MAIN] ~ Error: you need to indicate the partitioning as the last parameter");
            return 1;
        }
    } else { // Dynamic partitioning or CPU/GPU alone
        if (SM_GPU == searchMode)
        {
            // The GPU is alone so it takes all the work
            staticPartition = 1.0;
        } else {
            // The CPU is alone so it takes all the work, or it's the regular (dynamic) hybrid
            // so the staticPartition value does not matter
            staticPartition = 0.0;
        }
    }

    if(GPUNUMDIM != dim)
    {
        fprintf(stderr, "[MAIN] ~ Error: dim should be equals to GPUNUMDIM (see params.h)\n");
        return 1;
    }

    if(epsilon <= 0.0 || 1.0 < epsilon)
    {
        fprintf(stderr, "[MAIN] ~ Error: epsilon should be between ]0.0, 1.0]\n");
        return 1;
    }

    fprintf(stdout, "[MAIN] ~ Dataset: %s\n", filename);
    fprintf(stdout, "[MAIN] ~ Epsilon: %f\n", epsilon);
    fprintf(stdout, "[MAIN] ~ Dimensionality: %d\n", dim);
    fprintf(stdout, "[MAIN] ~ Search mode: %d\n", searchMode);
    if (SM_HYBRID_STATIC == searchMode)
    {
        fprintf(stdout, "[MAIN] ~ GPU part: %f, CPU part: %f\n", staticPartition, 1 - staticPartition);
    }

    Util::eps = epsilon;
    Util::eps2 = epsilon * epsilon;

    std::vector< std::vector<DTYPE> > NDdataPoints;
    double tBeginReadDataset = omp_get_wtime();
    // DTYPE * database;
    // unsigned int nbPoints = 0;
    // importNDDataset(&NDdataPoints, filename);
    importNDDatasetBinary(&NDdataPoints, filename);
    // importNDDatasetBinary(&database, filename, &nbPoints);
    double tEndReadDataset = omp_get_wtime();
    fprintf(stdout, "[MAIN] ~ Time to read the dataset: %f\n", tEndReadDataset - tBeginReadDataset);

    unsigned int DBSIZE = NDdataPoints.size();
    // unsigned int DBSIZE = nbPoints;
    setQueueIndexCPU(DBSIZE);

    // sortInNDBins(&NDdataPoints);

    Point * A;
    Point * B;
    if(SM_GPU != searchMode)
    {
        fprintf(stdout, "[MAIN] ~ Converting the dataset for Super-EGO\n");
        A = new Point[DBSIZE + 1];
        for(int i = 0; i < DBSIZE; ++i)
        {
            pPoint p = & A[i];
            p->id = i;
            for(int j = 0; j < GPUNUMDIM; ++j)
            {
                p->x[j] = NDdataPoints[i][j];
                // p->x[j] = database[i * GPUNUMDIM + j];
            }
        }
        B = A;
    }

    DTYPE * database = new DTYPE [DBSIZE * GPUNUMDIM];
    for(int i = 0; i < DBSIZE; ++i)
    {
        for(int j = 0; j < GPUNUMDIM; ++j)
        {
            database[i * GPUNUMDIM + j] = NDdataPoints[i][j];
        }
        // std::copy(NDdataPoints[i].begin(), NDdataPoints[i].end(), database + i * GPUNUMDIM);
    }

    DTYPE * minArr = new DTYPE [NUMINDEXEDDIM];
    DTYPE * maxArr = new DTYPE [NUMINDEXEDDIM];
    unsigned int * nCells = new unsigned int [NUMINDEXEDDIM];
    uint64_t totalCells = 0;
    unsigned int nNonEmptyCells = 0;

    generateNDGridDimensions(&NDdataPoints, epsilon, minArr, maxArr, nCells, &totalCells);
    fprintf(stdout, "[MAIN] ~ Total cells (including empty): %lu\n", totalCells);

    struct grid * index;
    struct gridCellLookup * gridCellLookupArr;
    unsigned int * indexLookupArr = new unsigned int[NDdataPoints.size()];

    DTYPE * dev_epsilon;
    DTYPE * dev_database;
    struct grid * dev_index;
    unsigned int * dev_indexLookupArr;
    struct gridCellLookup * dev_gridCellLookupArr;
    DTYPE * dev_minArr;
    unsigned int * dev_nCells;
    unsigned int * dev_nNonEmptyCells;

    gridIndexingGPU(&DBSIZE, totalCells, database, &dev_database, &epsilon, &dev_epsilon, minArr, &dev_minArr, &index, &dev_index,
            indexLookupArr, &dev_indexLookupArr, &gridCellLookupArr, &dev_gridCellLookupArr, &nNonEmptyCells, &dev_nNonEmptyCells,
            nCells, &dev_nCells);

    //Neighbortable storage -- the result
    neighborTableLookup * neighborTable = new neighborTableLookup[NDdataPoints.size()];
    std::vector<struct neighborDataPtrs> pointersToNeighbors;

    unsigned int * originPointIndex;
    unsigned int * dev_originPointIndex;

    uint64_t totalNeighbors = 0;
    uint64_t totalNeighborsCPU = 0;

    struct schedulingCell * sortedDatabaseTmp;

    double sortTime, gpuTime, egoTime, egoReorder, egoSort;

    double tStartSort = omp_get_wtime();
    #if SORT_BY_WORKLOAD
        sortByWorkLoad(searchMode, &DBSIZE, staticPartition, &sortedDatabaseTmp, &epsilon, &dev_epsilon,
                database, &dev_database, index, &dev_index, indexLookupArr, &dev_indexLookupArr,
                gridCellLookupArr, &dev_gridCellLookupArr, minArr, &dev_minArr, nCells, &dev_nCells,
                &nNonEmptyCells, &dev_nNonEmptyCells, &originPointIndex, &dev_originPointIndex);
    #endif
    double tEndSort = omp_get_wtime();
    sortTime = tEndSort - tStartSort;

    fprintf(stdout, "\n\n[MAIN] ~ Time to do everything before computing: %f\n\n\n", tEndSort - tStartStart);

    unsigned int nbCandidatesGPU = 0;

    omp_set_nested(1);
	omp_set_dynamic(0);

    double tStart = omp_get_wtime();
    double tEndGPU, tEndEgo;
    #pragma omp parallel num_threads(2)
    {
        int tid = omp_get_thread_num();

        if(0 == tid) // GPU part
        {
            if(searchMode != SM_CPU)
            {
                double tBeginGPU = omp_get_wtime();
                #if SORT_BY_WORKLOAD
                    distanceTableNDGridBatches(searchMode, staticPartition, &DBSIZE, &epsilon, dev_epsilon, database, dev_database,
                            index, dev_index, indexLookupArr, dev_indexLookupArr, gridCellLookupArr, dev_gridCellLookupArr,
                            minArr, dev_minArr, nCells, dev_nCells, &nNonEmptyCells, dev_nNonEmptyCells,
                            originPointIndex, dev_originPointIndex, neighborTable, &pointersToNeighbors, &totalNeighbors, &nbCandidatesGPU);
                #else
                    distanceTableNDGridBatches(searchMode, staticPartition, &DBSIZE, &epsilon, dev_epsilon, database, dev_database,
                            index, dev_index, indexLookupArr, dev_indexLookupArr, gridCellLookupArr, dev_gridCellLookupArr,
                            minArr, dev_minArr, nCells, dev_nCells, &nNonEmptyCells, dev_nNonEmptyCells,
                            nullptr, nullptr, neighborTable, &pointersToNeighbors, &totalNeighbors, &nbCandidatesGPU);
                #endif
                tEndGPU = omp_get_wtime();
                gpuTime = tEndGPU - tBeginGPU;
            }
        }
        else // Super-EGO part
        {
            if(searchMode != SM_GPU)
            {
                if(searchMode == SM_CPU)
                {
                    setQueueIndex(0);
                }

                unsigned int A_sz = DBSIZE;
                unsigned int B_sz = DBSIZE;

                fprintf(stdout, "[EGO] ~ Reordering the dimensions\n");
                double tStartReorder = omp_get_wtime();
                Util::reorderDim(A, A_sz, B, B_sz);
                double tEndReorder = omp_get_wtime();
                egoReorder = tEndReorder - tStartReorder;
                fprintf(stdout, "[EGO] ~ Done reordering in %f\n", egoReorder);

                fprintf(stdout, "[EGO] ~ EGO-sorting of A\n");
                double tStartEGOSort = omp_get_wtime();
                // std::stable_sort(A, A + A_sz, egoSortFunction);
                boost::sort::sample_sort(A, A + A_sz, egoSortFunction, CPU_THREADS);
                double tEndEGOSort = omp_get_wtime();
                egoSort = tEndEGOSort - tStartEGOSort;
                fprintf(stdout, "[EGO] ~ Done EGO-sorting in %f\n", egoSort);

                fprintf(stdout, "[EGO] ~ Creating the mapping\n");
                double tStartMapping = omp_get_wtime();
                unsigned int * egoMapping = new unsigned int[DBSIZE];
                for(int i = 0; i < DBSIZE; ++i)
                {
                    pPoint p = &A[i];
                    egoMapping[p->id] = i;
                }
                double tEndMapping = omp_get_wtime();
                fprintf(stdout, "[EGO] ~ Done creating the mapping in %f\n", tEndMapping - tStartMapping);

                double tBeginEgo = omp_get_wtime();

                fprintf(stdout, "[EGO] ~ Beginning the computation\n");
                #if SORT_BY_WORKLOAD
                    totalNeighborsCPU = Util::multiThreadJoinWorkQueue(searchMode, A, A_sz, B, B_sz, egoMapping, originPointIndex, neighborTable);
                #else
                    totalNeighborsCPU = Util::multiThreadJoinWorkQueue(searchMode, A, A_sz, B, B_sz, egoMapping, nullptr, neighborTable);
                #endif
                fprintf(stdout, "[EGO] ~ Done with the computation\n");

                tEndEgo = omp_get_wtime();
                egoTime = tEndEgo - tBeginEgo;
            } // searchMode
        } // Super-EGO
        #pragma omp barrier
    } // parallel section
    double tEnd = omp_get_wtime();
    double computeTime = tEnd - tStart;

    displayIndexes();

    fprintf(stdout, "[RESULT] ~ Total result set size: %lu\n", totalNeighbors + totalNeighborsCPU);
    fprintf(stdout, "   [RESULT] ~ Total result set size on the GPU: %lu\n", totalNeighbors);
    fprintf(stdout, "   [RESULT] ~ Total result set size on the CPU: %lu\n", totalNeighborsCPU);

    #if COUNT_CANDIDATES_GPU
        if (searchMode == SM_HYBRID || searchMode == SM_HYBRID_STATIC)
        {
            uint64_t nbCandidatesGPU = 0;
            for (int i = 0; i < nNonEmptyCells; ++i)
            {
                int cellId = sortedDatabaseTmp[i].cellId;
                int nbNeighbor = index[cellId].indexmax - index[cellId].indexmin + 1;
                nbCandidatesGPU += (nbNeighbor * sortedDatabaseTmp[i].nbPoints);
            }
            fprintf(stdout, "   [RESULT] ~ Total number of candidate points refined by the GPU: %lu\n", nbCandidatesGPU);
        }
    #endif

    fprintf(stdout, "[RESULT] ~ Total execution time: %f\n", computeTime + sortTime);
    fprintf(stdout, "   [RESULT] ~ Total execution time to SortByWL: %f\n", sortTime);
    fprintf(stdout, "   [RESULT] ~ Total execution time for the GPU: %f\n", gpuTime);
    fprintf(stdout, "   [RESULT] ~ Total execution time for the CPU: %f (reorder: %f, sort: %f, total = %f)\n", egoTime, egoReorder, egoSort, egoTime + egoReorder + egoSort);

    if(tEndGPU < tEndEgo)
    {
        fprintf(stdout, "[RESULT] ~ The GPU ended before the CPU, with a difference of: %f\n", tEndEgo - tEndGPU);
    }else{
        fprintf(stdout, "[RESULT] ~ The CPU ended before the GPU, with a difference of: %f\n", tEndGPU - tEndEgo);
    }

    // printNeighborTable(neighborTable, 0, 20);

    NDdataPoints.clear();
    NDdataPoints.shrink_to_fit();
    pointersToNeighbors.clear();
    pointersToNeighbors.shrink_to_fit();

    delete[] minArr;
    delete[] maxArr;
    delete[] nCells;
    delete[] indexLookupArr;
    delete[] neighborTable;
    delete[] database;

    if(SM_GPU != searchMode)
    {
        delete[] A;
    }

    hipFree(dev_epsilon);
    hipFree(dev_database);
    hipFree(dev_index);
    hipFree(dev_indexLookupArr);
    hipFree(dev_gridCellLookupArr);
    hipFree(dev_minArr);
    hipFree(dev_nCells);
    hipFree(dev_nNonEmptyCells);

    #if SORT_BY_WORKLOAD
        delete[] originPointIndex;
        hipFree(dev_originPointIndex);
    #endif

    return 0;
}


void generateNDGridDimensions(
        std::vector< std::vector <DTYPE> > * NDdataPoints,
        DTYPE epsilon,
        DTYPE* minArr,
        DTYPE* maxArr,
        unsigned int * nCells,
        uint64_t * totalCells)
{

    printf("\n\n*****************************  Generating grid dimensions  *****************************\n");

    printf("\nNumber of dimensions data: %d, Number of dimensions indexed: %d", GPUNUMDIM, NUMINDEXEDDIM);

    //make the min/max values for each grid dimension the first data element
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        minArr[j] = (*NDdataPoints)[0][j];
        maxArr[j] = (*NDdataPoints)[0][j];
    }



    for(int i = 1; i < NDdataPoints->size(); ++i)
    {
        for(int j = 0; j < NUMINDEXEDDIM; j++)
        {
            if ((*NDdataPoints)[i][j] < minArr[j])
            {
                minArr[j] = (*NDdataPoints)[i][j];
            }
            if((*NDdataPoints)[i][j] > maxArr[j])
            {
                maxArr[j] = (*NDdataPoints)[i][j];
            }
        }
    }

    printf("\n");
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        printf("Data Dim: %d, min/max: %f, %f\n", j, minArr[j], maxArr[j]);
    }

    //add buffer around each dim so no weirdness later with putting data into cells
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        minArr[j] -= epsilon;
        maxArr[j] += epsilon;
    }

    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        printf("Appended by epsilon Dim: %d, min/max: %f, %f\n", j, minArr[j], maxArr[j]);
    }

    //calculate the number of cells:
    for(int j = 0; j < NUMINDEXEDDIM; j++)
    {
        nCells[j] = ceil((maxArr[j] - minArr[j]) / epsilon);
        printf("Number of cells dim: %d: %d\n", j, nCells[j]);
    }

    //calc total cells: num cells in each dim multiplied
    uint64_t tmpTotalCells = nCells[0];
    for(int j = 1; j < NUMINDEXEDDIM; j++)
    {
        tmpTotalCells *= nCells[j];
    }

    *totalCells = tmpTotalCells;

}


void printNeighborTable(
    struct neighborTableLookup * neighborTable,
    unsigned int begin,
    unsigned int end)
{
    printf("\n");
    for (int i = begin; i < end; ++i)
    {
	 	// sort to compare against CPU implementation
	 	std::sort(neighborTable[i].dataPtr + neighborTable[i].indexmin, neighborTable[i].dataPtr + neighborTable[i].indexmax + 1);
	 	printf("point id: %d, neighbors: %d\n", i, neighborTable[i].indexmax - neighborTable[i].indexmin);
	 	for (int j = neighborTable[i].indexmin; j < neighborTable[i].indexmax - 1; j++)
        {
	 		printf("%d, ", neighborTable[i].dataPtr[j]);
	 	}
        printf("%d\n", neighborTable[i].dataPtr[ neighborTable[i].indexmax - 1 ]);
    }
}


struct cmpStruct
{
    cmpStruct(std::vector< std::vector<DTYPE> > points) {this->points = points;}
    bool operator() (int a, int b)
    {
        return points[a][0] < points[b][0];
    }

    std::vector< std::vector<DTYPE> > points;
};


void populateNDGridIndexAndLookupArray(
        std::vector<std::vector <DTYPE> > * NDdataPoints,
        DTYPE epsilon,
        struct gridCellLookup ** gridCellLookupArr,
        struct grid ** index,
        unsigned int * indexLookupArr,
        DTYPE* minArr,
        unsigned int * nCells,
        uint64_t totalCells,
        unsigned int * nNonEmptyCells,
        unsigned int ** gridCellNDMask,
        unsigned int * gridCellNDMaskOffsets,
        unsigned int * nNDMaskElems)
{

    /////////////////////////////////
    // Populate grid lookup array
    // and corresponding indicies in the lookup array
    /////////////////////////////////
    printf("\n\n*****************************  Populating Grid Index and lookup array  *****************************\n");
    // printf("\nSize of dataset: %lu", NDdataPoints->size());


    ///////////////////////////////
    // First, we need to figure out how many non-empty cells there will be
    // For memory allocation
    // Need to do a scan of the dataset and calculate this
    // Also need to keep track of the list of uniquie linear grid cell IDs for inserting into the grid
    ///////////////////////////////
    std::set<uint64_t> uniqueGridCellLinearIds;
    std::vector<uint64_t>uniqueGridCellLinearIdsVect; //for random access

    for (int i = 0; i < NDdataPoints->size(); ++i)
    {
        unsigned int tmpNDCellIdx[NUMINDEXEDDIM];
        for (int j = 0; j < NUMINDEXEDDIM; j++)
        {
            tmpNDCellIdx[j] = (((*NDdataPoints)[i][j] - minArr[j]) / epsilon);
        }
        uint64_t linearID = getLinearID_nDimensions(tmpNDCellIdx, nCells, NUMINDEXEDDIM);
        uniqueGridCellLinearIds.insert(linearID);

    }

    // printf("uniqueGridCellLinearIds: %d",uniqueGridCellLinearIds.size());

    //copy the set to the vector (sets can't do binary searches -- no random access)
    std::copy(uniqueGridCellLinearIds.begin(), uniqueGridCellLinearIds.end(), std::back_inserter(uniqueGridCellLinearIdsVect));


    ///////////////////////////////////////////////


    std::vector<uint64_t> * gridElemIDs;
    gridElemIDs = new std::vector<uint64_t>[uniqueGridCellLinearIds.size()];

    //Create ND array mask:
    //This mask determines which cells in each dimension has points in them.
    std::set<unsigned int> NDArrMask[NUMINDEXEDDIM];

    std::vector<uint64_t>::iterator lower;


    for (int i = 0; i < NDdataPoints->size(); i++)
    {
        unsigned int tmpNDCellID[NUMINDEXEDDIM];
        for (int j = 0; j < NUMINDEXEDDIM; j++)
        {
            tmpNDCellID[j] = (((*NDdataPoints)[i][j] - minArr[j]) / epsilon);

            //add value to the ND array mask
            NDArrMask[j].insert(tmpNDCellID[j]);
        }

        //get the linear id of the cell
        uint64_t linearID = getLinearID_nDimensions(tmpNDCellID, nCells, NUMINDEXEDDIM);
        //printf("\nlinear id: %d",linearID);
        //if (linearID > totalCells){

        //	printf("\n\nERROR Linear ID is: %lu, total cells is only: %lu\n\n", linearID, totalCells);
        //}

        //find the index in gridElemIds that corresponds to this grid cell linear id

        lower = std::lower_bound(uniqueGridCellLinearIdsVect.begin(), uniqueGridCellLinearIdsVect.end(), linearID);
        uint64_t gridIdx = lower - uniqueGridCellLinearIdsVect.begin();
        gridElemIDs[gridIdx].push_back(i);
    }




    ///////////////////////////////
    //Here we fill a temporary index with points, and then copy the non-empty cells to the actual index
    ///////////////////////////////

    struct grid * tmpIndex = new grid[uniqueGridCellLinearIdsVect.size()];

    int cnt = 0;



    //populate temp index and lookup array

    for (int i = 0; i < uniqueGridCellLinearIdsVect.size(); i++)
    {
        tmpIndex[i].indexmin = cnt;
        for (int j = 0; j < gridElemIDs[i].size(); j++)
        {
            if ((NDdataPoints->size() - 1) < j)
            {
                printf("\n\n***ERROR Value of a data point is larger than the dataset! %d\n\n", j);
                return;
            }
            indexLookupArr[cnt] = gridElemIDs[i][j];
            cnt++;
        }
        tmpIndex[i].indexmax = cnt - 1;
    }

    // printf("\nExiting grid populate method early!");
    // return;

    printf("\nFull cells: %d (%f, fraction full)", (unsigned int)uniqueGridCellLinearIdsVect.size(), uniqueGridCellLinearIdsVect.size() * 1.0 / double(totalCells));
    printf("\nEmpty cells: %ld (%f, fraction empty)", totalCells - (unsigned int)uniqueGridCellLinearIdsVect.size(), (totalCells - uniqueGridCellLinearIdsVect.size() * 1.0) / double(totalCells));

    *nNonEmptyCells = uniqueGridCellLinearIdsVect.size();


    printf("\nSize of index that would be sent to GPU (GiB) -- (if full index sent), excluding the data lookup arr: %f", (double)sizeof(struct grid) * (totalCells) / (1024.0 * 1024.0 * 1024.0));
    printf("\nSize of compressed index to be sent to GPU (GiB) , excluding the data and grid lookup arr: %f", (double)sizeof(struct grid) * (uniqueGridCellLinearIdsVect.size() * 1.0) / (1024.0 * 1024.0 * 1024.0));


    /////////////////////////////////////////
    //copy the tmp index into the actual index that only has the non-empty cells

    //allocate memory for the index that will be sent to the GPU
    *index = new grid[uniqueGridCellLinearIdsVect.size()];
    *gridCellLookupArr = new struct gridCellLookup[uniqueGridCellLinearIdsVect.size()];

    cmpStruct theStruct(*NDdataPoints);

    for (int i = 0; i < uniqueGridCellLinearIdsVect.size(); i++)
    {
        (*index)[i].indexmin = tmpIndex[i].indexmin;
        (*index)[i].indexmax = tmpIndex[i].indexmax;
        (*gridCellLookupArr)[i].idx = i;
        (*gridCellLookupArr)[i].gridLinearID = uniqueGridCellLinearIdsVect[i];
    }

    printf("\nWhen copying from entire index to compressed index: number of non-empty cells: %lu", uniqueGridCellLinearIdsVect.size());

    //copy NDArrMask from set to an array

    //find the total size and allocate the array

    unsigned int cntNDOffsets = 0;
    unsigned int cntNonEmptyNDMask = 0;
    for (int i = 0; i < NUMINDEXEDDIM; i++)
    {
        cntNonEmptyNDMask += NDArrMask[i].size();
    }
    *gridCellNDMask = new unsigned int[cntNonEmptyNDMask];

    *nNDMaskElems = cntNonEmptyNDMask;


    //copy the offsets to the array
    for (int i = 0; i < NUMINDEXEDDIM; i++)
    {
        //Min
        gridCellNDMaskOffsets[(i * 2)] = cntNDOffsets;
        for (std::set<unsigned int>::iterator it = NDArrMask[i].begin(); it != NDArrMask[i].end(); ++it)
        {
            (*gridCellNDMask)[cntNDOffsets] = *it;
            cntNDOffsets++;
        }
        //max
        gridCellNDMaskOffsets[(i * 2) + 1]  =cntNDOffsets - 1;
    }

    delete [] tmpIndex;
}


uint64_t getLinearID_nDimensions(
        unsigned int * indexes,
        unsigned int * dimLen,
        unsigned int nDimensions)
{
    uint64_t index = 0;
    uint64_t multiplier = 1;
    for (int i = 0; i < nDimensions; i++)
    {
        index += (uint64_t)indexes[i] * multiplier;
        multiplier *= dimLen[i];
    }

    return index;
}
