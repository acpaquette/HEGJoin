#include "hip/hip_runtime.h"
//precompute direct neighbors with the GPU:
#include "GPU.h"
#include "kernel.h"
#include "SortByWorkload.h"
#include "structs.h"
#include "params.h"
#include "WorkQueue.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <unistd.h>
#include "omp.h"

#include <pthread.h>

// //thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> // for streams for thrust (added with Thrust v1.8)
//
// //for warming up GPU:
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

//elements for the result set
//FOR A SINGLE KERNEL INVOCATION
//NOT FOR THE BATCHED ONE
#define BUFFERELEM 300000000 //400000000-original (when removing the data from the device before putting it back for the sort)

//FOR THE BATCHED EXECUTION:
//#define BATCHTOTALELEM 1200000000 //THE TOTAL SIZE ALLOCATED ON THE HOST
//THE NUMBER OF BATCHES AND THE SIZE OF THE BUFFER FOR EACH KERNEL EXECUTION ARE NOT RELATED TO THE TOTAL NUMBER
//OF ELEMENTS (ABOVE).
// #define NUMBATCHES 20
// #define BATCHBUFFERELEM 100000000 //THE SMALLER SIZE ALLOCATED ON THE DEVICE FOR EACH KERNEL EXECUTION

// #define GPUSTREAMS 1 //number of concurrent gpu streams, now defined in params.h

using std::cout;
using std::endl;

//sort ascending
bool compareByPointValue(const key_val_sort &a, const key_val_sort &b)
{
    return a.value_at_dim < b.value_at_dim;
}

uint64_t getLinearID_nDimensions2(unsigned int * indexes, unsigned int * dimLen, unsigned int nDimensions) {
    uint64_t index = 0;
	uint64_t multiplier = 1;
	for (int i = 0; i<nDimensions; i++){
  	     index += (uint64_t)indexes[i] * multiplier;
  	      multiplier *= dimLen[i];
	}

	return index;
}





////////////////////////////////////////////////////////////////////////////////





void gridIndexingGPU(
    unsigned int * DBSIZE,
    uint64_t totalCells,
    DTYPE * database,
    DTYPE ** dev_database,
    DTYPE * epsilon,
    DTYPE ** dev_epsilon,
    DTYPE * minArr,
    DTYPE ** dev_minArr,
    struct grid ** index,
    struct grid ** dev_index,
    unsigned int * indexLookupArr,
    unsigned int ** dev_indexLookupArr,
    struct gridCellLookup ** gridCellLookupArr,
    struct gridCellLookup ** dev_gridCellLookupArr,
    unsigned int * nNonEmptyCells,
    unsigned int ** dev_nNonEmptyCells,
    unsigned int * nCells,
    unsigned int ** dev_nCells)
{

    hipError_t errCode;

    double tStartAllocGPU = omp_get_wtime();

    errCode = hipMalloc( (void**)dev_database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc database -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_epsilon, sizeof(DTYPE));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc epsilon -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_minArr, sizeof(DTYPE) * (NUMINDEXEDDIM));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc minArr -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_indexLookupArr, sizeof(unsigned int) * (*DBSIZE));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: lookup array allocation -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_nNonEmptyCells, sizeof(unsigned int));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc nNonEmptyCells -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_nCells, sizeof(unsigned int) * (NUMINDEXEDDIM));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc nCells -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    uint64_t * dev_pointCellArr;
    errCode = hipMalloc((void**)&dev_pointCellArr, sizeof(uint64_t) * (*DBSIZE));
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: point cell array alloc -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    unsigned int * dev_databaseVal;
	errCode = hipMalloc((void**)&dev_databaseVal, sizeof(unsigned int) * (*DBSIZE));
	if (errCode != hipSuccess) {
    	cout << "[INDEX] ~ Error: Alloc databaseVal -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    unsigned int * N = new unsigned int;
	unsigned int * dev_N;
	errCode = hipMalloc((void**)&dev_N, sizeof(unsigned int) * GPUSTREAMS);
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc dev_N -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    double tEndAllocGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to allocate on the GPU: " << tEndAllocGPU - tStartAllocGPU << "\n\n";
    cout.flush();



    ////////////////////////////////////////////////////////////////////////////



    double tStartCopyGPU = omp_get_wtime();

    errCode = hipMemcpy( (*dev_database), database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE), hipMemcpyHostToDevice );
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: database copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_epsilon), epsilon, sizeof(DTYPE), hipMemcpyHostToDevice );
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: epsilon copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_minArr), minArr, sizeof(DTYPE) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Copy minArr to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_nCells), nCells, sizeof(unsigned int) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Copy nCells to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(dev_N, DBSIZE, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: database size Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    double tEndCopyGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to copy to the GPU: " << tEndCopyGPU - tStartCopyGPU << "\n\n";
    cout.flush();



    ////////////////////////////////////////////////////////////////////////////



    const int TOTALBLOCKS = ceil((1.0 * (*DBSIZE)) / (1.0 * BLOCKSIZE));
	printf("[INDEX] ~ Total blocks: %d\n",TOTALBLOCKS);

	kernelIndexComputeNonemptyCells<<<TOTALBLOCKS, BLOCKSIZE>>>((*dev_database), dev_N, (*dev_epsilon), (*dev_minArr),
            (*dev_nCells), dev_pointCellArr, nullptr, false);
    hipDeviceSynchronize();

    thrust::device_ptr<uint64_t> dev_pointCellArr_ptr(dev_pointCellArr);
	thrust::device_ptr<uint64_t> dev_new_end;

	try
    {
		//first sort
		thrust::sort(thrust::device, dev_pointCellArr_ptr, dev_pointCellArr_ptr + (*DBSIZE)); //, thrust::greater<uint64_t>()
		//then unique
		dev_new_end = thrust::unique(thrust::device, dev_pointCellArr_ptr, dev_pointCellArr_ptr + (*DBSIZE));
	}
	catch(std::bad_alloc &e)
	{
	 	std::cerr << "[INDEX] ~ Ran out of memory while sorting" << std::endl;
	    exit(-1);
    }

    uint64_t * new_end = thrust::raw_pointer_cast(dev_new_end);
    uint64_t numNonEmptyCells = std::distance(dev_pointCellArr_ptr, dev_new_end);
    printf("[INDEX] ~ Number of full cells (non-empty): %lu\n", numNonEmptyCells);
    *nNonEmptyCells = numNonEmptyCells;

    (*gridCellLookupArr) = new struct gridCellLookup[numNonEmptyCells];
    uint64_t * pointCellArrTmp = new uint64_t[numNonEmptyCells];
    errCode = hipMemcpy(pointCellArrTmp, dev_pointCellArr, sizeof(uint64_t) * numNonEmptyCells, hipMemcpyDeviceToHost);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: pointCellArrTmp memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	for (uint64_t i = 0; i < numNonEmptyCells; ++i)
	{
		(*gridCellLookupArr)[i].idx = i;
		(*gridCellLookupArr)[i].gridLinearID = pointCellArrTmp[i];
	}

    kernelIndexComputeNonemptyCells<<<TOTALBLOCKS, BLOCKSIZE>>>((*dev_database), dev_N, (*dev_epsilon), (*dev_minArr),
            (*dev_nCells), dev_pointCellArr, dev_databaseVal, true);

    try
	{
    	thrust::sort_by_key(thrust::device, dev_pointCellArr, dev_pointCellArr + (*DBSIZE), dev_databaseVal);
	}
	catch(std::bad_alloc &e)
	{
		std::cerr << "[INDEX] ~ Ran out of memory while sorting key/value pairs" << std::endl;
	    exit(-1);
	}

    uint64_t * cellKey = new uint64_t[(*DBSIZE)];
    errCode = hipMemcpy(cellKey, dev_pointCellArr, sizeof(uint64_t) * (*DBSIZE), hipMemcpyDeviceToHost);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: pointCellArr memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(indexLookupArr, dev_databaseVal, sizeof(unsigned int) * (*DBSIZE), hipMemcpyDeviceToHost);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: databaseIDValue memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    (*index) = new grid[numNonEmptyCells];
    (*index)[0].indexmin = 0;
	uint64_t cnt=0;
	for (uint64_t i = 1; i < (*DBSIZE); ++i)
    {
		if (cellKey[i - 1] != cellKey[i])
		{
			//grid index
			cnt++;
			(*index)[cnt].indexmin = i;
			(*index)[cnt - 1].indexmax = i - 1;
		}
	}
    delete[] cellKey;
	(*index)[numNonEmptyCells - 1].indexmax = (*DBSIZE) - 1;

    printf("[INDEX] ~ Full cells: %d (%f, fraction full)\n", (unsigned int)numNonEmptyCells, numNonEmptyCells * 1.0 / double(totalCells));
	printf("[INDEX] ~ Empty cells: %ld (%f, fraction empty)\n", totalCells - (unsigned int)numNonEmptyCells, (totalCells - numNonEmptyCells * 1.0) / double(totalCells));
	printf("[INDEX] ~ Size of index that would be sent to GPU (GiB) -- (if full index sent), excluding the data lookup arr: %f\n",
        (double)sizeof(struct grid) * (totalCells) / (1024.0 * 1024.0 * 1024.0));
	printf("[INDEX] ~ Size of compressed index to be sent to GPU (GiB), excluding the data and grid lookup arr: %f\n",
        (double)sizeof(struct grid) * (numNonEmptyCells * 1.0) / (1024.0 * 1024.0 * 1024.0));
	printf("[INDEX] ~ When copying from entire index to compressed index: number of non-empty cells: %lu\n", numNonEmptyCells);

    ////////////////////////////////////////////////////////////////////////////

    errCode = hipMalloc( (void**)dev_index, sizeof(struct grid) * (*nNonEmptyCells));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc grid index -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_gridCellLookupArr, sizeof(struct gridCellLookup) * (*nNonEmptyCells));
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: copy grid cell lookup array allocation -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    ////////////////////////////////////////////////////////////////////////////

    errCode = hipMemcpy( (*dev_nNonEmptyCells), nNonEmptyCells, sizeof(unsigned int), hipMemcpyHostToDevice );
	if (errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: nNonEmptyCells copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy((*dev_index), (*index), sizeof(struct grid) * numNonEmptyCells, hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: index copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy((*dev_indexLookupArr), indexLookupArr, sizeof(unsigned int) * (*DBSIZE), hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: index lookup array copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy((*dev_gridCellLookupArr), (*gridCellLookupArr), sizeof(struct gridCellLookup) * numNonEmptyCells, hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: grid lookup array copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    ////////////////////////////////////////////////////////////////////////////

    delete N;
    delete[] pointCellArrTmp;
    hipFree(dev_pointCellArr);
    hipFree(dev_databaseVal);
    hipFree(dev_N);

    double tEndIndexGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to index using the GPU (including allocating and transfering memory): " << tEndIndexGPU - tStartAllocGPU << '\n';
    cout.flush();

}




unsigned long long GPUBatchEst_v2(
    int searchMode,
    unsigned int * DBSIZE,
    float staticPartition,
    DTYPE * dev_database,
    unsigned int * dev_originPointIndex,
    DTYPE * dev_epsilon,
    struct grid * dev_grid,
    unsigned int * dev_indexLookupArr,
    struct gridCellLookup * dev_gridCellLookupArr,
    DTYPE * dev_minArr,
    unsigned int * dev_nCells,
    unsigned int * dev_nNonEmptyCells,
    unsigned int * retNumBatches,
    unsigned int * retGPUBufferSize,
    std::vector< std::pair<unsigned int, unsigned int> > * batches)
{

    hipError_t errCode;

    cout << "[GPU] ~ Estimating batches\n";

    // Parameters for the batch size estimation.
    double sampleRate = 0.10;
    int offsetRate = 1.0 / sampleRate;
    cout << "[GPU] ~ Sample rate: " << sampleRate << ", offset: " << offsetRate << '\n';

    /////////////////
	// N GPU threads
	////////////////

    unsigned int * dev_N_batchEst;
    unsigned int * N_batchEst = new unsigned int;

    unsigned int partitionedDBSIZE = (*DBSIZE) * staticPartition;

    if (SM_HYBRID_STATIC == searchMode && STATIC_SPLIT_QUERIES)
    {
        // Split the worked based on the number of queries, so also reduce the number of queries to estimate
        (*N_batchEst) = partitionedDBSIZE * sampleRate;
    } else {
        // Searchmode is either GPU alone, dynamic hybrid, or the workload is statically split
        //  based on the number of candidate points to refine, and so we estimate all the query points
        //  in all mentionned cases
        (*N_batchEst) = (*DBSIZE) * sampleRate;
    }

    errCode = hipMalloc((void**)&dev_N_batchEst, sizeof(unsigned int));
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_N_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(dev_N_batchEst, N_batchEst, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
	    cout << "[GPU] ~ Error: N batchEST Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    /////////////
	// count the result set size
	////////////

    unsigned int * dev_cnt_batchEst;
    unsigned int * cnt_batchEst = new unsigned int;
    (*cnt_batchEst) = 0;

    errCode = hipMalloc((void**)&dev_cnt_batchEst, sizeof(unsigned int));
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_cnt_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(dev_cnt_batchEst, cnt_batchEst, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_cnt_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    //////////////////
	// Sample offset - To sample the data to estimate the total number of key/value pairs
	/////////////////

    unsigned int * dev_sampleOffset;
    unsigned int * sampleOffset = new unsigned int;
    (*sampleOffset) = offsetRate;

    errCode = hipMalloc((void**)&dev_sampleOffset, sizeof(unsigned int));
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: sample offset Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(dev_sampleOffset, sampleOffset, sizeof(unsigned int), hipMemcpyHostToDevice);
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_sampleOffset Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    //////////////////
    // To save the estimated number of neighbors of points
    //////////////////

    unsigned int * dev_estimatedResult;
    unsigned int * estimatedResult = new unsigned int[(*N_batchEst)];

    errCode = hipMalloc((void**)&dev_estimatedResult, (*N_batchEst) * sizeof(unsigned int));
	if (errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: estimated result Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    int nbBlockTmp;
    if (searchMode == SM_HYBRID_STATIC)
    {
        #if STATIC_SPLIT_QUERIES
            nbBlockTmp = ceil((1.0 * partitionedDBSIZE * sampleRate) / (1.0 * BLOCKSIZE));
        #else
            nbBlockTmp = ceil((1.0 * (*DBSIZE) * sampleRate) / (1.0 * BLOCKSIZE));
        #endif
    } else {
        nbBlockTmp = ceil((1.0 * (*DBSIZE) * sampleRate) / (1.0 * BLOCKSIZE));
    }
    cout << "[GPU] ~ Total blocks: " << nbBlockTmp << '\n';
    cout.flush();

    cout << "[GPU] ~ Estimating batch without using pattern\n";
    cout.flush();

    const int TOTALBLOCKSBATCHEST = nbBlockTmp;


    #if SORT_BY_WORKLOAD
        kernelNDGridIndexBatchEstimator_v2<<<TOTALBLOCKSBATCHEST, BLOCKSIZE>>>(dev_N_batchEst, dev_sampleOffset,
            dev_database, dev_originPointIndex, dev_epsilon, dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr,
            dev_nCells, dev_cnt_batchEst, dev_nNonEmptyCells, dev_estimatedResult);
    #else
        kernelNDGridIndexBatchEstimator_v2<<<TOTALBLOCKSBATCHEST, BLOCKSIZE>>>(dev_N_batchEst, dev_sampleOffset,
            dev_database, nullptr, dev_epsilon, dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr,
            dev_nCells, dev_cnt_batchEst, dev_nNonEmptyCells, dev_estimatedResult);
    #endif


    cout << "[GPU] ~ ERROR FROM KERNEL LAUNCH OF BATCH ESTIMATOR: " << hipGetLastError() << '\n';
    cout.flush();

    errCode = hipMemcpy(cnt_batchEst, dev_cnt_batchEst, sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (errCode != hipSuccess)
    {
	    cout << "[GPU] ~ Error: getting cnt for batch estimate from GPU Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	} else {
        cout << "[GPU] ~ Result set size for estimating the number of batches (sampled): " << *cnt_batchEst << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(estimatedResult, dev_estimatedResult, (*N_batchEst) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (errCode != hipSuccess)
    {
	    cout << "[GPU] ~ Error: getting estimated results for batch estimate from GPU Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    unsigned int GPUBufferSize = 50000000;
    // unsigned int GPUBufferSize = 100000000;

    // uint64_t estimatedNeighbors = (uint64_t)*cnt_batchEst * (uint64_t)offsetRate;
    // cout << "[GPU] ~ From GPU cnt: " << *cnt_batchEst <<", offset rate: " << offsetRate << '\n';
    // cout.flush();

    unsigned long long fullEst = 0;
    unsigned int * estimatedFull;

    unsigned int nbUnestimatedSequences;

    if (SM_HYBRID_STATIC == searchMode)
    {
        #if STATIC_SPLIT_QUERIES
            nbUnestimatedSequences = partitionedDBSIZE / (*sampleOffset);
            estimatedFull = new unsigned int[partitionedDBSIZE];
        #else
            nbUnestimatedSequences = (*DBSIZE) / (*sampleOffset);
            estimatedFull = new unsigned int[(*DBSIZE)];
        #endif
    } else {
        nbUnestimatedSequences = (*DBSIZE) / (*sampleOffset);
        estimatedFull = new unsigned int[(*DBSIZE)];
    }

    for (int i = 0; i < nbUnestimatedSequences - 1; ++i)
    {
        unsigned int nbEstBefore = estimatedResult[i];
        unsigned int nbEstAfter = estimatedResult[i + 1];
        unsigned int maxEst = (nbEstBefore < nbEstAfter) ? nbEstAfter : nbEstBefore;

        unsigned int estBefore = i * (*sampleOffset);
        unsigned int estAfter = (i + 1) * (*sampleOffset);
        estimatedFull[estBefore] = nbEstBefore;
        fullEst += nbEstBefore;

        for (int j = estBefore + 1; j < estAfter; ++j)
        {
            #if SORT_BY_WORKLOAD
                estimatedFull[j] = maxEst;
                fullEst += maxEst;
            #else
                // If we do not sort by workload, then we can not assume that the work is in non-increasing order,
                // and thus that the used estimator is "correct", so we overestimate the estimation to compensate,
                // similarly as in the original algorithm
                estimatedFull[j] = maxEst + maxEst * sampleRate;
                fullEst += maxEst + maxEst * sampleRate;
            #endif
        }

    }

    cout << "[GPU | RESULT] ~ Total estimated workload: " << fullEst << '\n';

    if (searchMode == SM_HYBRID_STATIC)
    {
        // Not enough work to fill at least GPUSTREAMS batches, so reducing GPUBufferSize so the
        // GPU can fully use its GPUSTREAMS streams
        // Used if the work is statically partitioned, as the CPU will always have some work reserved
        if (fullEst < (GPUBufferSize * GPUSTREAMS))
        {
            GPUBufferSize = fullEst / (GPUSTREAMS);
            cout << "[GPU] ~ Too few batches, reducing GPUBufferSize to " << GPUBufferSize << '\n';
        }
    } else {
        // Not enough work to fill at least 6 batches (2 * GPUSTREAMS)
        // So we force to have at least 6 batches so all streams can be used, and the CPU as well
        // Used if the work is dynamically partitioned (work queue), so the CPU can have some work
        if (fullEst < (GPUBufferSize * GPUSTREAMS * 2))
        {
            GPUBufferSize = fullEst / (GPUSTREAMS * 2);
            cout << "[GPU] ~ Too few batches, reducing GPUBufferSize to " << GPUBufferSize << '\n';
        }
    }

    unsigned int batchBegin = 0;
    unsigned int batchEnd = 0;
    unsigned long long runningEst = 0;
    // Keeping 5% of margin to avoid a potential overflow of the buffer
    unsigned int reserveBuffer = GPUBufferSize * 0.05;

    if (searchMode == SM_HYBRID_STATIC)
    {
        #if STATIC_SPLIT_QUERIES
            for (int i = 0; i < partitionedDBSIZE; ++i)
            {
                runningEst += estimatedFull[i];
                // fullEst += estimatedFull[i];
                if ((GPUBufferSize - reserveBuffer) <= runningEst)
                {
                    batchEnd = i;
                    batches->push_back(std::make_pair(batchBegin, batchEnd));
                    batchBegin = i;
                    runningEst = 0;
                } else {
                    // The last batch may not fulfill the above condition of filling a result buffer
                    if (partitionedDBSIZE - 1 == i)
                    {
                        batchEnd = partitionedDBSIZE;
                        batches->push_back(std::make_pair(batchBegin, batchEnd));
                    }
                }
            }
            printf("[GPU | RESULT] ~ %u query points allocated to the GPU, with %llu estimated candidates\n", partitionedDBSIZE, runningEst);
            printf("[GPU | RESULT] ~ %u query points allocated to the CPU, with %llu estimated candidates\n", (*DBSIZE) - partitionedDBSIZE, fullEst - runningEst);
            setQueueIndex(partitionedDBSIZE);
        #else // Static partitioning based on the number candidate points to refine
            // unsigned long long partitionedCandidates = fullEst * staticPartition;
            // runningEst = 0;
            // unsigned long long runningEstBatch = 0;
            // unsigned int queryPoint = 0;
            // while (runningEst < partitionedCandidates)
            // {
            //     runningEst += estimatedFull[queryPoint];
            //     runningEstBatch += estimatedFull[queryPoint];
            //     if ((GPUBufferSize - reserveBuffer) <= runningEstBatch)
            //     {
            //         batchEnd = queryPoint;
            //         batches->push_back(std::make_pair(batchBegin, batchEnd));
            //         batchBegin = queryPoint;
            //         runningEstBatch = 0;
            //     }
            //     queryPoint++;
            // }
            // batchEnd = queryPoint;
            // batches->push_back(std::make_pair(batchBegin, batchEnd));
            for (int i = 0; i < (*DBSIZE); ++i)
            {
                runningEst += estimatedFull[i];
                // fullEst += estimatedFull[i];
                if ((GPUBufferSize - reserveBuffer) <= runningEst)
                {
                    batchEnd = i;
                    batches->push_back(std::make_pair(batchBegin, batchEnd));
                    batchBegin = i;
                    runningEst = 0;
                } else {
                    // The last batch may not fulfill the above condition of filling a result buffer
                    if ((*DBSIZE) - 1 == i)
                    {
                        batchEnd = (*DBSIZE);
                        batches->push_back(std::make_pair(batchBegin, batchEnd));
                    }
                }
            }

            // printf("[GPU | RESULT] ~ %u query points allocated to the GPU, with %llu estimated candidates\n", queryPoint, runningEst);
            // printf("[GPU | RESULT] ~ %u query points allocated to the CPU, with %llu estimated candidates\n", (*DBSIZE) - queryPoint, fullEst - runningEst);
            setQueueIndex((*DBSIZE));
        #endif
        fullEst = runningEst;
    } else {
        for (int i = 0; i < (*DBSIZE); ++i)
        {
            runningEst += estimatedFull[i];
            // fullEst += estimatedFull[i];
            if ((GPUBufferSize - reserveBuffer) <= runningEst)
            {
                batchEnd = i;
                batches->push_back(std::make_pair(batchBegin, batchEnd));
                batchBegin = i;
                runningEst = 0;
            } else {
                // The last batch may not fulfill the above condition of filling a result buffer
                if ((*DBSIZE) - 1 == i)
                {
                    batchEnd = (*DBSIZE);
                    batches->push_back(std::make_pair(batchBegin, batchEnd));
                }
            }
        }
        // setQueueIndex((batches[GPUSTREAMS]).first);
    }

    cout << "[GPU] ~ Estimated total result set size: " << fullEst << '\n';
    cout << "[GPU] ~ Number of batches: " << batches->size() << '\n';
    cout.flush();

    (*retNumBatches) = batches->size();
    (*retGPUBufferSize) = GPUBufferSize;

    cout << "[GPU] ~ Done estimating batches\n";

    hipFree(dev_cnt_batchEst);
    hipFree(dev_N_batchEst);
    hipFree(dev_sampleOffset);
    hipFree(dev_estimatedResult);

    delete[] estimatedResult;
    delete[] estimatedFull;
    delete N_batchEst;
    delete cnt_batchEst;
    delete sampleOffset;

    return fullEst;

}






//modified from: makeDistanceTableGPUGridIndexBatchesAlternateTest
void distanceTableNDGridBatches(
        int searchMode,
        float staticPartition,
        unsigned int * DBSIZE,
        DTYPE * epsilon,
        DTYPE * dev_epsilon,
        DTYPE * database,
        DTYPE * dev_database,
        struct grid * grid,
        struct grid * dev_grid,
        unsigned int * indexLookupArr,
        unsigned int * dev_indexLookupArr,
        struct gridCellLookup * gridCellLookupArr,
        struct gridCellLookup * dev_gridCellLookupArr,
        DTYPE * minArr,
        DTYPE * dev_minArr,
        unsigned int * nCells,
        unsigned int * dev_nCells,
        unsigned int * nNonEmptyCells,
        unsigned int * dev_nNonEmptyCells,
        // unsigned int * gridCellNDMask,
        // unsigned int * dev_gridCellNDMask,
        // unsigned int * gridCellNDMaskOffsets,
        // unsigned int * dev_gridCellNDMaskOffsets,
        // unsigned int * nNDMaskElems,
        unsigned int * originPointIndex,
        unsigned int * dev_originPointIndex,
        struct neighborTableLookup * neighborTable,
        std::vector<struct neighborDataPtrs> * pointersToNeighbors,
        uint64_t * totalNeighbors,
        unsigned int * nbQueriesGPU)
{
    double tKernelResultsStart = omp_get_wtime();

  	//CUDA error code:
  	hipError_t errCode;

  	cout << "\n[GPU] ~ Sometimes the GPU will error on a previous execution and you won't know. \n[GPU] ~ Last error start of function: " << hipGetLastError() << '\n';
    cout.flush();



  	///////////////////////////////////
  	//COUNT VALUES -- RESULT SET SIZE FOR EACH KERNEL INVOCATION
  	///////////////////////////////////

  	//total size of the result set as it's batched
  	//this isnt sent to the GPU
  	unsigned int * totalResultSetCnt = new unsigned int;
  	*totalResultSetCnt = 0;

  	//count values - for an individual kernel launch
  	//need different count values for each stream
  	unsigned int * cnt;
  	cnt = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);
  	*cnt = 0;

  	unsigned int * dev_cnt;
  	dev_cnt = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);
  	*dev_cnt = 0;

    for (int i = 0; i < (GPUSTREAMS); i++) {
        cnt[i] = 0;
        dev_cnt[i] = 0;
    }

  	//allocate on the device
  	errCode = hipMalloc((void**)&dev_cnt, sizeof(unsigned int) * GPUSTREAMS);
  	if (errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc cnt -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
  	}

  	///////////////////////////////////
  	//END COUNT VALUES -- RESULT SET SIZE FOR EACH KERNEL INVOCATION
  	///////////////////////////////////



  	////////////////////////////////////
  	//NUMBER OF THREADS PER GPU STREAM
  	////////////////////////////////////

  	//THE NUMBER OF THREADS THAT ARE LAUNCHED IN A SINGLE KERNEL INVOCATION
  	//CAN BE FEWER THAN THE NUMBER OF ELEMENTS IN THE DATABASE IF MORE THAN 1 BATCH
  	unsigned int * N = new unsigned int[GPUSTREAMS];

  	unsigned int * dev_N;
  	// dev_N = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

  	//allocate on the device
  	errCode = hipMalloc((void**)&dev_N, sizeof(unsigned int) * GPUSTREAMS);
  	if (errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc dev_N -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

  	////////////////////////////////////
  	//NUMBER OF THREADS PER GPU STREAM
  	////////////////////////////////////



    /////////////////////////////////////////////////////////
  	//BEGIN BATCH ESTIMATOR
  	/////////////////////////////////////////////////////////

  	unsigned long long estimatedNeighbors = 0;
  	unsigned int numBatches = 0;
  	unsigned int GPUBufferSize = 0;

    std::vector< std::pair<unsigned int, unsigned int> > batchesVector;

	double tstartbatchest = omp_get_wtime();
    if (SM_HYBRID_STATIC == searchMode)
    {
        #if STATIC_SPLIT_QUERIES
            #if SORT_BY_WORKLOAD
                estimatedNeighbors = GPUBatchEst_v2(searchMode, DBSIZE, staticPartition, dev_database, dev_originPointIndex, dev_epsilon, dev_grid, dev_indexLookupArr,
                        dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
            #else
                estimatedNeighbors = GPUBatchEst_v2(searchMode, DBSIZE, staticPartition, dev_database, nullptr, dev_epsilon, dev_grid, dev_indexLookupArr,
                        dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
            #endif
        #else
            unsigned int nbQueryPointsStatic = getStaticQueryPoint();
            cout << "[GPU | DEBUG] ~ Number of queries for the GPU: " << nbQueryPointsStatic << '\n';
            #if SORT_BY_WORKLOAD
                estimatedNeighbors = GPUBatchEst_v2(searchMode, &nbQueryPointsStatic, staticPartition, dev_database, dev_originPointIndex, dev_epsilon, dev_grid, dev_indexLookupArr,
                        dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
            #else
                estimatedNeighbors = GPUBatchEst_v2(searchMode, &nbQueryPointsStatic, staticPartition, dev_database, nullptr, dev_epsilon, dev_grid, dev_indexLookupArr,
                        dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
            #endif
        #endif
    } else {
        #if SORT_BY_WORKLOAD
            estimatedNeighbors = GPUBatchEst_v2(searchMode, DBSIZE, staticPartition, dev_database, dev_originPointIndex, dev_epsilon, dev_grid, dev_indexLookupArr,
                    dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
        #else
            estimatedNeighbors = GPUBatchEst_v2(searchMode, DBSIZE, staticPartition, dev_database, nullptr, dev_epsilon, dev_grid, dev_indexLookupArr,
                    dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, &numBatches, &GPUBufferSize, &batchesVector);
        #endif
    }
    double tendbatchest = omp_get_wtime();

    cout << "[GPU] ~ Time to estimate batches: " << tendbatchest - tstartbatchest << '\n';
    cout.flush();

    cout << "[GPU] ~ In calling function: Estimated neighbors = " << estimatedNeighbors
         << ", num. batches = " << numBatches << ", GPU buffer size = " << GPUBufferSize << '\n';
    cout.flush();

    // cout << "[GPU] ~ Batches: \n";
    // for (int i = 0; i < batchesVector.size(); ++i)
    // {
    //     cout << "   [GPU] ~ " << batchesVector[i].first << ", " << batchesVector[i].second <<  '\n';
    // }

    // sets the batch size for the queue and the queue index, considering the offset reserved for the GPU
    // shouldn't happen anymore as we always have at least 2*GPUSTREAMS batches now
    // setQueueIndex(GPUSTREAMS * (*DBSIZE / numBatches));
    // if (batchesVector.size() < GPUSTREAMS)
    // {
    //     setQueueIndex((*DBSIZE)); // the GPU reserves all the computation
    // } else {
    if (searchMode != SM_HYBRID_STATIC)
    {
        setQueueIndex(batchesVector[GPUSTREAMS].first);
    }
      // }

    // setQueueIndex(0);

  	/////////////////////////////////////////////////////////
  	//END BATCH ESTIMATOR
  	/////////////////////////////////////////////////////////



  	///////////////////
  	//ALLOCATE POINTERS TO INTEGER ARRAYS FOR THE VALUES FOR THE NEIGHBORTABLES
  	///////////////////

	//THE NUMBER OF POINTERS IS EQUAL TO THE NUMBER OF BATCHES
    unsigned int datasetSize = *DBSIZE;
    size_t freeMem = 0;
    size_t totalMem = 0;
    int memGetSuccess = hipMemGetInfo(&freeMem, &totalMem);
    if (memGetSuccess != hipSuccess) {
        cout << "You probably don't have a gpu, in which case, how did you get this far?" << endl;
        exit(memGetSuccess);
    }
    #if !SILENT_GPU
        cout << "[GPU] ~ Total Memory: " << totalMem << '\n';
        cout.flush();
    #endif

    // Compute Total blocks computable in a single instances based on estimated size
    // Have to convert esitmatedNeighbors into bytes
    const unsigned int tpp = 32;
    cout << "DEFINED TPP AS " << tpp << endl;

    numBatches = ceil(1.0 * (estimatedNeighbors * 2 * sizeof(int)) / (0.9 * GPUBufferSize));
    uint64_t pointsPerBlock = BLOCKSIZE / tpp;
    uint64_t NBLOCKS = ceil(((1.0 * datasetSize) / (1.0 * numBatches)) / (1.0 * pointsPerBlock));
    // NBLOCKS = NBLOCKS + (NBLOCKS % BLOCKSIZE);
    // numBatches = datasetSize / NBLOCKS;
    cout << "[GPU] ~ Number of blocks for a single instance: " << NBLOCKS << endl;
    cout << "[GPU] ~ Total Iterations: " << numBatches << '\n';
    cout.flush();
    // #if !SILENT_GPU
    //     cout << "[GPU] ~ Number of blocks for a single instance: " << NBLOCKS << endl;
    //     cout << "[GPU] ~ Total Iterations: " << numBatches << '\n';
    //     cout.flush();
    // #endif
	for (int i = 0; i < numBatches; i++)
    {
		int *ptr;
		struct neighborDataPtrs tmpStruct;
		tmpStruct.dataPtr = ptr;
		tmpStruct.sizeOfDataArr = 0;

		pointersToNeighbors->push_back(tmpStruct);
	}

  	///////////////////
  	//END ALLOCATE POINTERS TO INTEGER ARRAYS FOR THE VALUES FOR THE NEIGHBORTABLES
  	///////////////////



  	///////////////////////////////////
  	//ALLOCATE MEMORY FOR THE RESULT SET USING THE BATCH ESTIMATOR
  	///////////////////////////////////

  	//NEED BUFFERS ON THE GPU AND THE HOST FOR THE NUMBER OF CONCURRENT STREAMS
  	//GPU BUFFER ON THE DEVICE
  	//BUFFER ON THE HOST WITH PINNED MEMORY FOR FAST MEMCPY
  	//BUFFER ON THE HOST TO DUMP THE RESULTS OF BATCHES SO THAT GPU THREADS CAN CONTINUE
  	//EXECUTING STREAMS ON THE HOST

	//GPU MEMORY ALLOCATION: key/value pairs

	int * dev_pointIDKey[GPUSTREAMS]; //key
	int * dev_pointInDistValue[GPUSTREAMS]; //value
	for (int i = 0; i < GPUSTREAMS; i++)
	{
		errCode = hipMalloc((void **)&dev_pointIDKey[i], 2 * sizeof(int) * GPUBufferSize);
		if (errCode != hipSuccess)
        {
			cout << "[GPU] ~ CUDA: Got error with code " << errCode << '\n'; //2 means not enough memory
            cout << "  Details: " << hipGetErrorString(errCode) << '\n';
            cout.flush();
		}

		errCode = hipMalloc((void **)&dev_pointInDistValue[i], 2 * sizeof(int) * GPUBufferSize);
		if (errCode != hipSuccess)
        {
			cout << "[GPU] ~ CUDA: Got error with code " << errCode << '\n'; //2 means not enough memory
            cout << "  Details: " << hipGetErrorString(errCode) << '\n';
            cout.flush();
		}

	}
    cout << "[GPU] ~ Allocation pointIDKey and pointInDistValue on the GPU, size = " << 2 * sizeof(int) * GPUBufferSize << '\n';
    cout.flush();

  	//HOST RESULT ALLOCATION FOR THE GPU TO COPY THE DATA INTO A PINNED MEMORY ALLOCATION
  	//ON THE HOST
  	//pinned result set memory for the host
  	//the number of elements are recorded for that batch in resultElemCountPerBatch
  	//NEED PINNED MEMORY ALSO BECAUSE YOU NEED IT TO USE STREAMS IN THRUST FOR THE MEMCOPY OF THE SORTED RESULTS
  	//can't do async copies without pinned memory

  	//PINNED MEMORY TO COPY FROM THE GPU
  	int * pointIDKey[GPUSTREAMS]; //key
  	int * pointInDistValue[GPUSTREAMS]; //value

  	double tstartpinnedresults = omp_get_wtime();

    #pragma omp parallel for num_threads(GPUSTREAMS)
  	for (int i = 0; i < GPUSTREAMS; i++)
  	{
  		  hipHostMalloc((void **) &pointIDKey[i], 2 * sizeof(int) * GPUBufferSize);
  		  hipHostMalloc((void **) &pointInDistValue[i], 2 * sizeof(int) * GPUBufferSize);
  	}

  	double tendpinnedresults = omp_get_wtime();

    cout << "[GPU] ~ Time to allocate pinned memory for results: " << tendpinnedresults - tstartpinnedresults << '\n';
    cout.flush();

  	// hipMalloc((void **) &pointIDKey, sizeof(int)*GPUBufferSize*NUMBATCHES);
  	// hipMalloc((void **) &pointInDistValue, sizeof(int)*GPUBufferSize*NUMBATCHES);

    cout << "[GPU] ~ Memory request for results on GPU (GiB): " << (double)(sizeof(int) * 2 * GPUBufferSize * GPUSTREAMS) / (1024 * 1024 * 1024) << '\n';
    cout.flush();
    cout << "[GPU] ~ Memory requested for results in MAIN MEMORY (GiB): " << (double)(sizeof(int) * 2 * GPUBufferSize * GPUSTREAMS) / (1024 * 1024 * 1024) << '\n';
    cout.flush();

  	///////////////////////////////////
  	//END ALLOCATE MEMORY FOR THE RESULT SET
  	///////////////////////////////////



  	/////////////////////////////////
  	//SET OPENMP ENVIRONMENT VARIABLES
  	////////////////////////////////

  	omp_set_num_threads(GPUSTREAMS);

  	/////////////////////////////////
  	//END SET OPENMP ENVIRONMENT VARIABLES
  	////////////////////////////////



  	/////////////////////////////////
  	//CREATE STREAMS
  	////////////////////////////////

  	hipStream_t stream[GPUSTREAMS];

  	for (int i = 0; i < GPUSTREAMS; i++)
    {
        hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
  	}



  	///////////////////////////////////
  	//LAUNCH KERNEL IN BATCHES
  	///////////////////////////////////

  	//since we use the strided scheme, some of the batch sizes
  	//are off by 1 of each other, a first group of batches will
  	//have 1 extra data point to process, and we calculate which batch numbers will
  	//have that.  The batchSize is the lower value (+1 is added to the first ones)


    // unsigned int batchSize = (*DBSIZE) / numBatches;
    unsigned int batchSize = datasetSize / numBatches;
    // unsigned int batchesThatHaveOneMore = (*DBSIZE) - (batchSize * numBatches); //batch number 0- < this value have one more
    unsigned int batchesThatHaveOneMore = datasetSize - (batchSize * numBatches);
    cout << "[GPU] ~ Batches that have one more GPU thread: " << batchesThatHaveOneMore << " batchSize(N): " << batchSize << '\n';
    cout.flush();

    uint64_t totalResultsLoop = 0;

    unsigned int * batchBegin = new unsigned int[GPUSTREAMS];
    for (int i = 0; i < GPUSTREAMS; i++)
    {
        batchBegin[i] = 0;
    }

    unsigned int * dev_batchBegin;
    errCode = hipMalloc( (void**)&dev_batchBegin, GPUSTREAMS * sizeof(unsigned int));
    if (errCode != hipSuccess)
    {
        cout << "[GPU] ~ Error: Alloc queue index -- error with code " << errCode << '\n';
        cout.flush();
    }

    hipEvent_t * startKernel = new hipEvent_t[GPUSTREAMS];
    hipEvent_t * stopKernel = new hipEvent_t[GPUSTREAMS];
    float * kernelTimes = new float[GPUSTREAMS];
    unsigned int * nbKernelInvocation = new unsigned int[GPUSTREAMS];
    unsigned int * nbQueryPoint = new unsigned int [GPUSTREAMS];
    double computeTime = 0;

    for (int i = 0; i < GPUSTREAMS; ++i)
    {
        hipEventCreate(&startKernel[i]);
        hipEventCreate(&stopKernel[i]);
        kernelTimes[i] = 0;
        nbKernelInvocation[i] = 0;
        nbQueryPoint[i] = 0;
    }

    #if GPU_LOCKING
    pthread_mutex_t gpu_lock = PTHREAD_MUTEX_INITIALIZER;
    pthread_mutex_init(&gpu_lock, NULL);
    #endif

    if (SM_HYBRID == searchMode)
    {
        unsigned int globalBatchCounter = GPUSTREAMS;

        double tStartCompute = omp_get_wtime();
        #pragma omp parallel reduction(+: totalResultsLoop) num_threads(GPUSTREAMS)
        {
            unsigned int tid = omp_get_thread_num();
            // std::pair<unsigned int, unsigned int> gpuBatch = std::make_pair(tid * batchSize, tid * batchSize + batchSize);
            std::pair<unsigned int, unsigned int> gpuBatch = batchesVector[tid];

            unsigned int localBatchCounter = tid;

            do
            {
                nbQueryPoint[tid] += gpuBatch.second - gpuBatch.first;
                #if !SILENT_GPU
                    printf("[GPU | T_%d] ~ New batch: begin = %d, end = %d\n", tid, gpuBatch.first, gpuBatch.second);
                #endif

                errCode = hipMemcpy( &dev_batchBegin[tid], &gpuBatch.first, sizeof(unsigned int), hipMemcpyHostToDevice );
            	if (errCode != hipSuccess)
                {
            		cout << "[GPU] ~ Error: queue index copy to device -- error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
            	}

                // N[tid] = batchSize;
                N[tid] = gpuBatch.second - gpuBatch.first;
                errCode = hipMemcpyAsync( &dev_N[tid], &N[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if (errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: N Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // the batched result set size (reset to 0):
        		cnt[tid] = 0;
        		errCode = hipMemcpyAsync( &dev_cnt[tid], &cnt[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if (errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: dev_cnt memcpy Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                const int TOTALBLOCKS = ceil( (1.0 * (N[tid])) / (1.0 * BLOCKSIZE) );
                #if !SILENT_GPU
                    cout << "[GPU] ~ Total blocks: " << TOTALBLOCKS << '\n';
                    cout.flush();
                #endif

                #if GPU_LOCKING
                pthread_mutex_lock(&gpu_lock);
                #endif
                // double beginKernel = omp_get_wtime();
                hipEventRecord(startKernel[tid], stream[tid]);
                #if SORT_BY_WORKLOAD
                    kernelNDGridIndexGlobal<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                        dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                        dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells,
                        dev_pointIDKey[tid], dev_pointInDistValue[tid], tpp);
                #else
                    kernelNDGridIndexGlobal<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                        dev_database, nullptr, nullptr, dev_epsilon, dev_grid,
                        dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells,
                        dev_pointIDKey[tid], dev_pointInDistValue[tid], tpp);
                #endif
                hipEventRecord(stopKernel[tid], stream[tid]);


                errCode = hipGetLastError();
                #if !SILENT_GPU
            		cout << "\n\n[GPU] ~ KERNEL LAUNCH RETURN: " << errCode << '\n';
                    cout.flush();
                #endif
        		if ( hipSuccess != hipGetLastError() )
                {
        			cout << "\n\n[GPU] ~ ERROR IN KERNEL LAUNCH. ERROR: " << hipSuccess << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // find the size of the number of results
        		errCode = hipMemcpyAsync( &cnt[tid], &dev_cnt[tid], sizeof(unsigned int), hipMemcpyDeviceToHost, stream[tid] );
        		if (errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: getting cnt from GPU Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}
                #if !SILENT_GPU
        		else{
                    cout << "[GPU] ~ Result set size within epsilon: " << cnt[tid] << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}
                #endif

        		// double endKernel = omp_get_wtime();

                nbKernelInvocation[tid]++;

                hipEventSynchronize(stopKernel[tid]);
                float timeKernel = 0;
                hipEventElapsedTime(&timeKernel, startKernel[tid], stopKernel[tid]);
                kernelTimes[tid] += timeKernel;

                thrust::device_ptr<int> dev_keys_ptr(dev_pointIDKey[tid]);
        		thrust::device_ptr<int> dev_data_ptr(dev_pointInDistValue[tid]);

                try{
        			thrust::sort_by_key(thrust::cuda::par.on(stream[tid]), dev_keys_ptr, dev_keys_ptr + cnt[tid], dev_data_ptr);
        		}
        		catch(std::bad_alloc &e)
        		{
                    #if GPU_LOCKING
                    pthread_mutex_unlock(&gpu_lock);
                    #endif
        			std::cerr << "[GPU] ~ Ran out of memory while sorting, " << GPUBufferSize << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        			exit(1);
        		}
                #if GPU_LOCKING
                pthread_mutex_unlock(&gpu_lock);
                #endif

                hipMemcpyAsync(thrust::raw_pointer_cast(pointIDKey[tid]), thrust::raw_pointer_cast(dev_keys_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);
            		hipMemcpyAsync(thrust::raw_pointer_cast(pointInDistValue[tid]), thrust::raw_pointer_cast(dev_data_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);

                // hipStreamSynchronize(stream[tid]);

                double tableconstuctstart = omp_get_wtime();
        		//set the number of neighbors in the pointer struct:
        		(*pointersToNeighbors)[localBatchCounter].sizeOfDataArr = cnt[tid];
        		(*pointersToNeighbors)[localBatchCounter].dataPtr = new int[cnt[tid]];

                constructNeighborTableKeyValueWithPtrs(pointIDKey[tid], pointInDistValue[tid], neighborTable, (*pointersToNeighbors)[localBatchCounter].dataPtr, &cnt[tid]);

                double tableconstuctend = omp_get_wtime();

                #if !SILENT_GPU
                    cout << "[GPU] ~ Table construct time: " << tableconstuctend - tableconstuctstart << '\n';
                    cout.flush();
                #endif

                // add the batched result set size to the total count
        		totalResultsLoop += cnt[tid];

                #if !SILENT_GPU
                    cout << "[GPU] ~ Running total of total size of result array, tid: " << tid << ", " << totalResultsLoop << '\n';
                    cout.flush();
                #endif

                // gpuBatch = getBatchFromQueue(*DBSIZE, batchSize);
                gpuBatch = getBatchFromQueue_v2(batchesVector);
                // gpuBatch = getBatchFromQueue(9 * batchSize, batchSize);

                #pragma omp critical
                {
                    localBatchCounter = globalBatchCounter;
                    globalBatchCounter++;
                }

            } while(0 != gpuBatch.second);

        } // parallel section
        double tEndCompute = omp_get_wtime();
        computeTime = tEndCompute - tStartCompute;
    }
    else
    { // searchModes that have a fixed number of queries (e.g., original GPU kernel or static partitioning)
        // errCode = hipMemcpy( &dev_batchBegin[0], batchBegin, sizeof(unsigned int), hipMemcpyHostToDevice );
        // if (errCode != hipSuccess)
        // {
        //     cout << "[GPU] ~ Error: queue index copy to device -- error with code " << errCode << '\n';
        //     cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        //     cout.flush();
        // }

        double computeTimeStart = omp_get_wtime();

        double * computeTimeArray = new double[GPUSTREAMS];
        for(int i = 0; i < GPUSTREAMS; ++i)
        {
            computeTimeArray[i] = 0;
        }

        //FOR LOOP OVER THE NUMBER OF BATCHES STARTS HERE
    	//i=0...numBatches
        #pragma omp parallel for schedule(dynamic, 1) reduction(+: totalResultsLoop) num_threads(GPUSTREAMS)
    	for (int i = 0; i < 1; i++)
        // for (int i = 0; i < 9; ++i)
    	{
            int tid = omp_get_thread_num();

            double tStartLoop = omp_get_wtime();

            #if !SILENT_GPU
                cout << "[GPU] ~ tid " << tid << ", starting iteration " << i << " with block: " << (i * NBLOCKS) <<'\n';
                cout.flush();
            #endif

    		//N NOW BECOMES THE NUMBER OF POINTS TO PROCESS PER BATCH
    		//AS ONE GPU THREAD PROCESSES A SINGLE POINT
            // int blockResultSize = NBLOCKS;
            batchBegin[tid] = (i * NBLOCKS);
            errCode = hipMemcpy( &dev_batchBegin[tid], &batchBegin[tid], sizeof(unsigned int), hipMemcpyHostToDevice );
            if (errCode != hipSuccess)
            {
                cout << "[GPU] ~ Error: queue index copy to device -- error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
            }

            N[tid] = NBLOCKS;
            if (((i + 1) * NBLOCKS) > datasetSize) {
                N[tid] = datasetSize - (i * NBLOCKS);
            }
            #if !SILENT_GPU
                cout << "[GPU] ~ N (1 less): " << N[tid] << ", tid " << tid << '\n';
                cout.flush();
            #endif

            nbQueryPoint[tid] += N[tid];

    		//set relevant parameters for the batched execution that get reset

    		//copy N to device
    		//N IS THE NUMBER OF THREADS
    		errCode = hipMemcpyAsync( &dev_N[tid], &N[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if (errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: N Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		//the batched result set size (reset to 0):
    		cnt[tid] = 0;
    		errCode = hipMemcpyAsync( &dev_cnt[tid], &cnt[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if (errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: dev_cnt memcpy Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		// const int TOTALBLOCKS = ceil( (1.0 * (N[tid])) / (1.0 * BLOCKSIZE) );
            #if !SILENT_GPU
                cout << "[GPU] ~ Total blocks: " << NBLOCKS << '\n';
                cout.flush();
            #endif

            #if GPU_LOCKING
            pthread_mutex_lock(&gpu_lock);
            #endif

    		//execute kernel
    		//0 is shared memory pool
            hipEventRecord(startKernel[tid], stream[tid]);
            #if SORT_BY_WORKLOAD
                kernelNDGridIndexGlobal<<< NBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid], 
                    dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                    dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells,
                    dev_pointIDKey[tid], dev_pointInDistValue[tid], tpp);
            #else
                kernelNDGridIndexGlobal<<< NBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                    dev_database, nullptr, nullptr, dev_epsilon, dev_grid,
                    dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells,
                    dev_pointIDKey[tid], dev_pointInDistValue[tid], tpp);
            #endif
            hipEventRecord(stopKernel[tid], stream[tid]);


            errCode = hipGetLastError();
            #if !SILENT_GPU
        		cout << "\n\n[GPU] ~ KERNEL LAUNCH RETURN: " << errCode << '\n';
                cout.flush();
            #endif
    		if ( hipSuccess != hipGetLastError() )
            {
    			cout << "\n\n[GPU] ~ ERROR IN KERNEL LAUNCH. ERROR: " << hipSuccess << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		// find the size of the number of results
    		errCode = hipMemcpyAsync( &cnt[tid], &dev_cnt[tid], sizeof(unsigned int), hipMemcpyDeviceToHost, stream[tid] );
    		if (errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: getting cnt from GPU Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}
            #if !SILENT_GPU
    		else {
                cout << "[GPU] ~ Result set size within epsilon: " << cnt[tid] << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}
            #endif

    		// double endKernel = omp_get_wtime();
            // kernelTime[tid] += endKernel - beginKernel;
    		//cout << "Single kernel execution time = " << endKernel - beginKernel << " ms" << endl;

            nbKernelInvocation[tid]++;

            hipEventSynchronize(stopKernel[tid]);
            float timeKernel = 0;
            hipEventElapsedTime(&timeKernel, startKernel[tid], stopKernel[tid]);
            kernelTimes[tid] += timeKernel;


    		////////////////////////////////////
    		//SORT THE TABLE DATA ON THE GPU
    		//THERE IS NO ORDERING BETWEEN EACH POINT AND THE ONES THAT IT'S WITHIN THE DISTANCE OF
    		////////////////////////////////////

    		/////////////////////////////
    		//ONE PROBLEM WITH NOT TRANSFERING THE RESULT OFF OF THE DEVICE IS THAT
    		//YOU CAN'T RESIZE THE RESULTS TO BE THE SIZE OF *CNT
    		//SO THEN YOU HAVE POTENTIALLY LOTS OF WASTED SPACE
    		/////////////////////////////

    		//sort by key with the data already on the device:
    		//wrap raw pointer with a device_ptr to use with Thrust functions
    		thrust::device_ptr<int> dev_keys_ptr(dev_pointIDKey[tid]);
    		thrust::device_ptr<int> dev_data_ptr(dev_pointInDistValue[tid]);

    		//XXXXXXXXXXXXXXXX
    		//THRUST USING STREAMS REQUIRES THRUST V1.8
    		//XXXXXXXXXXXXXXXX

    		try {
    			thrust::sort_by_key(thrust::cuda::par.on(stream[tid]), dev_keys_ptr, dev_keys_ptr + cnt[tid], dev_data_ptr);

    		} catch(std::bad_alloc &e) {
                #if GPU_LOCKING
                pthread_mutex_unlock(&gpu_lock);
                #endif
    			std::cerr << "[GPU] ~ Ran out of memory while sorting, " << GPUBufferSize << '\n';
                cout.flush();
    			exit(1);
    		}

            #if GPU_LOCKING
            pthread_mutex_unlock(&gpu_lock);
            #endif
            // cout << "[GPU] ~ Thrust sort by key\n";
            // cout.flush();
            // cout << "[GPU] ~ Copy size: " << cnt[tid] * sizeof(int) << '\n';
            // cout.flush();

    		// thrust with streams into individual buffers for each batch
    		hipMemcpyAsync(thrust::raw_pointer_cast(pointIDKey[tid]), thrust::raw_pointer_cast(dev_keys_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);
    		hipMemcpyAsync(thrust::raw_pointer_cast(pointInDistValue[tid]), thrust::raw_pointer_cast(dev_data_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);

            // cout << "[GPU] ~ Async memcpy of pointers\n";
            // cout.flush();

    		//need to make sure the data is copied before constructing portion of the neighbor table
    		// hipStreamSynchronize(stream[tid]);

            // cout << "[GPU] ~ Stream synchronization\n";
            // cout.flush();

    		double tableconstuctstart = omp_get_wtime();
    		//set the number of neighbors in the pointer struct:
    		(*pointersToNeighbors)[i].sizeOfDataArr = cnt[tid];
    		(*pointersToNeighbors)[i].dataPtr = new int[cnt[tid]];

    		constructNeighborTableKeyValueWithPtrs(pointIDKey[tid], pointInDistValue[tid], neighborTable, (*pointersToNeighbors)[i].dataPtr, &cnt[tid]);
    		// cout <<"In make neighbortable. Data array ptr: "<<(*pointersToNeighbors)[i].dataPtr<<" , size of data array: "<<(*pointersToNeighbors)[i].sizeOfDataArr;cout.flush();

    		double tableconstuctend = omp_get_wtime();

            #if !SILENT_GPU
                cout << "[GPU] ~ Table construct time: " << tableconstuctend - tableconstuctstart << '\n';
                cout.flush();
            #endif

    		//add the batched result set size to the total count
            totalResultsLoop+=cnt[tid];

            #if !SILENT_GPU
                cout << "[GPU] ~ Running total of total size of result array, tid: " << tid << ", " << totalResultsLoop << '\n';
                cout.flush();
            #endif

            double tEndLoop = omp_get_wtime();
            // computeTimeArray[tid] += tEndLoop - tStartLoop;

    	} //END LOOP OVER THE GPU BATCHES

        double computeEndTime = omp_get_wtime();
        computeTime = computeEndTime - computeTimeStart;
        // cout << "[GPU | RESULT] ~ Compute time for the GPU = " << computeEndTime - computeTimeStart << '\n';
        // cout.flush();

        cout << "[BENCH] ~ Compute time for the GPU: " << computeTime << '\n';
        // for(int i = 0; i < GPUSTREAMS; ++i)
        // {
        //     cout << "   [BENCH | Stream " << i << "] ~ Compute time = " << computeTimeArray[i] << ", kernel time = " << kernelTimes[i] << '\n';
        // }

    }
    std::ofstream outfile ("new_mod_neighbor_table.csv",std::ofstream::binary);
    outfile << "pointIdx|originalPointIdx|neighborCnt|neighbors" << endl;
    int neighborCnt = 0;
    for (int i = 0; i < (*DBSIZE); i++) {
        neighborTableLookup tableRecord = neighborTable[originPointIndex[i]];
        neighborCnt = tableRecord.indexmax - tableRecord.indexmin;

        outfile << i << "|" << originPointIndex[i] << "|" << neighborCnt << "|";
        std::vector<unsigned int> neighbors = {};
        for (int j = tableRecord.indexmin; j < tableRecord.indexmax; j++) {
            neighbors.push_back(tableRecord.dataPtr[j]);
        }
        std::sort(neighbors.begin(), neighbors.end());
        for (int j = 0; j < neighborCnt; j++) {
            outfile << neighbors[j];
            if (j == neighborCnt - 1) {
                outfile << endl;
            }
            else {
                outfile << ",";
            }
        }
    }

    unsigned int nbQueryPointTotal = 0;
    for (int i = 0; i < GPUSTREAMS; ++i)
    {
        nbQueryPointTotal += nbQueryPoint[i];
    }

    (*nbQueriesGPU) = nbQueryPointTotal;

    for (int i = 0; i < GPUSTREAMS; ++i)
    {
        printf("[GPU] ~ Kernel execution time on stream %d: %f\n", i, kernelTimes[i]);
    }

    printf("[GPU | RESULT] ~ Query points computed by the GPU: %d (f: %f)\n", nbQueryPointTotal, (nbQueryPointTotal * 1.0) / ((*DBSIZE) * 1.0));
    printf("[GPU | RESULT] ~ Compute time for the GPU: %f\n", computeTime);

    cout << "[GPU] ~ Total result set size on host: " << totalResultsLoop << "\033[00m\n";
    cout.flush();

	*totalNeighbors = totalResultsLoop;

	double tKernelResultsEnd = omp_get_wtime();

    cout << "[GPU] ~ Time to launch kernel and execute everything except freeing memory: " << tKernelResultsEnd - tKernelResultsStart << '\n';
    cout.flush();

	///////////////////////////////////
	//END GET RESULT SET
	///////////////////////////////////





	///////////////////////////////////
	//FREE MEMORY FROM THE GPU
	///////////////////////////////////
	// if (NUM_TRIALS>1)
	// {

	double tFreeStart = omp_get_wtime();

	for (int i = 0; i < GPUSTREAMS; i++)
    {
		errCode = hipStreamDestroy(stream[i]);
		if (errCode != hipSuccess) {
			cout << "[GPU] ~ Error: destroying stream" << errCode << '\n';
            cout.flush();
		}
	}

	delete totalResultSetCnt;
	delete[] cnt;
	delete[] N;

	//free the data on the device

	hipFree(dev_N);
	hipFree(dev_cnt);
	// hipFree(dev_offset);
	// hipFree(dev_batchNumber);


	//free data related to the individual streams for each batch
	for (int i = 0; i < GPUSTREAMS; i++)
    {
		//free the data on the device
		hipFree(dev_pointIDKey[i]);
		hipFree(dev_pointInDistValue[i]);

		//free on the host
		hipHostFree(pointIDKey[i]);
		hipHostFree(pointInDistValue[i]);
	}

    // hipFree(dev_pointIDKey);
    // hipFree(dev_pointInDistValue);

	//free pinned memory on host
	// hipHostFree(pointIDKey); 
	// hipHostFree(pointInDistValue);

  	double tFreeEnd = omp_get_wtime();

    cout << "[GPU] ~ Time freeing memory: " << tFreeEnd - tFreeStart << '\n';
    cout.flush();
  	// printf("\nTime freeing memory: %f", tFreeEnd - tFreeStart);
  	// }
  	cout << "\n[GPU] ~ ** last error at end of fn batches (could be from freeing memory): " << hipGetLastError() << "\n\n";
    cout.flush();

} // NDGridIndexGlobal





void warmUpGPU() {
    // initialize all ten integers of a device_vector to 1
  	thrust::device_vector<int> D(10, 1);
  	// set the first seven elements of a vector to 9
  	thrust::fill(D.begin(), D.begin() + 7, 9);
  	// initialize a host_vector with the first five elements of D
  	thrust::host_vector<int> H(D.begin(), D.begin() + 5);
  	// set the elements of H to 0, 1, 2, 3, ...
  	thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D
  	thrust::copy(H.begin(), H.end(), D.begin());
  	// print D
  	for (int i = 0; i < D.size(); i++)
    {
        cout << " D[" << i << "] = " << D[i];
    }

	return;
}





void constructNeighborTableKeyValueWithPtrs(
    int * pointIDKey,
    int * pointInDistValue,
    struct neighborTableLookup * neighborTable,
    int * pointersToNeighbors,
    unsigned int * cnt)
{
    //copy the value data:
  	std::copy(pointInDistValue, pointInDistValue + (*cnt), pointersToNeighbors);

	//Step 1: find all of the unique keys and their positions in the key array
	unsigned int numUniqueKeys = 0;

	std::vector<keyData> uniqueKeyData;

	keyData tmp;
	tmp.key = pointIDKey[0];
	tmp.position = 0;
	uniqueKeyData.push_back(tmp);

	//we assign the ith data item when iterating over i+1th data item,
	//so we go 1 loop iteration beyond the number (*cnt)
	for (int i = 1; i < (*cnt) + 1; i++)
    {
		if (pointIDKey[i - 1] != pointIDKey[i]){
			numUniqueKeys++;
			tmp.key = pointIDKey[i];
			tmp.position = i;
			uniqueKeyData.push_back(tmp);
		}
	}

	//insert into the neighbor table the values based on the positions of
	//the unique keys obtained above.
	for (int i = 0; i < uniqueKeyData.size() - 1; i++)
    {
		int keyElem = uniqueKeyData[i].key;
		neighborTable[keyElem].pointID = keyElem;
		neighborTable[keyElem].indexmin = uniqueKeyData[i].position;
		neighborTable[keyElem].indexmax = uniqueKeyData[i + 1].position;

		//update the pointer to the data array for the values
		neighborTable[keyElem].dataPtr = pointersToNeighbors;
	}
}
