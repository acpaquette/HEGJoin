#include "hip/hip_runtime.h"
//precompute direct neighbors with the GPU:
#include "GPU.h"
#include "kernel.h"
#include "SortByWorkload.h"
#include "structs.h"
#include "params.h"
#include "WorkQueue.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <math.h>
#include <algorithm>
#include <unistd.h>
#include "omp.h"

// //thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/system/cuda/execution_policy.h> // for streams for thrust (added with Thrust v1.8)
//
// //for warming up GPU:
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

//elements for the result set
//FOR A SINGLE KERNEL INVOCATION
//NOT FOR THE BATCHED ONE
#define BUFFERELEM 300000000 //400000000-original (when removing the data from the device before putting it back for the sort)

//FOR THE BATCHED EXECUTION:
//#define BATCHTOTALELEM 1200000000 //THE TOTAL SIZE ALLOCATED ON THE HOST
//THE NUMBER OF BATCHES AND THE SIZE OF THE BUFFER FOR EACH KERNEL EXECUTION ARE NOT RELATED TO THE TOTAL NUMBER
//OF ELEMENTS (ABOVE).
// #define NUMBATCHES 20
// #define BATCHBUFFERELEM 100000000 //THE SMALLER SIZE ALLOCATED ON THE DEVICE FOR EACH KERNEL EXECUTION

// #define GPUSTREAMS 1 //number of concurrent gpu streams, now defined in params.h

using std::cout;
using std::endl;

//sort ascending
bool compareByPointValue(const key_val_sort &a, const key_val_sort &b)
{
    return a.value_at_dim < b.value_at_dim;
}

uint64_t getLinearID_nDimensions2(unsigned int * indexes, unsigned int * dimLen, unsigned int nDimensions) {
    uint64_t index = 0;
	uint64_t multiplier = 1;
	for (int i = 0; i<nDimensions; i++){
  	     index += (uint64_t)indexes[i] * multiplier;
  	      multiplier *= dimLen[i];
	}

	return index;
}





////////////////////////////////////////////////////////////////////////////////





void gridIndexingGPU(
    unsigned int * DBSIZE,
    uint64_t totalCells,
    DTYPE * database,
    DTYPE ** dev_database,
    DTYPE * epsilon,
    DTYPE ** dev_epsilon,
    DTYPE * minArr,
    DTYPE ** dev_minArr,
    struct grid ** index,
    struct grid ** dev_index,
    unsigned int * indexLookupArr,
    unsigned int ** dev_indexLookupArr,
    struct gridCellLookup ** gridCellLookupArr,
    struct gridCellLookup ** dev_gridCellLookupArr,
    unsigned int * nNonEmptyCells,
    unsigned int ** dev_nNonEmptyCells,
    unsigned int * nCells,
    unsigned int ** dev_nCells)
{

    hipError_t errCode;

    double tStartAllocGPU = omp_get_wtime();

    errCode = hipMalloc( (void**)dev_database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc database -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_epsilon, sizeof(DTYPE));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc epsilon -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_minArr, sizeof(DTYPE) * (NUMINDEXEDDIM));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc minArr -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_index, sizeof(struct grid) * (*nNonEmptyCells));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc grid index -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_indexLookupArr, sizeof(unsigned int) * (*DBSIZE));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: lookup array allocation -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc( (void**)dev_gridCellLookupArr, sizeof(struct gridCellLookup) * (*nNonEmptyCells));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: copy grid cell lookup array allocation -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_nNonEmptyCells, sizeof(unsigned int));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc nNonEmptyCells -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMalloc((void**)dev_nCells, sizeof(unsigned int) * (NUMINDEXEDDIM));
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc nCells -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    uint64_t * dev_pointCellArr;
    errCode = hipMalloc((void**)&dev_pointCellArr, sizeof(uint64_t) * (*DBSIZE));
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: point cell array alloc -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    unsigned int * dev_databaseVal;
	errCode = hipMalloc((void**)&dev_databaseVal, sizeof(unsigned int) * (*DBSIZE));
	if(errCode != hipSuccess) {
    	cout << "[INDEX] ~ Error: Alloc databaseVal -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    unsigned int * N = new unsigned int;
	unsigned int * dev_N;
	errCode = hipMalloc((void**)&dev_N, sizeof(unsigned int) * GPUSTREAMS);
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Alloc dev_N -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    double tEndAllocGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to allocate on the GPU: " << tEndAllocGPU - tStartAllocGPU << "\n\n";
    cout.flush();



    ////////////////////////////////////////////////////////////////////////////



    double tStartCopyGPU = omp_get_wtime();

    errCode = hipMemcpy( (*dev_database), database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: database copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_epsilon), epsilon, sizeof(DTYPE), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: epsilon copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_minArr), minArr, sizeof(DTYPE) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Copy minArr to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_index), index, sizeof(struct grid) * (*nNonEmptyCells), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: grid index copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_indexLookupArr), indexLookupArr, sizeof(unsigned int) * (*DBSIZE), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: copy lookup array to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_gridCellLookupArr), gridCellLookupArr, sizeof(struct gridCellLookup) * (*nNonEmptyCells), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: copy grid cell lookup array to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_nNonEmptyCells), nNonEmptyCells, sizeof(unsigned int), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: nNonEmptyCells copy to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy( (*dev_nCells), nCells, sizeof(unsigned int) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	if(errCode != hipSuccess)
    {
		cout << "[INDEX] ~ Error: Copy nCells to device -- error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(dev_N, DBSIZE, sizeof(unsigned int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: database size Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    double tEndCopyGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to copy to the GPU: " << tEndCopyGPU - tStartCopyGPU << "\n\n";
    cout.flush();



    ////////////////////////////////////////////////////////////////////////////



    const int TOTALBLOCKS = ceil((1.0 * (*DBSIZE)) / (1.0 * BLOCKSIZE));
	printf("[INDEX] ~ Total blocks: %d\n",TOTALBLOCKS);

	kernelIndexComputeNonemptyCells<<<TOTALBLOCKS, BLOCKSIZE>>>((*dev_database), dev_N, (*dev_epsilon), (*dev_minArr),
            (*dev_nCells), dev_pointCellArr, nullptr, false);
    hipDeviceSynchronize();

    thrust::device_ptr<uint64_t> dev_pointCellArr_ptr(dev_pointCellArr);
	thrust::device_ptr<uint64_t> dev_new_end;

	try
    {
		//first sort
		thrust::sort(thrust::device, dev_pointCellArr_ptr, dev_pointCellArr_ptr + (*DBSIZE)); //, thrust::greater<uint64_t>()
		//then unique
		dev_new_end = thrust::unique(thrust::device, dev_pointCellArr_ptr, dev_pointCellArr_ptr + (*DBSIZE));
	}
	catch(std::bad_alloc &e)
	{
	 	std::cerr << "[INDEX] ~ Ran out of memory while sorting" << std::endl;
	    exit(-1);
    }

    uint64_t * new_end = thrust::raw_pointer_cast(dev_new_end);
    uint64_t numNonEmptyCells = std::distance(dev_pointCellArr_ptr, dev_new_end);
    printf("[INDEX] ~ Number of full cells (non-empty): %lu\n", numNonEmptyCells);
    *nNonEmptyCells = numNonEmptyCells;
    (*gridCellLookupArr) = new struct gridCellLookup[numNonEmptyCells];
    uint64_t * pointCellArrTmp = new uint64_t[numNonEmptyCells];
    errCode = hipMemcpy(pointCellArrTmp, dev_pointCellArr, sizeof(uint64_t) * numNonEmptyCells, hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: pointCellArrTmp memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	for (uint64_t i = 0; i < numNonEmptyCells; ++i)
	{
		(*gridCellLookupArr)[i].idx = i;
		(*gridCellLookupArr)[i].gridLinearID = pointCellArrTmp[i];
	}

    kernelIndexComputeNonemptyCells<<<TOTALBLOCKS, BLOCKSIZE>>>((*dev_database), dev_N, (*dev_epsilon), (*dev_minArr),
            (*dev_nCells), dev_pointCellArr, dev_databaseVal, true);

    try
	{
    	thrust::sort_by_key(thrust::device, dev_pointCellArr, dev_pointCellArr + (*DBSIZE), dev_databaseVal);
	}
	catch(std::bad_alloc &e)
	{
		std::cerr << "[INDEX] ~ Ran out of memory while sorting key/value pairs" << std::endl;
	    exit(-1);
	}

    uint64_t * cellKey = new uint64_t[(*DBSIZE)];
    errCode = hipMemcpy(cellKey, dev_pointCellArr, sizeof(uint64_t) * (*DBSIZE), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: pointCellArr memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy(indexLookupArr, dev_databaseVal, sizeof(unsigned int) * (*DBSIZE), hipMemcpyDeviceToHost);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: databaseIDValue memcpy Got error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    (*index) = new grid[numNonEmptyCells];
    (*index)[0].indexmin = 0;
	uint64_t cnt=0;
	for (uint64_t i = 1; i < (*DBSIZE); ++i)
    {
		if (cellKey[i - 1] != cellKey[i])
		{
			//grid index
			cnt++;
			(*index)[cnt].indexmin = i;
			(*index)[cnt - 1].indexmax = i - 1;
		}
	}
	(*index)[numNonEmptyCells - 1].indexmax = (*DBSIZE) - 1;

    printf("[INDEX] ~ Full cells: %d (%f, fraction full)\n", (unsigned int)numNonEmptyCells, numNonEmptyCells * 1.0 / double(totalCells));
	printf("[INDEX] ~ Empty cells: %ld (%f, fraction empty)\n", totalCells - (unsigned int)numNonEmptyCells, (totalCells - numNonEmptyCells * 1.0) / double(totalCells));
	printf("[INDEX] ~ Size of index that would be sent to GPU (GiB) -- (if full index sent), excluding the data lookup arr: %f\n",
        (double)sizeof(struct grid) * (totalCells) / (1024.0 * 1024.0 * 1024.0));
	printf("[INDEX] ~ Size of compressed index to be sent to GPU (GiB), excluding the data and grid lookup arr: %f\n",
        (double)sizeof(struct grid) * (numNonEmptyCells * 1.0) / (1024.0 * 1024.0 * 1024.0));
	printf("[INDEX] ~ When copying from entire index to compressed index: number of non-empty cells: %lu\n", numNonEmptyCells);

    errCode = hipMemcpy((*dev_index), (*index), sizeof(struct grid) * (*nNonEmptyCells), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: index copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy((*dev_indexLookupArr), dev_databaseVal, sizeof(unsigned int) * (*DBSIZE), hipMemcpyDeviceToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: index lookup array copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    errCode = hipMemcpy((*dev_gridCellLookupArr), (*gridCellLookupArr), sizeof(struct gridCellLookup) * (*nNonEmptyCells), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[INDEX] ~ Error: grid lookup array copy to the GPU error with code " << errCode << '\n';
        cout << "[INDEX] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

    delete N;
    delete[] pointCellArrTmp;
    hipFree(dev_pointCellArr);
    hipFree(dev_databaseVal);
    hipFree(dev_N);

    double tEndIndexGPU = omp_get_wtime();
    cout << "[INDEX] ~ Time to index using the GPU (including allocating and transfering memory): " << tEndIndexGPU - tStartAllocGPU << '\n';
    cout.flush();

}





unsigned long long callGPUBatchEst(
    unsigned int * DBSIZE,
    unsigned int sampleBegin,
    unsigned int sampleEnd,
    DTYPE * dev_database,
    DTYPE * dev_sortedDatabase,
    unsigned int * dev_originPointIndex,
    DTYPE * dev_epsilon,
    struct grid * dev_grid,
	unsigned int * dev_indexLookupArr,
    struct gridCellLookup * dev_gridCellLookupArr,
    DTYPE * dev_minArr,
	unsigned int * dev_nCells,
    unsigned int * dev_nNonEmptyCells,
    // unsigned int * dev_gridCellNDMask,
	// unsigned int * dev_gridCellNDMaskOffsets,
    unsigned int * retNumBatches,
    unsigned int * retGPUBufferSize)
{
	//CUDA error code:
	hipError_t errCode;

    cout << "[GPU] ~ ***********************************\n[GPU] ~ Estimating Batches:\n";
	cout << "[GPU] ~ BATCH ESTIMATOR: Sometimes the GPU will error on a previous execution and you won't know. \n[GPU] ~ Last error start of function: " << hipGetLastError() << '\n';
    cout.flush();

    unsigned int sampleSize = sampleEnd - sampleBegin;

    cout << "[GPU] ~ Sampling the portion [" << sampleBegin << ", " << sampleEnd << "] of the dataset (" << sampleSize << " elements)\n";
    cout.flush();

    //////////////////////////////////////////////////////////
	//ESTIMATE THE BUFFER SIZE AND NUMBER OF BATCHES ETC BY COUNTING THE NUMBER OF RESULTS
	//TAKE A SAMPLE OF THE DATA POINTS, NOT ALL OF THEM
	//Use sampleRate for this
	/////////////////////////////////////////////////////////

	//Parameters for the batch size estimation.
	// double sampleRate = 0.01; //sample 1% of the points in the dataset sampleRate=0.01.
						//Sample the entire dataset(no sampling) sampleRate=1
    double sampleRate = 0.05; // sample 5% of the dataset
	int offsetRate = 1.0 / sampleRate;
    cout << "[GPU] ~ Offset: " << offsetRate << '\n';
    cout.flush();

	/////////////////
	//N GPU threads
	////////////////

	unsigned int * dev_N_batchEst;
	unsigned int * N_batchEst = (unsigned int*)malloc(sizeof(unsigned int));
    // For 'basic' versions, sampleEnd = DBSIZE and sampleBegin = 0
    // For static partitioning, sampleEnd = DBSIZE * fraction and sampleBegin = 0
    // For dynamic partitioning, sampleEnd and sampleBegin correspond to the batch currently taken by the GPU
    *N_batchEst = (sampleEnd - sampleBegin) * sampleRate;
	// *N_batchEst = *DBSIZE * sampleRate;

	//allocate on the device
	errCode = hipMalloc((void**)&dev_N_batchEst, sizeof(unsigned int));
	if(errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_N_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	//copy N to device
	//N IS THE NUMBER OF THREADS
	errCode = hipMemcpy(dev_N_batchEst, N_batchEst, sizeof(unsigned int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
	    cout << "[GPU] ~ Error: N batchEST Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	/////////////
	//count the result set size
	////////////

	unsigned int * dev_cnt_batchEst;

	unsigned int * cnt_batchEst;
	cnt_batchEst = (unsigned int*)malloc(sizeof(unsigned int));
	*cnt_batchEst = 0;

	//allocate on the device
	errCode = hipMalloc((void**)&dev_cnt_batchEst, sizeof(unsigned int));
	if(errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_cnt_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	//copy cnt to device
	errCode = hipMemcpy(dev_cnt_batchEst, cnt_batchEst, sizeof(unsigned int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_cnt_batchEst Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	//////////////////
	//SAMPLE OFFSET - TO SAMPLE THE DATA TO ESTIMATE THE TOTAL NUMBER OF KEY VALUE PAIRS
	/////////////////

	//offset into the database when batching the results
	unsigned int * sampleOffset;
	sampleOffset = (unsigned int*)malloc(sizeof(unsigned int));
	*sampleOffset = offsetRate;

	unsigned int * dev_sampleOffset;

	//allocate on the device
	errCode = hipMalloc((void**)&dev_sampleOffset, sizeof(unsigned int));
	if(errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: sample offset Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	//copy offset to device
	errCode = hipMemcpy(dev_sampleOffset, sampleOffset, sizeof(unsigned int), hipMemcpyHostToDevice);
	if(errCode != hipSuccess)
    {
    	cout << "[GPU] ~ Error: dev_sampleOffset Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}


	// const int TOTALBLOCKSBATCHEST = ceil((1.0 * (*DBSIZE) * sampleRate) / (1.0 * BLOCKSIZE));
    const int TOTALBLOCKSBATCHEST = ceil((1.0 * sampleSize * sampleRate) / (1.0 * BLOCKSIZE));
    cout << "[GPU] ~ Total blocks: " << TOTALBLOCKSBATCHEST << '\n';
    cout.flush();

    #if UNICOMP
        cout << "[GPU] ~ Estimating batch using the Unicomp pattern\n";
        cout.flush();
        kernelNDGridIndexBatchEstimatorUnicompAdaptive<<<TOTALBLOCKSBATCHEST, BLOCKSIZE>>>(sampleBegin, sampleEnd, dev_N_batchEst, dev_sampleOffset,
            dev_database, dev_sortedDatabase, dev_epsilon, dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells,
            dev_cnt_batchEst, dev_nNonEmptyCells/*, dev_gridCellNDMask, dev_gridCellNDMaskOffsets*/);
    #elif LID_UNICOMP
        cout << "[GPU] ~ Estimating batch using the Lid-Unicomp pattern\n";
        cout.flush();
        kernelNDGridIndexBatchEstimatorLidUnicompAdaptive<<<TOTALBLOCKSBATCHEST, BLOCKSIZE>>>(sampleBegin, sampleEnd, dev_N_batchEst, dev_sampleOffset,
            dev_database, dev_sortedDatabase, dev_epsilon, dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells,
            dev_cnt_batchEst, dev_nNonEmptyCells/*, dev_gridCellNDMask, dev_gridCellNDMaskOffsets*/);
    #else
        cout << "[GPU] ~ Estimating batch without using pattern\n";
        cout.flush();
        kernelNDGridIndexBatchEstimatorAdaptive<<<TOTALBLOCKSBATCHEST, BLOCKSIZE>>>(sampleBegin, sampleEnd, dev_N_batchEst, dev_sampleOffset,
            dev_database, dev_sortedDatabase, dev_originPointIndex, dev_epsilon, dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr,
            dev_nCells, dev_cnt_batchEst, dev_nNonEmptyCells/*, dev_gridCellNDMask, dev_gridCellNDMaskOffsets*/);
    #endif

	cout << "[GPU] ~ ERROR FROM KERNEL LAUNCH OF BATCH ESTIMATOR: " << hipGetLastError() << '\n';
    cout.flush();
	// find the size of the number of results

	errCode = hipMemcpy(cnt_batchEst, dev_cnt_batchEst, sizeof(unsigned int), hipMemcpyDeviceToHost);

	if(errCode != hipSuccess)
    {
	    cout << "[GPU] ~ Error: getting cnt for batch estimate from GPU Got error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}else{
        cout << "[GPU] ~ Result set size for estimating the number of batches (sampled): " << *cnt_batchEst << '\n';
        cout.flush();
	}

	uint64_t estimatedNeighbors = (uint64_t)*cnt_batchEst * (uint64_t)offsetRate;
    cout << "[GPU] ~ From GPU cnt: " << *cnt_batchEst <<", offset rate: " << offsetRate << '\n';
    cout.flush();
	//initial

    // unsigned int GPUBufferSize;
    // if(initialEst)
    // {
    	// GPUBufferSize = 40000000; //size in HPBDC paper (low-D)
    // }else{
    //     GPUBufferSize = 1000000;
    // }
	unsigned int GPUBufferSize = 50000000;
    // unsigned int GPUBufferSize = 100000000;

    // #if SORT_BY_WORKLOAD
    //     #if LID_UNICOMP
    //     	double alpha = 0.05; //overestimation factor
    //     #else
    //         double alpha = 0.5; // higher overestimation to compensate the fact that a cell access pattern adds 2 points for each neighbor within epsilon
    //     #endif
    // #else
        // double alpha = 0.05; //overestimation factor
    // #endif

    double alpha = 0.05;

	uint64_t estimatedTotalSizeWithAlpha = estimatedNeighbors * (1.0 + alpha * 1.0);
    cout << "[GPU] ~ Estimated total result set size: " << estimatedNeighbors << '\n';
    cout << "[GPU] ~ Estimated total restult set size (with Alpha " << alpha << "): " << estimatedTotalSizeWithAlpha << '\n';
    cout.flush();

	if (estimatedNeighbors < (GPUBufferSize * GPUSTREAMS))
	{
        cout << "[GPU] ~ Small buffer size, increasing alpha to: " << alpha * 3.0 << '\n';
        cout.flush();
		GPUBufferSize = estimatedNeighbors * (1.0 + (alpha * 2.0)) / (GPUSTREAMS);		//we do 2*alpha for small datasets because the
																		//sampling will be worse for small datasets
																		//but we fix the 3 streams still (thats why divide by 3).
	}

	unsigned int numBatches = ceil(((1.0 + alpha) * estimatedNeighbors * 1.0) / ((uint64_t)GPUBufferSize * 1.0));
    cout << "[GPU] ~ Number of batches: " << numBatches << ", buffer size: " << GPUBufferSize << '\n';
    cout.flush();

	*retNumBatches = numBatches + 1;
	// *retGPUBufferSize = 1.5 * GPUBufferSize;
    *retGPUBufferSize = GPUBufferSize;

    cout << "[GPU] ~ End Batch Estimator\n***********************************\n";
    cout.flush();

	hipFree(dev_cnt_batchEst);
	hipFree(dev_N_batchEst);
	hipFree(dev_sampleOffset);

    return estimatedTotalSizeWithAlpha;

}







//modified from: makeDistanceTableGPUGridIndexBatchesAlternateTest
void distanceTableNDGridBatches(
        int searchMode,
        unsigned int * DBSIZE,
        DTYPE * epsilon,
        DTYPE * dev_epsilon,
        DTYPE * database,
        DTYPE * dev_database,
        struct grid * grid,
        struct grid * dev_grid,
        unsigned int * indexLookupArr,
        unsigned int * dev_indexLookupArr,
        struct gridCellLookup * gridCellLookupArr,
        struct gridCellLookup * dev_gridCellLookupArr,
        DTYPE * minArr,
        DTYPE * dev_minArr,
        unsigned int * nCells,
        unsigned int * dev_nCells,
        unsigned int * nNonEmptyCells,
        unsigned int * dev_nNonEmptyCells,
        // unsigned int * gridCellNDMask,
        // unsigned int * dev_gridCellNDMask,
        // unsigned int * gridCellNDMaskOffsets,
        // unsigned int * dev_gridCellNDMaskOffsets,
        // unsigned int * nNDMaskElems,
        unsigned int * originPointIndex,
        unsigned int * dev_originPointIndex,
        struct neighborTableLookup * neighborTable,
        std::vector<struct neighborDataPtrs> * pointersToNeighbors,
        uint64_t * totalNeighbors)
{

	double tKernelResultsStart = omp_get_wtime();

	//CUDA error code:
	hipError_t errCode;

	cout << "\n[GPU] ~ Sometimes the GPU will error on a previous execution and you won't know. \n[GPU] ~ Last error start of function: " << hipGetLastError() << '\n';
    cout.flush();



	///////////////////////////////////
	//COUNT VALUES -- RESULT SET SIZE FOR EACH KERNEL INVOCATION
	///////////////////////////////////

	//total size of the result set as it's batched
	//this isnt sent to the GPU
	unsigned int * totalResultSetCnt;
	totalResultSetCnt = (unsigned int*)malloc(sizeof(unsigned int));
	*totalResultSetCnt = 0;

	//count values - for an individual kernel launch
	//need different count values for each stream
	unsigned int * cnt;
	cnt = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);
	*cnt = 0;

	unsigned int * dev_cnt;
	dev_cnt = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);
	*dev_cnt = 0;

	//allocate on the device
	errCode = hipMalloc((void**)&dev_cnt, sizeof(unsigned int) * GPUSTREAMS);
	if(errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc cnt -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	///////////////////////////////////
	//END COUNT VALUES -- RESULT SET SIZE FOR EACH KERNEL INVOCATION
	///////////////////////////////////



	////////////////////////////////////
	//NUMBER OF THREADS PER GPU STREAM
	////////////////////////////////////

	//THE NUMBER OF THREADS THAT ARE LAUNCHED IN A SINGLE KERNEL INVOCATION
	//CAN BE FEWER THAN THE NUMBER OF ELEMENTS IN THE DATABASE IF MORE THAN 1 BATCH
	unsigned int * N;
	N = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	unsigned int * dev_N;
	// dev_N = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	//allocate on the device
	errCode = hipMalloc((void**)&dev_N, sizeof(unsigned int) * GPUSTREAMS);
	if(errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc dev_N -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	////////////////////////////////////
	//NUMBER OF THREADS PER GPU STREAM
	////////////////////////////////////



	////////////////////////////////////
	//OFFSET INTO THE DATABASE FOR BATCHING THE RESULTS
	//BATCH NUMBER
	////////////////////////////////////
	unsigned int * batchOffset;
	batchOffset = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	unsigned int * dev_offset;
	// dev_offset = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	//allocate on the device
	errCode = hipMalloc((void**)&dev_offset, sizeof(unsigned int) * GPUSTREAMS);
	if(errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc offset -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	//Batch number to calculate the point to process (in conjunction with the offset)
	//offset into the database when batching the results
	unsigned int * batchNumber;
	batchNumber = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	unsigned int * dev_batchNumber;
	// dev_batchNumber = (unsigned int*)malloc(sizeof(unsigned int) * GPUSTREAMS);

	//allocate on the device
	errCode = hipMalloc((void**)&dev_batchNumber, sizeof(unsigned int) * GPUSTREAMS);
	if(errCode != hipSuccess)
    {
		cout << "[GPU] ~ Error: Alloc batch number -- error with code " << errCode << '\n';
        cout << "  Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
	}

	////////////////////////////////////
	//END OFFSET INTO THE DATABASE FOR BATCHING THE RESULTS
	//BATCH NUMBER
	////////////////////////////////////



    /////////////////////////////////////////////////////////
	//BEGIN BATCH ESTIMATOR
	/////////////////////////////////////////////////////////

	unsigned long long estimatedNeighbors = 0;
	unsigned int numBatches = 0;
	unsigned int GPUBufferSize = 0;

	double tstartbatchest = omp_get_wtime();
    estimatedNeighbors = callGPUBatchEst(DBSIZE, 0, *DBSIZE, dev_database, nullptr, dev_originPointIndex, dev_epsilon,
            dev_grid, dev_indexLookupArr, dev_gridCellLookupArr, dev_minArr, dev_nCells, dev_nNonEmptyCells, /*dev_gridCellNDMask,
            dev_gridCellNDMaskOffsets,*/ &numBatches, &GPUBufferSize);
	double tendbatchest = omp_get_wtime();

    cout << "[GPU] ~ Time to estimate batches: " << tendbatchest - tstartbatchest << '\n';
    cout.flush();

    cout << "[GPU] ~ In calling function: Estimated neighbors = " << estimatedNeighbors
            << ", num. batches = " << numBatches << ", GPU buffer size = " << GPUBufferSize << '\n';
    cout.flush();

    // sets the batch size for the queue and the queue index, considering the offset reserved for the GPU
    setQueueIndex(GPUSTREAMS * (*DBSIZE / numBatches));
// setQueueIndex(0);

	/////////////////////////////////////////////////////////
	//END BATCH ESTIMATOR
	/////////////////////////////////////////////////////////



	///////////////////
	//ALLOCATE POINTERS TO INTEGER ARRAYS FOR THE VALUES FOR THE NEIGHBORTABLES
	///////////////////

	//THE NUMBER OF POINTERS IS EQUAL TO THE NUMBER OF BATCHES
	for (int i = 0; i < numBatches; i++)
    {
		int *ptr;
		struct neighborDataPtrs tmpStruct;
		tmpStruct.dataPtr = ptr;
		tmpStruct.sizeOfDataArr = 0;

		pointersToNeighbors->push_back(tmpStruct);
	}

	///////////////////
	//END ALLOCATE POINTERS TO INTEGER ARRAYS FOR THE VALUES FOR THE NEIGHBORTABLES
	///////////////////



	///////////////////////////////////
	//ALLOCATE MEMORY FOR THE RESULT SET USING THE BATCH ESTIMATOR
	///////////////////////////////////

	//NEED BUFFERS ON THE GPU AND THE HOST FOR THE NUMBER OF CONCURRENT STREAMS
	//GPU BUFFER ON THE DEVICE
	//BUFFER ON THE HOST WITH PINNED MEMORY FOR FAST MEMCPY
	//BUFFER ON THE HOST TO DUMP THE RESULTS OF BATCHES SO THAT GPU THREADS CAN CONTINUE
	//EXECUTING STREAMS ON THE HOST

	//GPU MEMORY ALLOCATION: key/value pairs

	int * dev_pointIDKey[GPUSTREAMS]; //key
	int * dev_pointInDistValue[GPUSTREAMS]; //value
    // GPUBufferSize = 100000000;
    // GPUBufferSize = 150000000;
	for (int i = 0; i < GPUSTREAMS; i++)
	{
		errCode = hipMalloc((void **)&dev_pointIDKey[i], 2 * sizeof(int) * GPUBufferSize);
		if(errCode != hipSuccess)
        {
			cout << "[GPU] ~ CUDA: Got error with code " << errCode << '\n'; //2 means not enough memory
            cout << "  Details: " << hipGetErrorString(errCode) << '\n';
            cout.flush();
		}

		errCode = hipMalloc((void **)&dev_pointInDistValue[i], 2 * sizeof(int) * GPUBufferSize);
		if(errCode != hipSuccess)
        {
			cout << "[GPU] ~ CUDA: Got error with code " << errCode << '\n'; //2 means not enough memory
            cout << "  Details: " << hipGetErrorString(errCode) << '\n';
            cout.flush();
		}

	}
    cout << "[GPU] ~ Allocation pointIDKey and pointInDistValue on the GPU, size = " << 2 * sizeof(int) * GPUBufferSize << '\n';
    cout.flush();

	//HOST RESULT ALLOCATION FOR THE GPU TO COPY THE DATA INTO A PINNED MEMORY ALLOCATION
	//ON THE HOST
	//pinned result set memory for the host
	//the number of elements are recorded for that batch in resultElemCountPerBatch
	//NEED PINNED MEMORY ALSO BECAUSE YOU NEED IT TO USE STREAMS IN THRUST FOR THE MEMCOPY OF THE SORTED RESULTS
	//can't do async copies without pinned memory

	//PINNED MEMORY TO COPY FROM THE GPU
	int * pointIDKey[GPUSTREAMS]; //key
	int * pointInDistValue[GPUSTREAMS]; //value

	double tstartpinnedresults = omp_get_wtime();

    #pragma omp parallel for num_threads(GPUSTREAMS)
	for (int i = 0; i < GPUSTREAMS; i++)
	{
		hipHostMalloc((void **) &pointIDKey[i], 2 * sizeof(int) * GPUBufferSize);
		hipHostMalloc((void **) &pointInDistValue[i], 2 * sizeof(int) * GPUBufferSize);
	}

	double tendpinnedresults = omp_get_wtime();

    cout << "[GPU] ~ Time to allocate pinned memory for results: " << tendpinnedresults - tstartpinnedresults << '\n';
    cout.flush();

	// hipMalloc((void **) &pointIDKey, sizeof(int)*GPUBufferSize*NUMBATCHES);
	// hipMalloc((void **) &pointInDistValue, sizeof(int)*GPUBufferSize*NUMBATCHES);

    cout << "[GPU] ~ Memory request for results on GPU (GiB): " << (double)(sizeof(int) * 2 * GPUBufferSize * GPUSTREAMS) / (1024 * 1024 * 1024) << '\n';
    cout.flush();
    cout << "[GPU] ~ Memory requested for results in MAIN MEMORY (GiB): " << (double)(sizeof(int) * 2 * GPUBufferSize * GPUSTREAMS) / (1024 * 1024 * 1024) << '\n';
    cout.flush();

	///////////////////////////////////
	//END ALLOCATE MEMORY FOR THE RESULT SET
	///////////////////////////////////



	/////////////////////////////////
	//SET OPENMP ENVIRONMENT VARIABLES
	////////////////////////////////

	omp_set_num_threads(GPUSTREAMS);

	/////////////////////////////////
	//END SET OPENMP ENVIRONMENT VARIABLES
	////////////////////////////////



	/////////////////////////////////
	//CREATE STREAMS
	////////////////////////////////

	hipStream_t stream[GPUSTREAMS];

	for (int i = 0; i < GPUSTREAMS; i++)
    {
		hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking);
	}



	///////////////////////////////////
	//LAUNCH KERNEL IN BATCHES
	///////////////////////////////////

	//since we use the strided scheme, some of the batch sizes
	//are off by 1 of each other, a first group of batches will
	//have 1 extra data point to process, and we calculate which batch numbers will
	//have that.  The batchSize is the lower value (+1 is added to the first ones)

    unsigned int datasetSize = *DBSIZE;

	// unsigned int batchSize = (*DBSIZE) / numBatches;
    unsigned int batchSize = datasetSize / numBatches;
	// unsigned int batchesThatHaveOneMore = (*DBSIZE) - (batchSize * numBatches); //batch number 0- < this value have one more
    unsigned int batchesThatHaveOneMore = datasetSize - (batchSize * numBatches);
    cout << "[GPU] ~ Batches that have one more GPU thread: " << batchesThatHaveOneMore << " batchSize(N): " << batchSize << '\n';
    cout.flush();

	uint64_t totalResultsLoop = 0;

    unsigned int * batchBegin = new unsigned int[GPUSTREAMS];
    for(int i = 0; i < GPUSTREAMS; i++)
    {
        batchBegin[i] = 0;
    }
    unsigned int * dev_batchBegin;
    errCode = hipMalloc( (void**)&dev_batchBegin, GPUSTREAMS * sizeof(unsigned int));
    if(errCode != hipSuccess)
    {
        cout << "[GPU] ~ Error: Alloc queue index -- error with code " << errCode << '\n';
        cout.flush();
    }

    hipEvent_t * startKernel = new hipEvent_t[GPUSTREAMS];
    hipEvent_t * stopKernel = new hipEvent_t[GPUSTREAMS];
    float * kernelTimes = new float[GPUSTREAMS];
    unsigned int * nbKernelInvocation = new unsigned int[GPUSTREAMS];
    unsigned int * nbQueryPoint = new unsigned int [GPUSTREAMS];
    double computeTime = 0;

    for(int i = 0; i < GPUSTREAMS; ++i)
    {
        hipEventCreate(&startKernel[i]);
        hipEventCreate(&stopKernel[i]);
        kernelTimes[i] = 0;
        nbKernelInvocation[i] = 0;
        nbQueryPoint[i] = 0;
    }

    if (SM_HYBRID == searchMode)
    {
        double tStartCompute = omp_get_wtime();
        #pragma omp parallel reduction(+: totalResultsLoop) num_threads(GPUSTREAMS)
        {
            unsigned int tid = omp_get_thread_num();
            std::pair<unsigned int, unsigned int> gpuBatch = std::make_pair(tid * batchSize, tid * batchSize + batchSize);
            // std::pair<unsigned int, unsigned int> gpuBatch = getBatchFromQueue(*DBSIZE, batchSize);

            unsigned int localBatchCounter = tid;

            do
            {
                nbQueryPoint[tid] += gpuBatch.second - gpuBatch.first;
                #if !SILENT_GPU
                    printf("[GPU | T_%d] ~ New batch: begin = %d, end = %d\n", tid, gpuBatch.first, gpuBatch.second);
                #endif

                errCode = hipMemcpy( &dev_batchBegin[tid], &gpuBatch.first, sizeof(unsigned int), hipMemcpyHostToDevice );
            	if(errCode != hipSuccess)
                {
            		cout << "[GPU] ~ Error: queue index copy to device -- error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
            	}

                // N[tid] = batchSize;
                N[tid] = gpuBatch.second - gpuBatch.first;
                errCode = hipMemcpyAsync( &dev_N[tid], &N[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if(errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: N Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // the batched result set size (reset to 0):
        		cnt[tid] = 0;
        		errCode = hipMemcpyAsync( &dev_cnt[tid], &cnt[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if(errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: dev_cnt memcpy Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // the offset for batching, which keeps track of where to start processing at each batch
        		batchOffset[tid] = numBatches; //for the strided
        		errCode = hipMemcpyAsync( &dev_offset[tid], &batchOffset[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if(errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: dev_offset memcpy Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // the batch number for batching with strided
        		batchNumber[tid] = localBatchCounter;
        		errCode = hipMemcpyAsync( &dev_batchNumber[tid], &batchNumber[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
        		if(errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: dev_batchNumber memcpy Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                const int TOTALBLOCKS = ceil( (1.0 * (N[tid])) / (1.0 * BLOCKSIZE) );
                #if !SILENT_GPU
                    cout << "[GPU] ~ Total blocks: " << TOTALBLOCKS << '\n';
                    cout.flush();
                #endif

                // double beginKernel = omp_get_wtime();

                // kernelNDGridIndexGlobalHybridDynamic<<< THREADPERPOINT * TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                //         dev_database, dev_sortedDatabase, dev_originPointIndex, dev_epsilon, dev_grid,
                //         dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, dev_gridCellNDMask,
                //         dev_gridCellNDMaskOffsets, dev_pointIDKey[tid], dev_pointInDistValue[tid]);

                hipEventRecord(startKernel[tid], stream[tid]);
                #if UNICOMP
                    kernelNDGridIndexGlobalUnicomp<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                        &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                        dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                        dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
                #elif LID_UNICOMP
                    kernelNDGridIndexGlobalLinearIDUnicomp<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                        &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                        dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                        dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
                #else
                    kernelNDGridIndexGlobal<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[tid], &dev_N[tid],
                        &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                        dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                        dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
                #endif
                hipEventRecord(stopKernel[tid], stream[tid]);

                errCode = hipGetLastError();
                #if !SILENT_GPU
            		cout << "\n\n[GPU] ~ KERNEL LAUNCH RETURN: " << errCode << '\n';
                    cout.flush();
                #endif
        		if ( hipSuccess != hipGetLastError() )
                {
        			cout << "\n\n[GPU] ~ ERROR IN KERNEL LAUNCH. ERROR: " << hipSuccess << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}

                // find the size of the number of results
        		errCode = hipMemcpyAsync( &cnt[tid], &dev_cnt[tid], sizeof(unsigned int), hipMemcpyDeviceToHost, stream[tid] );
        		if(errCode != hipSuccess)
                {
        			cout << "[GPU] ~ Error: getting cnt from GPU Got error with code " << errCode << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}
                #if !SILENT_GPU
        		else{
                    cout << "[GPU] ~ Result set size within epsilon: " << cnt[tid] << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        		}
                #endif

        		// double endKernel = omp_get_wtime();

                nbKernelInvocation[tid]++;

                hipEventSynchronize(stopKernel[tid]);
                float timeKernel = 0;
                hipEventElapsedTime(&timeKernel, startKernel[tid], stopKernel[tid]);
                kernelTimes[tid] += timeKernel;

                thrust::device_ptr<int> dev_keys_ptr(dev_pointIDKey[tid]);
        		thrust::device_ptr<int> dev_data_ptr(dev_pointInDistValue[tid]);

                try{
        			thrust::sort_by_key(thrust::cuda::par.on(stream[tid]), dev_keys_ptr, dev_keys_ptr + cnt[tid], dev_data_ptr);
        		}
        		catch(std::bad_alloc &e)
        		{
        			std::cerr << "[GPU] ~ Ran out of memory while sorting, " << GPUBufferSize << '\n';
                    cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                    cout.flush();
        			exit(1);
        		}

                hipMemcpyAsync(thrust::raw_pointer_cast(pointIDKey[tid]), thrust::raw_pointer_cast(dev_keys_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);
        		hipMemcpyAsync(thrust::raw_pointer_cast(pointInDistValue[tid]), thrust::raw_pointer_cast(dev_data_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);

                hipStreamSynchronize(stream[tid]);

                double tableconstuctstart = omp_get_wtime();
        		//set the number of neighbors in the pointer struct:
        		(*pointersToNeighbors)[localBatchCounter].sizeOfDataArr = cnt[tid];
        		(*pointersToNeighbors)[localBatchCounter].dataPtr = new int[cnt[tid]];

                constructNeighborTableKeyValueWithPtrs(pointIDKey[tid], pointInDistValue[tid], neighborTable, (*pointersToNeighbors)[localBatchCounter].dataPtr, &cnt[tid]);

                double tableconstuctend = omp_get_wtime();

                #if !SILENT_GPU
                    cout << "[GPU] ~ Table construct time: " << tableconstuctend - tableconstuctstart << '\n';
                    cout.flush();
                #endif

                // add the batched result set size to the total count
        		totalResultsLoop += cnt[tid];

                #if !SILENT_GPU
                    cout << "[GPU] ~ Running total of total size of result array, tid: " << tid << ", " << totalResultsLoop << '\n';
                    cout.flush();
                #endif

                gpuBatch = getBatchFromQueue(*DBSIZE, batchSize);
                // gpuBatch = getBatchFromQueue(9 * batchSize, batchSize);

            }while(0 != gpuBatch.second);

        } // parallel section
        double tEndCompute = omp_get_wtime();
        computeTime = tEndCompute - tStartCompute;
    }
    else
    { // searchModes that have a fixed number of queries (e.g., original GPU kernel or static partitioning)
        errCode = hipMemcpy( &dev_batchBegin[0], batchBegin, sizeof(unsigned int), hipMemcpyHostToDevice );
        if(errCode != hipSuccess)
        {
            cout << "[GPU] ~ Error: queue index copy to device -- error with code " << errCode << '\n';
            cout << "  Details: " << hipGetErrorString(errCode) << '\n';
            cout.flush();
        }

        double computeTimeStart = omp_get_wtime();

        //FOR LOOP OVER THE NUMBER OF BATCHES STARTS HERE
    	//i=0...numBatches
        #pragma omp parallel for schedule(dynamic, 1) reduction(+: totalResultsLoop) num_threads(GPUSTREAMS)
    	for (int i = 0; i < numBatches; ++i)
        // for(int i = 0; i < 9; ++i)
    	{
    		int tid = omp_get_thread_num();

            #if !SILENT_GPU
                cout << "[GPU] ~ tid " << tid << ", starting iteration " << i << '\n';
                cout.flush();
            #endif

    		//N NOW BECOMES THE NUMBER OF POINTS TO PROCESS PER BATCH
    		//AS ONE GPU THREAD PROCESSES A SINGLE POINT

    		if (i < batchesThatHaveOneMore)
    		{
    			N[tid] = batchSize + 1;
                #if !SILENT_GPU
                // cout << "[GPU] ~ N (GPU threads): " << N[tid] << ", tid " << tid << '\n';
                // cout.flush();
                #endif
    		}
    		else
    		{
    			N[tid] = batchSize;
                #if !SILENT_GPU
                    cout << "[GPU] ~ N (1 less): " << N[tid] << ", tid " << tid << '\n';
                    cout.flush();
                #endif
    		}

            nbQueryPoint[tid] += N[tid];

    		//set relevant parameters for the batched execution that get reset

    		//copy N to device
    		//N IS THE NUMBER OF THREADS
    		errCode = hipMemcpyAsync( &dev_N[tid], &N[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if(errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: N Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		//the batched result set size (reset to 0):
    		cnt[tid] = 0;
    		errCode = hipMemcpyAsync( &dev_cnt[tid], &cnt[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if(errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: dev_cnt memcpy Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		//the offset for batching, which keeps track of where to start processing at each batch
    		batchOffset[tid] = numBatches; //for the strided
    		errCode = hipMemcpyAsync( &dev_offset[tid], &batchOffset[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if(errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: dev_offset memcpy Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		//the batch number for batching with strided
    		batchNumber[tid] = i;
    		errCode = hipMemcpyAsync( &dev_batchNumber[tid], &batchNumber[tid], sizeof(unsigned int), hipMemcpyHostToDevice, stream[tid] );
    		if(errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: dev_batchNumber memcpy Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		const int TOTALBLOCKS = ceil( (1.0 * (N[tid])) / (1.0 * BLOCKSIZE) );
            #if !SILENT_GPU
                cout << "[GPU] ~ Total blocks: " << TOTALBLOCKS << '\n';
                cout.flush();
            #endif

    		//execute kernel
    		//0 is shared memory pool

    		// double beginKernel = omp_get_wtime();

            hipEventRecord(startKernel[tid]);
            #if UNICOMP
                kernelNDGridIndexGlobalUnicomp<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[0], &dev_N[tid],
                    &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                    dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                    dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
            #elif LID_UNICOMP
                kernelNDGridIndexGlobalLinearIDUnicomp<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[0], &dev_N[tid],
                    &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                    dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                    dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
            #else
                kernelNDGridIndexGlobal<<< TOTALBLOCKS, BLOCKSIZE, 0, stream[tid] >>>(&dev_batchBegin[0], &dev_N[tid],
                    &dev_offset[tid], &dev_batchNumber[tid], dev_database, nullptr, dev_originPointIndex, dev_epsilon, dev_grid,
                    dev_indexLookupArr,dev_gridCellLookupArr, dev_minArr, dev_nCells, &dev_cnt[tid], dev_nNonEmptyCells, /*dev_gridCellNDMask,
                    dev_gridCellNDMaskOffsets,*/ dev_pointIDKey[tid], dev_pointInDistValue[tid]);
            #endif
            hipEventRecord(stopKernel[tid]);

            errCode = hipGetLastError();
            #if !SILENT_GPU
        		cout << "\n\n[GPU] ~ KERNEL LAUNCH RETURN: " << errCode << '\n';
                cout.flush();
            #endif
    		if ( hipSuccess != hipGetLastError() )
            {
    			cout << "\n\n[GPU] ~ ERROR IN KERNEL LAUNCH. ERROR: " << hipSuccess << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}

    		// find the size of the number of results
    		errCode = hipMemcpyAsync( &cnt[tid], &dev_cnt[tid], sizeof(unsigned int), hipMemcpyDeviceToHost, stream[tid] );
    		if(errCode != hipSuccess)
            {
    			cout << "[GPU] ~ Error: getting cnt from GPU Got error with code " << errCode << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}
            #if !SILENT_GPU
    		else{
                cout << "[GPU] ~ Result set size within epsilon: " << cnt[tid] << '\n';
                cout << "  Details: " << hipGetErrorString(errCode) << '\n';
                cout.flush();
    		}
            #endif

    		// double endKernel = omp_get_wtime();
            // kernelTime[tid] += endKernel - beginKernel;
    		//cout << "Single kernel execution time = " << endKernel - beginKernel << " ms" << endl;

            nbKernelInvocation[tid]++;

            hipEventSynchronize(stopKernel[tid]);
            float timeKernel = 0;
            hipEventElapsedTime(&timeKernel, startKernel[tid], stopKernel[tid]);
            kernelTimes[tid] += timeKernel;


    		////////////////////////////////////
    		//SORT THE TABLE DATA ON THE GPU
    		//THERE IS NO ORDERING BETWEEN EACH POINT AND THE ONES THAT IT'S WITHIN THE DISTANCE OF
    		////////////////////////////////////

    		/////////////////////////////
    		//ONE PROBLEM WITH NOT TRANSFERING THE RESULT OFF OF THE DEVICE IS THAT
    		//YOU CAN'T RESIZE THE RESULTS TO BE THE SIZE OF *CNT
    		//SO THEN YOU HAVE POTENTIALLY LOTS OF WASTED SPACE
    		/////////////////////////////

    		//sort by key with the data already on the device:
    		//wrap raw pointer with a device_ptr to use with Thrust functions
    		thrust::device_ptr<int> dev_keys_ptr(dev_pointIDKey[tid]);
    		thrust::device_ptr<int> dev_data_ptr(dev_pointInDistValue[tid]);

    		//XXXXXXXXXXXXXXXX
    		//THRUST USING STREAMS REQUIRES THRUST V1.8
    		//XXXXXXXXXXXXXXXX

    		try{
    			thrust::sort_by_key(thrust::cuda::par.on(stream[tid]), dev_keys_ptr, dev_keys_ptr + cnt[tid], dev_data_ptr);
    		}
    		catch(std::bad_alloc &e)
    		{
    			std::cerr << "[GPU] ~ Ran out of memory while sorting, " << GPUBufferSize << '\n';
                cout.flush();
    			exit(1);
    		}

            // cout << "[GPU] ~ Thrust sort by key\n";
            // cout.flush();
            // cout << "[GPU] ~ Copy size: " << cnt[tid] * sizeof(int) << '\n';
            // cout.flush();

    		//thrust with streams into individual buffers for each batch
    		hipMemcpyAsync(thrust::raw_pointer_cast(pointIDKey[tid]), thrust::raw_pointer_cast(dev_keys_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);
    		hipMemcpyAsync(thrust::raw_pointer_cast(pointInDistValue[tid]), thrust::raw_pointer_cast(dev_data_ptr), cnt[tid] * sizeof(int), hipMemcpyDeviceToHost, stream[tid]);

            // cout << "[GPU] ~ Async memcpy of pointers\n";
            // cout.flush();

    		//need to make sure the data is copied before constructing portion of the neighbor table
    		hipStreamSynchronize(stream[tid]);

            // cout << "[GPU] ~ Stream synchronization\n";
            // cout.flush();

    		double tableconstuctstart = omp_get_wtime();
    		//set the number of neighbors in the pointer struct:
    		(*pointersToNeighbors)[i].sizeOfDataArr = cnt[tid];
    		(*pointersToNeighbors)[i].dataPtr = new int[cnt[tid]];

    		constructNeighborTableKeyValueWithPtrs(pointIDKey[tid], pointInDistValue[tid], neighborTable, (*pointersToNeighbors)[i].dataPtr, &cnt[tid]);

    		//cout <<"In make neighbortable. Data array ptr: "<<(*pointersToNeighbors)[i].dataPtr<<" , size of data array: "<<(*pointersToNeighbors)[i].sizeOfDataArr;cout.flush();

    		double tableconstuctend = omp_get_wtime();

            #if !SILENT_GPU
                cout << "[GPU] ~ Table construct time: " << tableconstuctend - tableconstuctstart << '\n';
                cout.flush();
            #endif

    		//add the batched result set size to the total count
    		totalResultsLoop += cnt[tid];

            #if !SILENT_GPU
                cout << "[GPU] ~ Running total of total size of result array, tid: " << tid << ", " << totalResultsLoop << '\n';
                cout.flush();
            #endif

    	} //END LOOP OVER THE GPU BATCHES

        double computeEndTime = omp_get_wtime();
        computeTime = computeEndTime - computeTimeStart;
        // cout << "[GPU | RESULT] ~ Compute time for the GPU = " << computeEndTime - computeTimeStart << '\n';
        // cout.flush();

    }

    unsigned int nbQueryPointTotal = 0;
    for(int i = 0; i < GPUSTREAMS; ++i)
    {
        nbQueryPointTotal += nbQueryPoint[i];
    }
    printf("[GPU | RESULT] ~ Query points computed by the GPU: %d\n", nbQueryPointTotal);
    printf("[GPU | RESULT] ~ Compute time for the GPU: %f\n", computeTime);

    cout << "[GPU] ~ Total result set size on host: " << totalResultsLoop << "\033[00m\n";
    cout.flush();

	*totalNeighbors = totalResultsLoop;

	double tKernelResultsEnd = omp_get_wtime();

    cout << "[GPU] ~ Time to launch kernel and execute everything except freeing memory: " << tKernelResultsEnd - tKernelResultsStart << '\n';
    cout.flush();

	///////////////////////////////////
	//END GET RESULT SET
	///////////////////////////////////



    // double kernelExecutionTime = 0;
    // for(int i = 0; i < GPUSTREAMS; ++i)
    // {
    //     kernelExecutionTime += kernelTimes[i];
    // }
    // cout << "[RESULT] ~ Total kernel execution times: " << kernelExecutionTime << ", average = " << kernelExecutionTime / nbKernelInvocation << '\n';
    // cout << "[RESULT] ~ Total kernel execution times: " << '\n';
    // for(int i = 0; i < GPUSTREAMS; ++i)
    // {
    //     cout << "  [RESULT] ~ Stream " << i << ", " << nbQueryPoint[i] << " queries computed: total kernel time = " << kernelTimes[i]
    //         << ", average = " << kernelTimes[i] / nbKernelInvocation[i] << '\n';
    // }
    // cout.flush();



	///////////////////////////////////
	//FREE MEMORY FROM THE GPU
	///////////////////////////////////
	// if (NUM_TRIALS>1)
	// {

	double tFreeStart = omp_get_wtime();

	for (int i = 0; i < GPUSTREAMS; i++)
    {
		errCode = hipStreamDestroy(stream[i]);
		if(errCode != hipSuccess) {
			cout << "[GPU] ~ Error: destroying stream" << errCode << '\n';
            cout.flush();
		}
	}

	free(totalResultSetCnt);
	free(cnt);
	free(N);
	free(batchOffset);
	free(batchNumber);

	//free the data on the device

	hipFree(dev_N);
	hipFree(dev_cnt);
	hipFree(dev_offset);
	hipFree(dev_batchNumber);


	//free data related to the individual streams for each batch
	for (int i = 0; i < GPUSTREAMS; i++)
    {
		//free the data on the device
		hipFree(dev_pointIDKey[i]);
		hipFree(dev_pointInDistValue[i]);

		//free on the host
		hipHostFree(pointIDKey[i]);
		hipHostFree(pointInDistValue[i]);
	}

    // hipFree(dev_pointIDKey);
    // hipFree(dev_pointInDistValue);

	//free pinned memory on host
	hipHostFree(pointIDKey);
	hipHostFree(pointInDistValue);

	double tFreeEnd = omp_get_wtime();

    cout << "[GPU] ~ Time freeing memory: " << tFreeEnd - tFreeStart << '\n';
    cout.flush();
	// printf("\nTime freeing memory: %f", tFreeEnd - tFreeStart);
	// }
	cout << "\n[GPU] ~ ** last error at end of fn batches (could be from freeing memory): " << hipGetLastError() << "\n\n";
    cout.flush();

} // NDGridIndexGlobal





void warmUpGPU(){
	// initialize all ten integers of a device_vector to 1
	thrust::device_vector<int> D(10, 1);
	// set the first seven elements of a vector to 9
	thrust::fill(D.begin(), D.begin() + 7, 9);
	// initialize a host_vector with the first five elements of D
	thrust::host_vector<int> H(D.begin(), D.begin() + 5);
	// set the elements of H to 0, 1, 2, 3, ...
	thrust::sequence(H.begin(), H.end()); // copy all of H back to the beginning of D
	thrust::copy(H.begin(), H.end(), D.begin());
	// print D
	for(int i = 0; i < D.size(); i++)
    {
		cout << " D[" << i << "] = " << D[i];
    }

	return;
}





void constructNeighborTableKeyValueWithPtrs(
    int * pointIDKey,
    int * pointInDistValue,
    struct neighborTableLookup * neighborTable,
    int * pointersToNeighbors,
    unsigned int * cnt)
{

	//copy the value data:
	std::copy(pointInDistValue, pointInDistValue + (*cnt), pointersToNeighbors);

	//Step 1: find all of the unique keys and their positions in the key array
	unsigned int numUniqueKeys = 0;

	std::vector<keyData> uniqueKeyData;

	keyData tmp;
	tmp.key = pointIDKey[0];
	tmp.position = 0;
	uniqueKeyData.push_back(tmp);

	//we assign the ith data item when iterating over i+1th data item,
	//so we go 1 loop iteration beyond the number (*cnt)
	for (int i = 1; i < (*cnt) + 1; i++)
    {
		if (pointIDKey[i - 1] != pointIDKey[i]){
			numUniqueKeys++;
			tmp.key = pointIDKey[i];
			tmp.position = i;
			uniqueKeyData.push_back(tmp);
		}
	}

	//insert into the neighbor table the values based on the positions of
	//the unique keys obtained above.
	for (int i = 0; i < uniqueKeyData.size() - 1; i++)
    {
		int keyElem = uniqueKeyData[i].key;
		neighborTable[keyElem].pointID = keyElem;
		neighborTable[keyElem].indexmin = uniqueKeyData[i].position;
		neighborTable[keyElem].indexmax = uniqueKeyData[i + 1].position - 1;

		//update the pointer to the data array for the values
		neighborTable[keyElem].dataPtr = pointersToNeighbors;
	}
}
