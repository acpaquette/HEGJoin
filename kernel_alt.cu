#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>

#include "kernel_alt.h"
#include "params.h"
#include "structs.h"

// Specific to tensor cores
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_fp16.h>
using namespace nvcuda;
using namespace cooperative_groups;



__device__ uint64_t getLinearID_nDimensionsGPUKernelAlt(
	unsigned int* indexes,
	unsigned int* dimLen,
	unsigned int nDimensions)
{
    uint64_t offset = 0;
	uint64_t multiplier = 1;

	for (int i = 0; i < nDimensions; ++i)
	{
		offset += (uint64_t) indexes[i] * multiplier;
		multiplier *= dimLen[i];
	}

	return offset;
}


__global__ void convertAndResizeDataset(
    DTYPE* in,
    half* out,
    unsigned int nbQueries)
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < nbQueries)
    {
	    // Copy the coordinates from the dataset
        for (int i = 0; i < GPUNUMDIM; ++i)
        {
            out[tid * COMPUTE_DIM + i] = (half)in[tid * GPUNUMDIM + i];
        }
		// Fill with 0s so the dimensionality of the dataset is a multiple of 16
		for (int i = GPUNUMDIM; i < COMPUTE_DIM; ++i)
		{
			out[tid * COMPUTE_DIM + i] = (half)0.0;
		}
    }
    // The original dataset does not have the 15 supplemental points, so need to do it in another step
    if (tid < 15)
    {
	    // Create "fake points" with 0s coordinates so the last query point will still have 15 points after when loading using load_matrix_sync
		for (int i = 0; i < COMPUTE_DIM; ++i)
		{
			out[tid * COMPUTE_DIM + i] = (half)0.0;
		}
    }
}



__global__ void convertMinArr(
	DTYPE* in,
	half* out)
{
	for (int i = 0; i < NUMINDEXEDDIM; ++i)
	{
		out[i] = (half)in[i];
	}
}


__global__ void convertMinArrHalf2(
	DTYPE* in,
	half2* out)
{
	for (int i = 0; i < NUMINDEXEDDIM / 2; ++i)
	{
		out[i] = __floats2half2_rn(in[i * 2], in[i * 2 + 1]);
	}
}


__global__ void convertFloatToHalf2(
    float* input,
    // half2* tmp,
    half2* output,
    unsigned int nbPoints)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < nbPoints)
    {
		if (0 == (GPUNUMDIM % 2))
		{
			for (int i = 0; i < HALF2_DIM; ++i)
			{
				output[tid * HALF2_DIM + i] = __floats2half2_rn(input[tid * GPUNUMDIM + i * 2], input[tid * GPUNUMDIM + i * 2 + 1]);
			}

			// for (int i = 0; i < GPUNUMDIM; i += 2)
			// {
			// 	output[tid * HALF2_DIM + (i / 2)] = __floats2half2_rn(input[tid * GPUNUMDIM + i], input[tid * GPUNUMDIM + i + 1]);
			// }
		} else {
			for (int i = 0; i < (GPUNUMDIM / 2); ++i)
			{
				output[tid * HALF2_DIM + i] = __floats2half2_rn(input[tid * GPUNUMDIM + i * 2], input[tid * GPUNUMDIM + i * 2 + 1]);
			}
			output[tid * HALF2_DIM + (GPUNUMDIM / 2) + 1] = __floats2half2_rn(input[tid * GPUNUMDIM + GPUNUMDIM - 1], 0.0f);
		}
    }
}


__global__ void batchEstimatorKernel_alt(
	unsigned int* N,
	unsigned int* sampleOffset,
	DTYPE* database,
	unsigned int* originPointIndex,
	DTYPE* epsilon,
	struct grid* grid,
	unsigned int* gridLookupArr,
	struct gridCellLookup* gridCellLookupArr,
	DTYPE* minArr,
	unsigned int* nCells,
	unsigned int* cnt,
	unsigned int* nNonEmptyCells,
	unsigned int* estimatedResult,
	unsigned int* candidatesCounter)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if ((*N) <= tid)
	{
		return;
	}

	unsigned int pointId = tid * (*sampleOffset);

	DTYPE point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; ++i)
	{
		point[i] = database[ originPointIndex[pointId] * GPUNUMDIM + i ];
	}

	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int nDMinCellIDs[NUMINDEXEDDIM];
	unsigned int nDMaxCellIDs[NUMINDEXEDDIM];

	for (int i = 0; i < NUMINDEXEDDIM; ++i)
	{
		nDCellIDs[i] = (point[i] - minArr[i]) / (*epsilon);
		nDMinCellIDs[i] = max(0, nDCellIDs[i] - 1);
		nDMaxCellIDs[i] = min(nCells[i] - 1, nDCellIDs[i] + 1);
	}

	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	unsigned int localNeighborCounter = 0;
	unsigned int localCandidateCounter = 0;

	for (loopRng[0] = nDMinCellIDs[0]; loopRng[0] <= nDMaxCellIDs[0]; loopRng[0]++)
		for (loopRng[1] = nDMinCellIDs[1]; loopRng[1] <= nDMaxCellIDs[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; ++x)
			{
				indexes[x] = loopRng[x];
			}

			uint64_t calcLinearID = getLinearID_nDimensionsGPUKernelAlt(indexes, nCells, NUMINDEXEDDIM);
			//compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
			//a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)

			struct gridCellLookup tmp;
			tmp.gridLinearID = calcLinearID;

			if (thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
			{
				struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
				unsigned int GridIndex = resultBinSearch->idx;

				for (int k = grid[GridIndex].indexmin; k <= grid[GridIndex].indexmax; ++k)
				{
					DTYPE runningTotalDist = 0;
					unsigned int dataIdx = gridLookupArr[k];

					for (int l = 0; l < GPUNUMDIM; ++l)
					{
						runningTotalDist += (database[dataIdx * GPUNUMDIM + l]  - point[l])
								* (database[dataIdx * GPUNUMDIM + l] - point[l]);
					}

					#if ACCUM_PREC == 16
					if (hsqrt(runningTotalDist) <= (*epsilon))
					#else
					if (sqrt(runningTotalDist) <= (*epsilon))
					#endif
					{
						unsigned int idx = atomicAdd(cnt, int(1));
						localNeighborCounter++;
					}
				}

				localCandidateCounter += grid[GridIndex].indexmax - grid[GridIndex].indexmin + 1;
			}
		} //end loop body

	estimatedResult[tid] = localNeighborCounter;
	candidatesCounter[tid] = localCandidateCounter;
}


__device__ void evaluateCell_alt(
	unsigned int* nCells,
	unsigned int* indexes,
	struct gridCellLookup* gridCellLookupArr,
	unsigned int* nNonEmptyCells,
	half* database,
	DTYPE* epsilon,
	struct grid* grid,
	unsigned int* gridLookupArr,
	half* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx,
	unsigned int* nDCellIDs)
{
	// Compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
	// a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)
	uint64_t calcLinearID = getLinearID_nDimensionsGPUKernelAlt(indexes, nCells, NUMINDEXEDDIM);

	struct gridCellLookup tmp;
	tmp.gridLinearID = calcLinearID;
	//find if the cell is non-empty
	if(thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
	{
		//compute the neighbors for the adjacent non-empty cell
		struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr,
                gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
		unsigned int GridIndex = resultBinSearch->idx;

		for(int k = grid[GridIndex].indexmin; k <= grid[GridIndex].indexmax; ++k)
        {
			#if ILP == 1
				evalPoint_alt(gridLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			#else
				evalPointILP_alt(gridLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			#endif
		}
	}
}


__device__ void evaluateCellHalf2(
	unsigned int* nCells,
	unsigned int* indexes,
	struct gridCellLookup* gridCellLookupArr,
	unsigned int* nNonEmptyCells,
	half2* database,
	DTYPE* epsilon,
	struct grid* grid,
	unsigned int* gridLookupArr,
	half2* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx,
	unsigned int* nDCellIDs)
{
	// Compare the linear ID with the gridCellLookupArr to determine if the cell is non-empty: this can happen because one point says
	// a cell in a particular dimension is non-empty, but that's because it was related to a different point (not adjacent to the query point)
	uint64_t calcLinearID = getLinearID_nDimensionsGPUKernelAlt(indexes, nCells, NUMINDEXEDDIM);

	struct gridCellLookup tmp;
	tmp.gridLinearID = calcLinearID;
	//find if the cell is non-empty
	if(thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
	{
		//compute the neighbors for the adjacent non-empty cell
		struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr,
                gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
		unsigned int GridIndex = resultBinSearch->idx;

		for(int k = grid[GridIndex].indexmin; k <= grid[GridIndex].indexmax; ++k)
        {
			#if ILP == 1
				evalPointHalf2(gridLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			#else
				evalPointHalf2ILP(gridLookupArr, k, database, epsilon, point, cnt, pointIDKey, pointInDistVal, pointIdx);
			#endif
		}
	}
}


__forceinline__ __device__ void evalPoint_alt(
	unsigned int* gridLookupArr,
	int k,
	half* database,
	DTYPE* epsilon,
	half* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx)
{
	DTYPE runningTotalDist = 0;
	unsigned int dataIdx = gridLookupArr[k];

	for(int l = 0; l < GPUNUMDIM; ++l)
    {
		runningTotalDist += (DTYPE)((database[dataIdx * COMPUTE_DIM + l] - point[l]) * (database[dataIdx * COMPUTE_DIM + l] - point[l]));
	}

    // if(runningTotalDist <= ((*epsilon) * (*epsilon)))
    #if DTYPE_PREC == 16
	if(hsqrt(runningTotalDist) <= (*epsilon))
    #else
    if(sqrt(runningTotalDist) <= (*epsilon))
    #endif
    {
		unsigned int idx = atomicAdd(cnt, int(1));
		pointIDKey[idx] = pointIdx;
		pointInDistVal[idx] = dataIdx;
	}
}


__forceinline__ __device__ void evalPointILP_alt(
	unsigned int* gridLookupArr,
	int k,
	half* database,
	DTYPE* epsilon,
	half* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx)
{
	unsigned int dataIdx = gridLookupArr[k];
	DTYPE runningTotalDist[ILP];

	// const unsigned int unrollSize = ILP;

	#pragma unroll
	for (int i = 0; i < ILP; ++i)
	{
		runningTotalDist[i] = 0.0;
	}

	for(int i = 0; i < GPUNUMDIM; i += ILP)
    {
		#pragma unroll
		for (int j = 0; j < ILP && (i + j) < GPUNUMDIM; ++j)
		{
			runningTotalDist[j] += (DTYPE)((database[dataIdx * COMPUTE_DIM + i + j] - point[i + j]) * (database[dataIdx * COMPUTE_DIM + i + j] - point[i + j]));
		}

		#if SHORT_CIRCUIT
			#pragma unroll
			for (int j = 1; j < ILP; ++j)
			{
				runningTotalDist[0] += runningTotalDist[j];
				runningTotalDist[j] = 0.0;
			}

			#if DTYPE_PREC == 16
			if (hsqrt(runningTotalDist[0]) > (*epsilon))
			#else
			if (sqrt(runningTotalDist[0]) > (*epsilon))
			#endif
			{
				return;
			}
		#endif
	}

	#if !SHORT_CIRCUIT
		#pragma unroll
		for (int i = 1; i < ILP; ++i)
		{
			runningTotalDist[0] += runningTotalDist[i];
		}
	#endif

    // if(runningTotalDist <= ((*epsilon) * (*epsilon)))
    #if DTYPE_PREC == 16
	if(hsqrt(runningTotalDist[0]) <= (*epsilon))
    #else
    if(sqrt(runningTotalDist[0]) <= (*epsilon))
    #endif
    {
		unsigned int idx = atomicAdd(cnt, int(1));
		pointIDKey[idx] = pointIdx;
		pointInDistVal[idx] = dataIdx;
	}
}


__forceinline__ __device__ void evalPointHalf2(
	unsigned int* gridLookupArr,
	int k,
	half2* database,
	DTYPE* epsilon,
	half2* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx)
{
	unsigned int dataIdx = gridLookupArr[k];

	#if DTYPE_PREC == 16
		half runningTotalDist = 0.0;
		for (int i = 0; i < HALF2_DIM; ++i)
		{
			half2 tmpDistance = __hsub2(__hmul2(point[i], database[dataIdx * HALF2_DIM + i]),
										__hmul2(point[i], database[dataIdx * HALF2_DIM + i]));
			runningTotalDist += __low2half(tmpDistance) + __high2half(tmpDistance);
		}
	#else
		float runningTotalDist = 0.0;
		for (int i = 0; i < HALF2_DIM; ++i)
		{
			float2 tmpDistance = __half22float2(__hsub2(__hmul2(point[i], database[dataIdx * HALF2_DIM + i]),
														__hmul2(point[i], database[dataIdx * HALF2_DIM + i])));
			runningTotalDist += tmpDistance.x + tmpDistance.y;
		}
	#endif

    // if(runningTotalDist <= ((*epsilon) * (*epsilon)))
    #if DTYPE_PREC == 16
	if(hsqrt(runningTotalDist) <= (*epsilon))
    #else
    if(sqrt(runningTotalDist) <= (*epsilon))
    #endif
    {
		unsigned int idx = atomicAdd(cnt, int(1));
		pointIDKey[idx] = pointIdx;
		pointInDistVal[idx] = dataIdx;
	}
}


__forceinline__ __device__ void evalPointHalf2ILP(
	unsigned int* gridLookupArr,
	int k,
	half2* database,
	DTYPE* epsilon,
	half2* point,
	unsigned int* cnt,
	int* pointIDKey,
	int* pointInDistVal,
	int pointIdx)
{
	#if ILP > 1

	unsigned int dataIdx = gridLookupArr[k];
	DTYPE runningTotalDist[ILP / 2];

	// const unsigned int unrollSize = ILP;

	#pragma unroll
	for (int i = 0; i < ILP / 2; ++i)
	{
		runningTotalDist[i] = (DTYPE)0.0;
	}

	for(int i = 0; i < HALF2_DIM; i += (ILP / 2))
    {
		#pragma unroll
		for (int j = 0; j < (ILP / 2) && (i + j) < HALF2_DIM; ++j)
		{
			#if DTYPE_PREC == 16
				half2 tmpDistance = __hsub2(__hmul2(point[i + j], database[dataIdx * HALF2_DIM + i + j]),
											__hmul2(point[i + j], database[dataIdx * HALF2_DIM + i + j]));
				runningTotalDist[j] += __low2half(tmpDistance) + __high2half(tmpDistance);
			#else
				float2 tmpDistance = __half22float2(__hsub2(__hmul2(point[i + j], database[dataIdx * HALF2_DIM + i + j]),
															__hmul2(point[i + j], database[dataIdx * HALF2_DIM + i + j])));
				runningTotalDist[j] += tmpDistance.x + tmpDistance.y;
			#endif
		}

		#if SHORT_CIRCUIT
			#pragma unroll
			for (int j = 1; j < (ILP / 2); ++j)
			{
				runningTotalDist[0] += runningTotalDist[j];
				runningTotalDist[j] = (DTYPE)0.0;
			}

			#if DTYPE_PREC == 16
			if (hsqrt(runningTotalDist[0]) > (*epsilon))
			#else
			if (sqrt(runningTotalDist[0]) > (*epsilon))
			#endif
			{
				return;
			}
		#endif
	}

	#if !SHORT_CIRCUIT
		#pragma unroll
		for (int i = 1; i < (ILP / 2); ++i)
		{
			runningTotalDist[0] += runningTotalDist[i];
		}
	#endif

    // if(runningTotalDist <= ((*epsilon) * (*epsilon)))
    #if DTYPE_PREC == 16
	if(hsqrt(runningTotalDist[0]) <= (*epsilon))
    #else
    if(sqrt(runningTotalDist[0]) <= (*epsilon))
    #endif
    {
		unsigned int idx = atomicAdd(cnt, int(1));
		pointIDKey[idx] = pointIdx;
		pointInDistVal[idx] = dataIdx;
	}

	#endif
}



//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\



// __global__ void distanceCalculationBruteForceCuda(
//     DTYPE* database,
// 	unsigned int* nbQueries,
//     unsigned int* queryOffset,
//     DTYPE* epsilon,
//     unsigned int* nbNeighbors)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//
//     if ((*nbQueries) <= tid)
//     {
//         return;
//     }
//
//     DTYPE point[GPUNUMDIM];
//     for (int i = 0; i < GPUNUMDIM; ++i)
//     {
//         point[i] = database[tid * COMPUTE_DIM + i];
//     }
//
//     DTYPE runningTotalDist[ILP];
// 	const unsigned int unrollSize = ILP;
//
// 	for (int i = 0; i < (*nbQueries); ++i)
// 	{
// 		#pragma unroll unrollSize
// 		for (int j = 0; j < ILP; ++j)
// 		{
// 			runningTotalDist[j] = 0.0;
// 		}
//
// 		for(int j = 0; j < GPUNUMDIM; j += ILP)
// 	    {
// 			#pragma unroll unrollSize
// 			for (int k = 0; k < ILP && (j + k) < GPUNUMDIM; ++k)
// 			{
// 				runningTotalDist[k] += (database[i * COMPUTE_DIM + j + k] - point[j + k]) * (database[i * COMPUTE_DIM + j + k] - point[j + k]);
// 			}
//
// 			#if SHORT_CIRCUIT
// 				#pragma unroll (unrollSize - 1)
// 				for (int k = 1; k < ILP; ++k)
// 				{
// 					runningTotalDist[0] += runningTotalDist[k];
// 					runningTotalDist[k] = 0.0;
// 				}
//
// 				#if ACCUM_PREC == 16
// 				if (hsqrt(runningTotalDist[0]) > (*epsilon))
// 				#else
// 				if (sqrt(runningTotalDist[0]) > (*epsilon))
// 				#endif
// 				{
// 					return;
// 				}
// 			#endif
// 		}
//
// 		#if !SHORT_CIRCUIT
// 			#pragma unroll unrollSize
// 			for (int j = 1; j < ILP; ++j)
// 			{
// 				runningTotalDist[0] += runningTotalDist[j];
// 			}
// 		#endif
//
// 	    // if(runningTotalDist <= ((*epsilon) * (*epsilon)))
// 	    #if DTYPE_PREC == 16
// 		if(hsqrt(runningTotalDist[0]) <= (*epsilon))
// 	    #else
// 	    if(sqrt(runningTotalDist[0]) <= (*epsilon))
// 	    #endif
// 	    {
// 			atomicAdd(nbNeighbors, int(1));
// 		}
// 	}
// }


__global__ void distanceCalculationBruteForceCudaHalf(
	half* database,
	unsigned int* nbQueries,
	unsigned int* queryOffset,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if ((*nbQueries) <= tid)
	{
		return;
	}

	half point[GPUNUMDIM];
	for (int i = 0; i < GPUNUMDIM; ++i)
	{
		point[i] = database[tid * COMPUTE_DIM + i];
	}

	for (int i = 0; i < (*nbQueries); ++i)
	{
		DTYPE resultDistance = 0.0;

		for (int j = 0; j < GPUNUMDIM; ++j)
		{
			resultDistance += (DTYPE)((point[j] * database[i * COMPUTE_DIM + j]) - (point[j] * database[i * COMPUTE_DIM + j]));
		}

		#if DTYPE_PREC == 16
		if (hsqrt(resultDistance) <= (*epsilon))
		#else
		if (sqrt(resultDistance) <= (*epsilon))
		#endif
		{
			atomicAdd(nbNeighbors, int(1));
		}
	}
}



__global__ void distanceCalculationBruteForceCuda_half2(
	half2* database,
	unsigned int* nbQueries,
	unsigned int* queryOffset,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if ((*nbQueries) <= tid)
	{
		return;
	}

	half2 point[HALF2_DIM];
	for (int i = 0; i < HALF2_DIM; ++i)
	{
		point[i] = database[tid * HALF2_DIM + i];
	}

	for (int i = 0; i < (*nbQueries); ++i)
	{
		#if DTYPE_PREC == 16
			half resultDistance = 0.0;
			for (int j = 0; j < HALF2_DIM; ++j)
			{
				half2 tmpResult = __hsub2(__hmul2(point[j], database[i * HALF2_DIM + i]), __hmul2(point[j], database[i * HALF2_DIM + i]));
				resultDistance += __low2half(tmpResult) + __high2half(tmpResult);
			}
		#else
			float resultDistance = 0.0;
			for (int j = 0; j < HALF2_DIM; ++j)
			{
				half2 tmpResult = __hsub2(__hmul2(point[j], database[i * HALF2_DIM + i]), __hmul2(point[j], database[i * HALF2_DIM + i]));
				resultDistance += __low2float(tmpResult) + __high2float(tmpResult);
			}
		#endif

		#if DTYPE_PREC == 16
		if (hsqrt(resultDistance) <= (*epsilon))
		#else
		if (sqrt(resultDistance) <= (*epsilon))
		#endif
		{
			atomicAdd(nbNeighbors, int(1));
		}
	}
}



__global__ void distanceCalculationBruteForceTensor_TwoStepsComputePagingOneQuery(
	half* dataset,
	unsigned int* nbQueries,
	half* identity,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	__shared__ half sharedArrayQueryPoint[WARP_PER_BLOCK * COMPUTE_DIM];
	__shared__ half sharedArrayResultFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];

	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int warpId = tid / WARP_SIZE;

	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());

	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> identityFragment;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half> firstStepAccumulator;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;

	wmma::load_matrix_sync(identityFragment, identity, TILE_SIZE_HALF);

	for (int i = 0; i < POINTS_PER_WARP; ++i)
	{
		unsigned int nbDimsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
		for (int j = 0; j < nbDimsToPage; ++j)
		{
			if ((warp.thread_rank() * nbDimsToPage + j) < COMPUTE_DIM)
			{
				sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j] =
					dataset[(tid / WARP_SIZE) * POINTS_PER_WARP * COMPUTE_DIM + i * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j];
			}
		}

		for (int j = 0; j < (*nbQueries); j += TILE_SIZE_HALF)
		{
			wmma::fill_fragment(secondStepAccumulator, 0.0);

			for (int k = 0; k < COMPUTE_DIM; k += TILE_SIZE_HALF)
			{
				wmma::load_matrix_sync(matrixAFragment, sharedArrayQueryPoint + (warpIdInBlock * COMPUTE_DIM + k), 0);

				wmma::load_matrix_sync(firstStepAccumulator, dataset + j * COMPUTE_DIM + k, COMPUTE_DIM, wmma::mem_row_major);
				for (int l = 0; l < firstStepAccumulator.num_elements; ++l)
				{
					firstStepAccumulator.x[l] = (half)-1.0 * firstStepAccumulator.x[l];
				}

				wmma::mma_sync(firstStepAccumulator, matrixAFragment, identityFragment, firstStepAccumulator);

				wmma::store_matrix_sync(sharedArrayResultFirstStep + sharedArrayResultOffset, firstStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);
				wmma::load_matrix_sync(matrixAFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
				wmma::load_matrix_sync(matrixBFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);

				wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);
			}

			wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

			if (warp.thread_rank() < TILE_SIZE_HALF)
			{
				DTYPE resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];

				#if DTYPE_PREC == 16
				if(hsqrt(resultDistance) <= (*epsilon))
				#else
				if(sqrt(resultDistance) <= (*epsilon))
				#endif
				{
					atomicAdd(nbNeighbors, int(1));
				}
			}
			warp.sync();
		}
	}
}



__global__ void distanceCalculationBruteForceTensor_TwoStepsComputePagingOneQueryOptim(
	half* dataset,
	unsigned int* nbQueries,
	half* identity,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	__shared__ half sharedArrayQueryPoint[WARP_PER_BLOCK * COMPUTE_DIM];
	__shared__ half sharedArrayResultFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];

	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int warpId = tid / WARP_SIZE;

	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());

	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> identityFragment;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half> firstStepAccumulator;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;

	wmma::load_matrix_sync(identityFragment, identity, TILE_SIZE_HALF);

	for (int i = 0; i < POINTS_PER_WARP; ++i)
	{
		unsigned int nbStepsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
		for (int j = 0; j < nbStepsToPage; ++j)
		{
			if ((j * WARP_SIZE + warp.thread_rank()) < COMPUTE_DIM)
			{
				sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()] =
					dataset[(tid / WARP_SIZE) * POINTS_PER_WARP * COMPUTE_DIM + i * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()];
			}
		}

		for (int j = 0; j < (*nbQueries); j += TILE_SIZE_HALF)
		{
			wmma::fill_fragment(secondStepAccumulator, 0.0);

			for (int k = 0; k < COMPUTE_DIM; k += TILE_SIZE_HALF)
			{
				wmma::load_matrix_sync(matrixAFragment, sharedArrayQueryPoint + (warpIdInBlock * COMPUTE_DIM + k), 0);

				wmma::load_matrix_sync(firstStepAccumulator, dataset + j * COMPUTE_DIM + k, COMPUTE_DIM, wmma::mem_row_major);
				for (int l = 0; l < firstStepAccumulator.num_elements; ++l)
				{
					firstStepAccumulator.x[l] = (half)-1.0 * firstStepAccumulator.x[l];
				}

				wmma::mma_sync(firstStepAccumulator, matrixAFragment, identityFragment, firstStepAccumulator);

				wmma::store_matrix_sync(sharedArrayResultFirstStep + sharedArrayResultOffset, firstStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);
				wmma::load_matrix_sync(matrixAFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
				wmma::load_matrix_sync(matrixBFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);

				wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);
			}

			wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

			if (warp.thread_rank() < TILE_SIZE_HALF)
			{
				DTYPE resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];

				#if DTYPE_PREC == 16
				if(hsqrt(resultDistance) <= (*epsilon))
				#else
				if(sqrt(resultDistance) <= (*epsilon))
				#endif
				{
					atomicAdd(nbNeighbors, int(1));
				}
			}
			warp.sync();
		}
	}
}



__global__ void distanceCalculationBruteForceTensor_OneStepComputePagingOneQuery(
	half* dataset,
	unsigned int* nbQueries,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	__shared__ half sharedArrayQueryPoint[WARP_PER_BLOCK * COMPUTE_DIM];
	__shared__ half sharedArrayTmpFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];

	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int warpId = tid / WARP_SIZE;

	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());

	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;

	for (int i = 0; i < POINTS_PER_WARP; ++i)
	{
		unsigned int nbDimsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
		for (int j = 0; j < nbDimsToPage; ++j)
		{
			if ((warp.thread_rank() * nbDimsToPage + j) < COMPUTE_DIM)
			{
				sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j] =
					dataset[(tid / WARP_SIZE) * POINTS_PER_WARP * COMPUTE_DIM + i * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j];
			}
		}

		for (int j = 0; j < (*nbQueries); j += TILE_SIZE_HALF)
		{
			wmma::fill_fragment(secondStepAccumulator, 0.0);

			for (int k = 0; k < COMPUTE_DIM; k += TILE_SIZE_HALF)
			{
				unsigned int nbElemsToPage = (TILE_SIZE_HALF * TILE_SIZE_HALF) / WARP_SIZE;
				unsigned int threadPerPoint = WARP_SIZE / TILE_SIZE_HALF;
				for (int l = 0; l < nbElemsToPage; ++l)
				{
					sharedArrayTmpFirstStep[sharedArrayResultOffset + warp.thread_rank() * nbElemsToPage + l] =
						sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + k + (warp.thread_rank() / threadPerPoint) * nbElemsToPage + l]
						- dataset[j * COMPUTE_DIM + k + (warp.thread_rank() / threadPerPoint) * COMPUTE_DIM + (warp.thread_rank() / threadPerPoint) * nbElemsToPage + l];
				}

				wmma::load_matrix_sync(matrixAFragment, sharedArrayTmpFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
				wmma::load_matrix_sync(matrixBFragment, sharedArrayTmpFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);

				wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);
			}

			wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

			if (warp.thread_rank() < TILE_SIZE_HALF)
			{
				DTYPE resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];

				#if DTYPE_PREC == 16
				if(hsqrt(resultDistance) <= (*epsilon))
				#else
				if(sqrt(resultDistance) <= (*epsilon))
				#endif
				{
					atomicAdd(nbNeighbors, int(1));
				}
			}
			warp.sync();
		}
	}
}



__global__ void distanceCalculationBruteForceTensor_OneStepComputePagingOneQueryOptim(
	half* dataset,
	unsigned int* nbQueries,
	DTYPE* epsilon,
	unsigned int* nbNeighbors)
{
	__shared__ half sharedArrayQueryPoint[WARP_PER_BLOCK * COMPUTE_DIM];
	__shared__ half sharedArrayTmpFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];

	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// unsigned int warpId = tid / WARP_SIZE;

	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());

	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;

	for (int i = 0; i < POINTS_PER_WARP; ++i)
	{
		unsigned int nbStepsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
		for (int j = 0; j < nbStepsToPage; ++j)
		{
			if ((j * WARP_SIZE + warp.thread_rank()) < COMPUTE_DIM)
			{
				sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()] =
					dataset[(tid / WARP_SIZE) * POINTS_PER_WARP * COMPUTE_DIM + i * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()];
			}
		}

		for (int j = 0; j < (*nbQueries); j += TILE_SIZE_HALF)
		{
			wmma::fill_fragment(secondStepAccumulator, 0.0);

			for (int k = 0; k < COMPUTE_DIM; k += TILE_SIZE_HALF)
			{
				unsigned int nbStepsOfPage = (TILE_SIZE_HALF * TILE_SIZE_HALF) / WARP_SIZE;
				for (int l = 0; l < nbStepsOfPage; ++l)
				{
					sharedArrayTmpFirstStep[sharedArrayResultOffset + l * WARP_SIZE + warp.thread_rank()] =
						sharedArrayQueryPoint[warpIdInBlock * COMPUTE_DIM + k + (warp.thread_rank() % TILE_SIZE_HALF)]
						- dataset[j * COMPUTE_DIM + k + 2 * l * COMPUTE_DIM + (warp.thread_rank() / TILE_SIZE_HALF) * COMPUTE_DIM + warp.thread_rank()];
				}

				wmma::load_matrix_sync(matrixAFragment, sharedArrayTmpFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
				wmma::load_matrix_sync(matrixBFragment, sharedArrayTmpFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);

				wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);
			}

			wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

			if (warp.thread_rank() < TILE_SIZE_HALF)
			{
				DTYPE resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];

				#if DTYPE_PREC == 16
				if(hsqrt(resultDistance) <= (*epsilon))
				#else
				if(sqrt(resultDistance) <= (*epsilon))
				#endif
				{
					atomicAdd(nbNeighbors, int(1));
				}
			}
			warp.sync();
		}
	}
}



//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\//\\



__global__ void distanceCalculationGridCudaHalf(
    unsigned int* batchBegin,
    unsigned int* batchSize,
    half* database,
    unsigned int* originPointIndex,
    DTYPE* epsilon,
    struct grid* grid,
    unsigned int* gridLookupArr,
    struct gridCellLookup* gridCellLookupArr,
    half* minArr,
    unsigned int* nCells,
    unsigned int* cnt,
    unsigned int* nNonEmptyCells,
    int* pointIDKey,
    int* pointInDistVal)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ((*batchSize) <= tid)
    {
        return;
    }

    // Get the next query point in the "local" queue
    unsigned int pointId = atomicAdd(batchBegin, int(1));

    half point[GPUNUMDIM];
    for (int i = 0; i < GPUNUMDIM; ++i)
    {
        point[i] = database[ originPointIndex[pointId] * COMPUTE_DIM + i ];
    }

    // Calculate the coords of the Cell for the point and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
    unsigned int nDMinCellIDs[NUMINDEXEDDIM];
	unsigned int nDMaxCellIDs[NUMINDEXEDDIM];

    for (int i = 0; i < NUMINDEXEDDIM; ++i)
    {
        nDCellIDs[i] = (DTYPE)(point[i] - minArr[i]) / (*epsilon);
		nDMinCellIDs[i] = max(0, nDCellIDs[i] - 1); // Boundary conditions (don't go beyond cell 0)
		nDMaxCellIDs[i] = min(nCells[i] - 1, nDCellIDs[i] + 1); // Boundary conditions (don't go beyond the maximum number of cells)
    }

    unsigned int indexes[NUMINDEXEDDIM];
    unsigned int loopRng[NUMINDEXEDDIM];

    for (loopRng[0] = nDMinCellIDs[0]; loopRng[0] <= nDMaxCellIDs[0]; loopRng[0]++)
		for (loopRng[1] = nDMinCellIDs[1]; loopRng[1] <= nDMaxCellIDs[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; x++)
			{
				indexes[x] = loopRng[x];
			}

			evaluateCell_alt(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, grid,
					gridLookupArr, point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], nDCellIDs);

		} //end loop body
}



__global__ void distanceCalculationGridCudaHalf2(
    unsigned int* batchBegin,
    unsigned int* batchSize,
    half2* database,
    unsigned int* originPointIndex,
    DTYPE* epsilon,
    struct grid* grid,
    unsigned int* gridLookupArr,
    struct gridCellLookup* gridCellLookupArr,
    half* minArr,
    unsigned int* nCells,
    unsigned int* cnt,
    unsigned int* nNonEmptyCells,
    int* pointIDKey,
    int* pointInDistVal)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ((*batchSize) <= tid)
    {
        return;
    }

    // Get the next query point in the "local" queue
    unsigned int pointId = atomicAdd(batchBegin, int(1));

    half2 point[HALF2_DIM];
    for (int i = 0; i < HALF2_DIM; ++i)
    {
        point[i] = database[ originPointIndex[pointId] * HALF2_DIM + i ];
    }

    // Calculate the coords of the Cell for the point and the min/max ranges in each dimension
	unsigned int nDCellIDs[NUMINDEXEDDIM];
    unsigned int nDMinCellIDs[NUMINDEXEDDIM];
	unsigned int nDMaxCellIDs[NUMINDEXEDDIM];

    for (int i = 0; i < NUMINDEXEDDIM; ++i)
    {
		if (0 == (i % 2))
		{
			nDCellIDs[i] = (DTYPE)(__low2half(point[i / 2]) - minArr[i]) / (*epsilon);
		} else {
			nDCellIDs[i] = (DTYPE)(__high2half(point[i / 2]) - minArr[i]) / (*epsilon);
		}
		nDMinCellIDs[i] = max(0, nDCellIDs[i] - 1); // Boundary conditions (don't go beyond cell 0)
		nDMaxCellIDs[i] = min(nCells[i] - 1, nDCellIDs[i] + 1); // Boundary conditions (don't go beyond the maximum number of cells)
    }

    unsigned int indexes[NUMINDEXEDDIM];
    unsigned int loopRng[NUMINDEXEDDIM];

    for (loopRng[0] = nDMinCellIDs[0]; loopRng[0] <= nDMaxCellIDs[0]; loopRng[0]++)
		for (loopRng[1] = nDMinCellIDs[1]; loopRng[1] <= nDMaxCellIDs[1]; loopRng[1]++)
		#include "kernelloops.h"
		{ //beginning of loop body

			for (int x = 0; x < NUMINDEXEDDIM; x++)
			{
				indexes[x] = loopRng[x];
			}

			evaluateCellHalf2(nCells, indexes, gridCellLookupArr, nNonEmptyCells, database, epsilon, grid,
					gridLookupArr, point, cnt, pointIDKey, pointInDistVal, originPointIndex[pointId], nDCellIDs);

		} //end loop body
}



__global__ void distanceCalculationGridTensor_TwoStepsComputePagingOneQuery(
	unsigned int* batchBegin,
	unsigned int* batchEnd,
	half* database,
	unsigned int* nbQueryPoints,
	unsigned int* originPointIndex,
	half* identityMatrix,
	DTYPE* epsilon,
	struct grid* grid,
	unsigned int* gridLookupArr,
	struct gridCellLookup* gridCellLookupArr,
	half* minArr,
	unsigned int* nCells,
	unsigned int* cnt,
	unsigned int* nNonEmptyCells,
	int* pointIDKey,
	int* pointInDistVal)
{
	__shared__ half sharedArrayQueryPoints[WARP_PER_BLOCK * COMPUTE_DIM];
	__shared__ half sharedArrayResultFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];

	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());

	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> identityFragment;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half> firstStepAccumulator;
	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;

	wmma::load_matrix_sync(identityFragment, identityMatrix, TILE_SIZE_HALF);

	unsigned int nDCellIDs[NUMINDEXEDDIM];
	unsigned int nDMinCellIDs[NUMINDEXEDDIM];
	unsigned int nDMaxCellIDs[NUMINDEXEDDIM];
	unsigned int indexes[NUMINDEXEDDIM];
	unsigned int loopRng[NUMINDEXEDDIM];

	unsigned int firstQueryId;
	if (0 == warp.thread_rank())
	{
		firstQueryId = atomicAdd(batchBegin, int(POINTS_PER_WARP));
	}
	firstQueryId = __shfl_sync(0xffffffff, firstQueryId, 0);

	for (int i = firstQueryId; i < (firstQueryId + POINTS_PER_WARP) && i < (*batchEnd); ++i)
	{
		unsigned int nbStepsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
		for (int j = 0; j < nbStepsToPage; ++j)
		{
			if ((j * WARP_SIZE + warp.thread_rank()) < COMPUTE_DIM)
			{
				sharedArrayQueryPoints[warpIdInBlock * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()] =
					database[originPointIndex[i] * COMPUTE_DIM + j * WARP_SIZE + warp.thread_rank()];
			}
		}

		for (int j = 0; j < NUMINDEXEDDIM; ++j)
		{
			nDCellIDs[j] = (sharedArrayQueryPoints[warpIdInBlock * COMPUTE_DIM + j] - minArr[j]) / (half)(*epsilon);
			nDMinCellIDs[j] = max(0, nDCellIDs[j] - 1);
			nDMaxCellIDs[j] = min(nCells[j] - 1, nDCellIDs[j] + 1);
		}

		DTYPE resultDistance = 0.0;

		for (loopRng[0] = nDMinCellIDs[0]; loopRng[0] <= nDMaxCellIDs[0]; loopRng[0]++)
			for (loopRng[1] = nDMinCellIDs[1]; loopRng[1] <= nDMaxCellIDs[1]; loopRng[1]++)
			#include "kernelloops.h"
			{ //beginning of loop body
				for (int x = 0; x < NUMINDEXEDDIM; ++x)
				{
					indexes[x] = loopRng[x];
				}

				uint64_t cellLinearId = getLinearID_nDimensionsGPUKernelAlt(indexes, nCells, NUMINDEXEDDIM);
				struct gridCellLookup tmp;
				tmp.gridLinearID = cellLinearId;

				// Find if the neighboring cell is empty or not
				if(thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
				{
					struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr,
						gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
					unsigned int gridIndex = resultBinSearch->idx;

					// For all the candidates in the cell, process them 16 by 16
					for (int k = grid[gridIndex].indexmin; k <= grid[gridIndex].indexmax; k += TILE_SIZE_HALF)
					{
						unsigned int nbCandidatesLeft = grid[gridIndex].indexmax - k + 1;

						wmma::fill_fragment(secondStepAccumulator, 0.0f);

						// For all dimensions of the candidates, process them 16 by 16
						for (int n = 0; n < COMPUTE_DIM; n += TILE_SIZE_HALF)
						{
							wmma::load_matrix_sync(matrixAFragment, sharedArrayQueryPoints + warpIdInBlock * COMPUTE_DIM + n, 0);

							// unsigned int candidateId;
							// thread_block_tile<16> halfWarp = tiled_partition<16>(warp);
							unsigned int halfWarpId = warp.thread_rank() / 16;
							unsigned int halfWarpThreadId = warp.thread_rank() % 16;

							for (int j = 0; j < TILE_SIZE_HALF; j += 2)
							{
								unsigned int candidateId;
								// if ((k + j + halfWarp.meta_group_rank()) < (*nbQueryPoints))
								// {
								// 	candidateId = gridLookupArr[k + j + halfWarp.meta_group_rank()];
								// } else {
								// 	candidateId = 0;
								// }
								if ((k + j + halfWarpId) < (*nbQueryPoints))
								{
									candidateId = gridLookupArr[k + j + halfWarpId];
								} else {
									candidateId = 0;
								}

								sharedArrayResultFirstStep[sharedArrayResultOffset + (j + halfWarpId) * TILE_SIZE_HALF + halfWarpThreadId] =
									database[candidateId * COMPUTE_DIM + n + halfWarpThreadId];
							}

							wmma::load_matrix_sync(firstStepAccumulator, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF, wmma::mem_row_major);
							for (int j = 0; j < firstStepAccumulator.num_elements; ++j)
							{
								firstStepAccumulator.x[j] = (half)-1.0 * firstStepAccumulator.x[j];
							}

							wmma::mma_sync(firstStepAccumulator, matrixAFragment, identityFragment, firstStepAccumulator);
							wmma::store_matrix_sync(sharedArrayResultFirstStep + sharedArrayResultOffset, firstStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

							wmma::load_matrix_sync(matrixAFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
							wmma::load_matrix_sync(matrixBFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);

							wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);

							#if SHORT_CIRCUIT
								wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

								int nbThreadsShortCircuit = 0;
								if (warp.thread_rank() < TILE_SIZE_HALF && warp.thread_rank() < nbCandidatesLeft)
								{
									resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];

									int shortCircuit = 0;
									#if DTYPE_PREC == 16
									if (hsqrt(resultDistance) > (*epsilon))
									#else
									if (sqrt(resultDistance) > (*epsilon))
									#endif
									{
										shortCircuit = 1;
									}

									// Match if all 16 candidate points short-circuited
									__match_all_sync(__activemask(), shortCircuit, &nbThreadsShortCircuit);
								}

								// Get from thread 0 if the threads that computed the distances short-circuited
								nbThreadsShortCircuit = __shfl_sync(0xffffffff, nbThreadsShortCircuit, 0);
								if (nbThreadsShortCircuit)
								{
									// Break the loop iterating over the dimensions of the current candidates
									n = COMPUTE_DIM;
								}
							#endif
						} // for COMPUTE_DIM

						#if SHORT_CIRCUIT
							if (warp.thread_rank() < TILE_SIZE_HALF && warp.thread_rank() < nbCandidatesLeft)
							{
								// The distance was already computed on the last short-circuit check
								#if DTYPE_PREC == 16
								if (hsqrt(resultDistance) <= (*epsilon))
								#else
								if (sqrt(resultDistance) <= (*epsilon))
								#endif
								{
									unsigned int tmpIdx = atomicAdd(cnt, int(1));
									pointIDKey[tmpIdx] = originPointIndex[i];
									pointInDistVal[tmpIdx] = gridLookupArr[k];
								}
							}
						#else
							wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);

							if (warp.thread_rank() < TILE_SIZE_HALF && warp.thread_rank() < nbCandidatesLeft)
							{
								// resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];
								resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank()];

								#if DTYPE_PREC == 16
								if(hsqrt(resultDistance) <= (*epsilon))
								#else
								if(sqrt(resultDistance) <= (*epsilon))
								#endif
								{
									unsigned int tmpIdx = atomicAdd(cnt, int(1));
									pointIDKey[tmpIdx] = originPointIndex[i];
									pointInDistVal[tmpIdx] = gridLookupArr[k];
								}
							}
							warp.sync();
						#endif
					} // for candidates
				} // if non-empty cell
			} // for neighboring cells
	} // for query points
}



// __global__ void distanceCalculationGridTensor_OneStepComputePagingOneQuery(
// 	unsigned int* batchBegin,
// 	unsigned int* batchSize,
// 	half* database,
// 	unsigned int* originPointIndex,
// 	DTYPE* epsilon,
// 	struct grid* grid,
// 	unsigned int* gridLookupArr,
// 	struct gridCellLookup* gridCellLookupArr,
// 	half* minArr,
// 	unsigned int* nCells,
// 	unsigned int* cnt,
// 	unsigned int* nNonEmptyCells,
// 	int* pointIDKey,
// 	int* pointInDistVal)
// {
// 	__shared__ half sharedArrayQueryPoints[WARP_PER_BLOCK * COMPUTE_DIM];
// 	__shared__ half sharedArrayResultFirstStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
// 	__shared__ DTYPE sharedArrayResultSecondStep[WARP_PER_BLOCK * TILE_SIZE_HALF * TILE_SIZE_HALF];
//
// 	unsigned int warpIdInBlock = threadIdx.x / WARP_SIZE;
// 	unsigned int sharedArrayResultOffset = warpIdInBlock * TILE_SIZE_HALF * TILE_SIZE_HALF;
// 	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
//
// 	thread_block_tile<WARP_SIZE> warp = tiled_partition<WARP_SIZE>(this_thread_block());
//
// 	wmma::fragment<wmma::matrix_a, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::row_major> matrixAFragment;
// 	wmma::fragment<wmma::matrix_b, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, half, wmma::col_major> matrixBFragment;
// 	wmma::fragment<wmma::accumulator, TILE_SIZE_HALF, TILE_SIZE_HALF, TILE_SIZE_HALF, DTYPE> secondStepAccumulator;
//
// 	unsigned int nDCellIDs[NUMINDEXEDDIM];
// 	unsigned int nDMinCellIDs[NUMINDEXEDDIM];
// 	unsigned int nDMaxCellIDs[NUMINDEXEDDIM];
// 	unsigned int indexes[NUMINDEXEDDIM];
// 	unsigned int loopRng[NUMINDEXEDDIM];
//
// 	unsigned int firstQueryId;
// 	if (0 == warp.thread_rank())
// 	{
// 		firstQueryId = atomicAdd(batchBegin, int(POINTS_PER_WARP));
// 	}
// 	firstQueryId = __shfl_sync(0xffffffff, firstQueryId, 0);
//
// 	for (int i = firstQueryId; i < (firstQueryId + POINTS_PER_WARP); ++i)
// 	{
// 		unsigned int nbDimsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
// 		for (int j = 0; j < nbDimsToPage; ++j)
// 		{
// 			if ((warp.thread_rank() * nbDimsToPage + j) < COMPUTE_DIM)
// 			{
// 				sharedArrayQueryPoints[warpIdInBlock * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j] =
// 					database[ originPointIndex[i] * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j];
// 			}
// 		}
//
// 		for (int j = 0; j < NUMINDEXEDDIM; ++j)
// 		{
// 			nDCellIDs[j] = (sharedArrayQueryPoints[warpIdInBlock * COMPUTE_DIM + j] - minArr[j]) / (half)(*epsilon);
// 			nDMinCellIDs[j] = max(0, nDCellIDs[j] - 1);
// 			nDMaxCellIDs[j] = min(nCells[j] - 1, nDCellIDs[j] + 1);
// 		}
//
// 		DTYPE resultDistance = 0.0;
//
// 		for (loopRng[0] = nDMinCellIDs[0]; loopRng[0] <= nDMaxCellIDs[0]; loopRng[0]++)
// 			for (loopRng[1] = nDMinCellIDs[1]; loopRng[1] <= nDMaxCellIDs[1]; loopRng[1]++)
// 			#include "kernelloops.h"
// 			{ //beginning of loop body
// 				for (int x = 0; x < NUMINDEXEDDIM; ++x)
// 				{
// 					indexes[x] = loopRng[x];
// 				}
//
// 				uint64_t cellLinearId = getLinearID_nDimensionsGPUKernelAlt(indexes, nCells, NUMINDEXEDDIM);
// 				struct gridCellLookup tmp;
// 				tmp.gridLinearID = cellLinearId;
//
// 				// Find if the neighboring cell is empty or not
// 				if(thrust::binary_search(thrust::seq, gridCellLookupArr, gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp)))
// 				{
// 					struct gridCellLookup * resultBinSearch = thrust::lower_bound(thrust::seq, gridCellLookupArr,
// 						gridCellLookupArr + (*nNonEmptyCells), gridCellLookup(tmp));
// 					unsigned int gridIndex = resultBinSearch->idx;
//
// 					for (int k = grid[gridIndex].indexmin; k <= grid[gridIndex].indexmax; k += TILE_SIZE_HALF)
// 					{
// 						unsigned int nbCandidatesLeft = grid[gridIndex].indexmax - k + 1;
//
// 						wmma::fill_fragment(secondStepAccumulator, 0.0f);
//
// 						unsigned int candidateId = gridLookupArr[k + (warp.thread_rank() / 2)];
//
// 						for (int n = 0; n < COMPUTE_DIM; n += TILE_SIZE_HALF)
// 						{
// 							wmma::load_matrix_sync(matrixAFragment, sharedArrayQueryPoints + (warpIdInBlock * COMPUTE_DIM + k), 0);
//
// 							// unsigned int nbDimsToPage = ceil((1.0 * COMPUTE_DIM) / (1.0 * WARP_SIZE));
// 							for (int j = 0; j < nbDimsToPage; ++j)
// 							{
// 								if ((warp.thread_rank() * nbDimsToPage + j) < COMPUTE_DIM)
// 								{
// 									sharedArrayResultFirstStep[sharedArrayResultOffset + warp.thread_rank() * nbDimsToPage + j] =
// 										database[candidateId * COMPUTE_DIM + warp.thread_rank() * nbDimsToPage + j] * (half)-1.0;
// 								}
// 							}
// 							wmma::load_matrix_sync(matrixBFragment, sharedArrayResultFirstStep + sharedArrayResultOffset, TILE_SIZE_HALF);
//
// 							wmma::mma_sync(secondStepAccumulator, matrixAFragment, matrixBFragment, secondStepAccumulator);
// 						}
//
// 						wmma::store_matrix_sync(sharedArrayResultSecondStep + sharedArrayResultOffset, secondStepAccumulator, TILE_SIZE_HALF, wmma::mem_row_major);
//
// 						if (warp.thread_rank() < TILE_SIZE_HALF)
// 						{
// 							resultDistance = sharedArrayResultSecondStep[sharedArrayResultOffset + warp.thread_rank() * TILE_SIZE_HALF + warp.thread_rank()];
//
// 							#if DTYPE_PREC == 16
// 							if(hsqrt(resultDistance) <= (*epsilon))
// 							#else
// 							if(sqrt(resultDistance) <= (*epsilon))
// 							#endif
// 							{
// 								unsigned int tmpIdx = atomicAdd(cnt, int(1));
// 								pointIDKey[tmpIdx] = originPointIndex[i];
// 								pointInDistVal[tmpIdx] = candidateId;
// 							}
// 						}
// 						warp.sync();
// 					}
// 				}
// 			}
// 	}
// }
