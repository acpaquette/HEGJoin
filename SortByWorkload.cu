#include "hip/hip_runtime.h"
#include "SortByWorkload.h"
#include "structs.h"
#include "params.h"
#include "kernel.h"
#include "WorkQueue.h"

#include <iostream>
#include <algorithm>

#include "omp.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

void sortByWorkLoad(
        unsigned int searchMode,
        unsigned int * DBSIZE,
        DTYPE * epsilon,
        DTYPE ** dev_epsilon,
        DTYPE * database,
        DTYPE ** dev_database,
        struct grid * index,
        struct grid ** dev_index,
        unsigned int * indexLookupArr,
        unsigned int ** dev_indexLookupArr,
        struct gridCellLookup * gridCellLookupArr,
        struct gridCellLookup ** dev_gridCellLookupArr,
        DTYPE * minArr,
        DTYPE ** dev_minArr,
        unsigned int * nCells,
        unsigned int ** dev_nCells,
        unsigned int * nNonEmptyCells,
        unsigned int ** dev_nNonEmptyCells,
        unsigned int ** originPointIndex,
        unsigned int ** dev_originPointIndex)
        // bool * isSortByWLDone,
        // unsigned int * nbPointsPreComputed,
        // CPU_State * cpuState)
{

    double tStartSortingCells = omp_get_wtime();

    hipError_t errCode;

    struct schedulingCell * sortedDatabaseTmp = new schedulingCell[sizeof(struct schedulingCell) * (*nNonEmptyCells)];
    struct schedulingCell * dev_sortedDatabaseTmp;

    hipEvent_t startKernel, endKernel;
    hipEventCreate(&startKernel);
    hipEventCreate(&endKernel);

    errCode = hipMalloc((void**)&dev_sortedDatabaseTmp, sizeof(struct schedulingCell) * (*nNonEmptyCells));
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: Alloc sortedSet -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    // Beginning of the sorting section
    int nbBlock = ((*nNonEmptyCells) / BLOCKSIZE) + 1;
    cout << "[SORT] ~ Using " << nbBlock << " blocks of " << BLOCKSIZE << " threads to sort on the GPU\n";
    cout.flush();

    hipEventRecord(startKernel);
    #if UNICOMP
        sortByWorkLoadUnicomp<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), dev_sortedDatabaseTmp);
    #elif LID_UNICOMP
        sortByWorkLoadLidUnicomp<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), dev_sortedDatabaseTmp);
    #else
        cout << "[SORT] ~ Not using a cell access pattern to sort by workload\n";
        sortByWorkLoadGlobal<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), dev_sortedDatabaseTmp);
    #endif
    hipEventRecord(endKernel);

    errCode = hipGetLastError();
    cout << "[SORT] ~ SORTING KERNEL LAUNCH RETURN: " << errCode << '\n';
    cout.flush();

    hipDeviceSynchronize();

    errCode = hipMemcpy(sortedDatabaseTmp, dev_sortedDatabaseTmp, sizeof(struct schedulingCell) * (*nNonEmptyCells), hipMemcpyDeviceToHost);
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: copy sorted cells from the GPU -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    #pragma omp critical
    {
        (*isSortByWLDone) = true;
    }

    hipEventSynchronize(endKernel);
    float timeKernel = 0;
    hipEventElapsedTime(&timeKernel, startKernel, endKernel);
    cout << "[SORT] ~ Kernel time to sort by workload: " << timeKernel << '\n';
    cout.flush();

    double tBeginSort = omp_get_wtime();
    std::sort(sortedDatabaseTmp, sortedDatabaseTmp + (*nNonEmptyCells),
            [](const schedulingCell& a, const schedulingCell& b){ return a.nbPoints > b.nbPoints; });
    double tEndSort = omp_get_wtime();
    printf("[SORT] ~ Time to call std::sort = %f\n", tEndSort - tBeginSort);

    unsigned int maxNeighbor = sortedDatabaseTmp[0].nbPoints;
    unsigned int minNeighbor = sortedDatabaseTmp[(*nNonEmptyCells) - 1].nbPoints;
    // cout << "max = " << maxNeighbor << '\n';
    // cout << "min = " << minNeighbor << '\n';
    uint64_t accNeighbor = 0;

    // unsigned int * nbNeighborPoints = new unsigned int[(*DBSIZE)];

    // unsigned int nbQueriesPreComputed;
    // bool cpuComputing = true;
    // #pragma omp critical
    // {
    //     if((*cpuState) < CPU_State::computing)
    //     {
    //         nbQueriesPreComputed = 0;
    //         cpuComputing = false;
    //     }
    // }
    //
    // while(cpuComputing)
    // {
    //     #pragma omp critical
    //     {
    //         cpuComputing = (CPU_State::computing == (*cpuState));
    //     }
    // }
    //
    //     // while((*cpuState) != CPU_State::doneComputing){}
    // #pragma omp critical
    // {
    //     nbQueriesPreComputed = (*nbPointsPreComputed);
    // }

    // (*originPointIndex) = new unsigned int [(*DBSIZE) - nbQueriesPreComputed];
    (*originPointIndex) = new unsigned int [(*DBSIZE)];

    int prec = 0;
    for(int i = 0; i < (*nNonEmptyCells); ++i)
    {
        int cellId = sortedDatabaseTmp[i].cellId;
        int nbNeighbor = index[cellId].indexmax - index[cellId].indexmin + 1;
        int nbPointsSkipped = 0;

        accNeighbor += (nbNeighbor * sortedDatabaseTmp[i].nbPoints);

        for(int j = 0; j < nbNeighbor; ++j)
        {
            int tmpId = indexLookupArr[ index[cellId].indexmin + j ];
            (*originPointIndex)[prec + j] = tmpId;
            // if(nbQueriesPreComputed < tmpId)
            // {
            //     (*originPointIndex)[prec + j] = tmpId;
            // }
            // else{
            //     nbPointsSkipped++;
            // }
        }
        prec += nbNeighbor;
        // prec += (nbNeighbor - nbPointsSkipped);
    }

    // Setting some stuff for the CPU so it can begin immediately
    // and allocate buffers to store temp results
    // However, this value is way overestimated as it is set to the max,
    // while the CPU computes from the end of the queue.
    setMaxNeighbors(maxNeighbor);
    setWorkQueueReady();

    errCode = hipMalloc((void**)dev_originPointIndex, ((*DBSIZE) - nbQueriesPreComputed) * sizeof(unsigned int));
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: Alloc point index -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    errCode = hipMemcpy( (*dev_originPointIndex), (*originPointIndex), ((*DBSIZE) - nbQueriesPreComputed) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: point index copy -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    unsigned int decileMark = (*nNonEmptyCells) / 10;
    cout << "[SORT] ~ Total number of candidate points to refine: " << accNeighbor << '\n';
    cout << "[SORT] ~ Number of candidates: min = " << minNeighbor << ", median = " << sortedDatabaseTmp[(*nNonEmptyCells) / 2].nbPoints << ", max = " << maxNeighbor << ", avg = " << accNeighbor / (*DBSIZE) << '\n';
    cout << "[SORT] ~ Deciles number of candidates: \n";
    for(int i = 1; i < 10; ++i)
    {
        cout << "   [SORT] ~ " << i * 10 << "% = " << sortedDatabaseTmp[decileMark * i].nbPoints << '\n';
    }
    cout.flush();

    hipFree(dev_sortedDatabaseTmp);

    delete[] sortedDatabaseTmp;
    // delete[] nbNeighborPoints;

    double tEndSortingCells = omp_get_wtime();

    cout << "[SORT] ~ Time to sort the cells by workload and copy to the GPU: " << tEndSortingCells - tStartSortingCells << '\n';
    cout.flush();

}
