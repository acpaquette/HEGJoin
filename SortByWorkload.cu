#include "hip/hip_runtime.h"
#include "SortByWorkload.h"
#include "structs.h"
#include "params.h"
#include "kernel.h"
#include "WorkQueue.h"

#include <iostream>
#include <algorithm>

#include "omp.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using std::cout;
using std::endl;

void sortByWorkLoad(
        unsigned int searchMode,
        unsigned int * DBSIZE,
        DTYPE * epsilon,
        DTYPE ** dev_epsilon,
        DTYPE * database,
        DTYPE ** dev_database,
        struct grid * index,
        struct grid ** dev_index,
        unsigned int * indexLookupArr,
        unsigned int ** dev_indexLookupArr,
        struct gridCellLookup * gridCellLookupArr,
        struct gridCellLookup ** dev_gridCellLookupArr,
        DTYPE * minArr,
        DTYPE ** dev_minArr,
        unsigned int * nCells,
        unsigned int ** dev_nCells,
        unsigned int * nNonEmptyCells,
        unsigned int ** dev_nNonEmptyCells,
        // unsigned int * gridCellNDMask,
        // unsigned int ** dev_gridCellNDMask,
        // unsigned int * gridCellNDMaskOffsets,
        // unsigned int ** dev_gridCellNDMaskOffsets,
        // unsigned int * nNDMaskElems,
        unsigned int ** originPointIndex,
        unsigned int ** dev_originPointIndex,
        DTYPE ** dev_sortedDatabase)
{

    hipError_t errCode;

    struct schedulingCell * sortedDatabaseTmp = new schedulingCell[sizeof(struct schedulingCell) * (*nNonEmptyCells)];
    struct schedulingCell * dev_sortedDatabaseTmp;

    hipEvent_t startKernel, endKernel;
    hipEventCreate(&startKernel);
    hipEventCreate(&endKernel);

    // Memory allocations needed by the GPU

    double tStartAllocGPU = omp_get_wtime();

    // errCode = hipMalloc( (void**)dev_epsilon, sizeof(DTYPE));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc epsilon -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc( (void**)dev_database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc database -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc( (void**)dev_index, sizeof(struct grid) * (*nNonEmptyCells));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc grid index -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc( (void**)dev_indexLookupArr, sizeof(unsigned int) * (*DBSIZE));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: lookup array allocation -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc( (void**)dev_gridCellLookupArr, sizeof(struct gridCellLookup) * (*nNonEmptyCells));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: copy grid cell lookup array allocation -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc((void**)dev_minArr, sizeof(DTYPE) * (NUMINDEXEDDIM));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc minArr -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc((void**)dev_nCells, sizeof(unsigned int) * (NUMINDEXEDDIM));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc nCells -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc((void**)dev_nNonEmptyCells, sizeof(unsigned int));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc nNonEmptyCells -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMalloc((void**)dev_gridCellNDMask, sizeof(unsigned int) * (*nNDMaskElems));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc gridCellNDMask -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }
    //
    // errCode = hipMalloc((void**)dev_gridCellNDMaskOffsets, sizeof(unsigned int) * (2 * NUMINDEXEDDIM));
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Alloc gridCellNDMaskOffsets -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    errCode = hipMalloc((void**)&dev_sortedDatabaseTmp, sizeof(struct schedulingCell) * (*nNonEmptyCells));
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: Alloc sortedSet -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    errCode = hipMalloc((void**)dev_originPointIndex, (*DBSIZE) * sizeof(unsigned int));
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: Alloc point index -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    double tEndAllocGPU = omp_get_wtime();
    cout << "[SORT] ~ Time to allocate on the GPU: " << tEndAllocGPU - tStartAllocGPU << '\n';
    cout.flush();


    // Memory copies needed by the GPU


    double tStartCopyGPU = omp_get_wtime();

    // errCode = hipMemcpy( (*dev_epsilon), epsilon, sizeof(DTYPE), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: epsilon copy to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_database), database, sizeof(DTYPE) * (GPUNUMDIM) * (*DBSIZE), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: database copy to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_index), index, sizeof(struct grid) * (*nNonEmptyCells), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: grid index copy to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }
    //
    // errCode = hipMemcpy( (*dev_indexLookupArr), indexLookupArr, sizeof(unsigned int) * (*DBSIZE), hipMemcpyHostToDevice);
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: copy lookup array to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }
    //
    // errCode = hipMemcpy( (*dev_gridCellLookupArr), gridCellLookupArr, sizeof(struct gridCellLookup) * (*nNonEmptyCells), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: copy grid cell lookup array to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_minArr), minArr, sizeof(DTYPE) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Copy minArr to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_nCells), nCells, sizeof(unsigned int) * (NUMINDEXEDDIM), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Copy nCells to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_nNonEmptyCells), nNonEmptyCells, sizeof(unsigned int), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: nNonEmptyCells copy to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    // errCode = hipMemcpy( (*dev_gridCellNDMask), gridCellNDMask, sizeof(unsigned int)*(*nNDMaskElems), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Copy gridCellNDMask to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }
    //
    // errCode = hipMemcpy( (*dev_gridCellNDMaskOffsets), gridCellNDMaskOffsets, sizeof(unsigned int) * (2 * NUMINDEXEDDIM), hipMemcpyHostToDevice );
	// if(errCode != hipSuccess)
    // {
	// 	cout << "[SORT] ~ Error: Copy gridCellNDMaskOffsets to device -- error with code " << errCode << '\n';
    //     cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
    //     cout.flush();
	// }

    double tEndCopyGPU = omp_get_wtime();
    cout << "[SORT] ~ Time to copy to the GPU: " << tEndCopyGPU - tStartCopyGPU << '\n';
    cout.flush();




    // Beginning of the sorting section
    double tStartSortingCells = omp_get_wtime();

    int nbBlock = ((*nNonEmptyCells) / BLOCKSIZE) + 1;
    cout << "[SORT] ~ Using " << nbBlock << " blocks of " << BLOCKSIZE << " threads to sort on the GPU\n";
    cout.flush();

    hipEventRecord(startKernel);
    #if UNICOMP
        sortByWorkLoadUnicomp<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), /*(*dev_gridCellNDMask), (*dev_gridCellNDMaskOffsets),*/
                dev_sortedDatabaseTmp);
    #elif LID_UNICOMP
        sortByWorkLoadLidUnicomp<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), /*(*dev_gridCellNDMask), (*dev_gridCellNDMaskOffsets),*/
                dev_sortedDatabaseTmp);
    #else
        cout << "[SORT] ~ Not using a cell access pattern to sort by workload\n";
        sortByWorkLoadGlobal<<<nbBlock, BLOCKSIZE>>>((*dev_database), (*dev_epsilon), (*dev_index),
                (*dev_indexLookupArr), (*dev_gridCellLookupArr), (*dev_minArr), (*dev_nCells),
                (*dev_nNonEmptyCells), /*(*dev_gridCellNDMask), (*dev_gridCellNDMaskOffsets),*/
                dev_sortedDatabaseTmp);
    #endif
    hipEventRecord(endKernel);

    errCode = hipGetLastError();
    cout << "[SORT] ~ SORTING KERNEL LAUNCH RETURN: " << errCode << '\n';
    cout.flush();

    hipDeviceSynchronize();

    errCode = hipMemcpy(sortedDatabaseTmp, dev_sortedDatabaseTmp, sizeof(struct schedulingCell) * (*nNonEmptyCells), hipMemcpyDeviceToHost);
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: copy sorted cells from the GPU -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    hipEventSynchronize(endKernel);
    float timeKernel = 0;
    hipEventElapsedTime(&timeKernel, startKernel, endKernel);
    cout << "[SORT] ~ Kernel time to sort by workload: " << timeKernel << '\n';
    cout.flush();

    double tBeginSort = omp_get_wtime();
    std::sort(sortedDatabaseTmp, sortedDatabaseTmp + (*nNonEmptyCells),
            [](const schedulingCell& a, const schedulingCell& b){ return a.nbPoints > b.nbPoints; });
    double tEndSort = omp_get_wtime();
    printf("[SORT] ~ Time to call std::sort = %f\n", tEndSort - tBeginSort);

    (*originPointIndex) = new unsigned int [(*DBSIZE)];

    // unsigned int maxNeighbor = 0;
    // unsigned int minNeighbor = (*DBSIZE);
    unsigned int maxNeighbor = sortedDatabaseTmp[0].nbPoints;
    unsigned int minNeighbor = sortedDatabaseTmp[(*nNonEmptyCells) - 1].nbPoints;
    cout << "max = " << maxNeighbor << '\n';
    cout << "min = " << minNeighbor << '\n';
    uint64_t accNeighbor = 0;

    unsigned int * nbNeighborPoints = new unsigned int[(*DBSIZE)];

    int prec = 0;
    for(int i = 0; i < (*nNonEmptyCells); ++i)
    {
        int cellId = sortedDatabaseTmp[i].cellId;
        int nbNeighbor = index[cellId].indexmax - index[cellId].indexmin + 1;

        // some stats about the number of neighbor
        accNeighbor += (nbNeighbor * sortedDatabaseTmp[i].nbPoints);

        for(int j = 0; j < nbNeighbor; ++j)
        {
            int tmpId = indexLookupArr[ index[cellId].indexmin + j ];
            nbNeighborPoints[tmpId] = nbNeighbor;
            (*originPointIndex)[prec + j] = tmpId;
        }
        prec += nbNeighbor;
    }

    // Setting some stuff for the CPU so it can begin immediately
    // and allocate buffers to store temp results
    // However, this value is way overestimated as it is set to the max,
    // while the CPU computes from the end of the queue.
    setMaxNeighbors(maxNeighbor);
    setWorkQueueReady();

    errCode = hipMemcpy( (*dev_originPointIndex), (*originPointIndex), (*DBSIZE) * sizeof(unsigned int), hipMemcpyHostToDevice);
    if(errCode != hipSuccess)
    {
        cout << "[SORT] ~ Error: point index copy -- error with code " << errCode << '\n';
        cout << "[SORT] ~   Details: " << hipGetErrorString(errCode) << '\n';
        cout.flush();
    }

    hipFree(dev_sortedDatabaseTmp);

    unsigned int decileMark = (*nNonEmptyCells) / 10;
    cout << "[SORT] ~ Total number of candidate points to refine: " << accNeighbor << '\n';
    cout << "[SORT] ~ Number of candidates: min = " << minNeighbor << ", median = " << sortedDatabaseTmp[(*nNonEmptyCells) / 2].nbPoints << ", max = " << maxNeighbor << ", avg = " << accNeighbor / (*DBSIZE) << '\n';
    cout << "[SORT] ~ Deciles number of candidates: \n";
    for(int i = 1; i < 10; ++i)
    {
        cout << "   [SORT] ~ " << i * 10 << "% = " << sortedDatabaseTmp[decileMark * i].nbPoints << '\n';
    }
    cout.flush();

    delete[] sortedDatabaseTmp;

    double tEndSortingCells = omp_get_wtime();

    cout << "[SORT] ~ Time to sort the cells by workload and copy to the GPU: " << tEndSortingCells - tStartSortingCells << '\n';
    cout.flush();

    cout << "\n\n\n";
    for(unsigned int i = 0; i < (*DBSIZE); ++i)
    {
        cout << nbNeighborPoints[i] << '\n';
    }
    cout << "\n\n\n";

    delete[] nbNeighborPoints;

}
